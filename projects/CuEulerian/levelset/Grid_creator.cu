#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/LevelSetUtils.tpp"
#include "zensim/geometry/SparseGrid.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

#include <zeno/VDBGrid.h>

#include "../utils.cuh"

namespace zeno {

struct ZSMakeSparseGrid : INode {
    void apply() override {
        auto attr = get_input2<std::string>("Attribute");
        auto dx = get_input2<float>("Dx");
        auto bg = get_input2<float>("background");
        auto type = get_input2<std::string>("type");
        auto structure = get_input2<std::string>("structure");

        auto zsSPG = std::make_shared<ZenoSparseGrid>();
        auto &spg = zsSPG->spg;

        int nc = 1;
        if (type == "scalar")
            nc = 1;
        else if (type == "vector3")
            nc = 3;

        spg = ZenoSparseGrid::spg_t{{{attr, nc}}, 0, zs::memsrc_e::device, 0};
        spg.scale(dx);
        spg._background = bg;

        if (structure == "vertex-centered") {
            auto trans = zs::vec<float, 3>::uniform(-dx / 2);
            // zs::vec<float, 3> trans{-dx / 2.f, -dx / 2.f, -dx / 2.f};

            spg.translate(trans);
        }

        set_output("Grid", zsSPG);
    }
};

ZENDEFNODE(ZSMakeSparseGrid, {/* inputs: */
                              {{"string", "Attribute", ""},
                               {"float", "Dx", "1.0"},
                               {"float", "background", "0"},
                               {"enum scalar vector3", "type", "scalar"},
                               {"enum cell-centered vertex-centered", "structure", "cell-centered "}},
                              /* outputs: */
                              {"Grid"},
                              /* params: */
                              {},
                              /* category: */
                              {"Eulerian"}});

struct ZSGridTopoCopy : INode {
    void apply() override {
        auto zs_grid = get_input<ZenoSparseGrid>("Grid");
        auto zs_topo = get_input<ZenoSparseGrid>("TopologyGrid");

        auto &grid = zs_grid->spg;
        auto &topo = zs_topo->spg;

        // topo copy
        grid._table = topo._table;
        grid._transform = topo._transform;
        grid._grid.resize(topo.numBlocks() * topo.block_size);

        set_output("Grid", zs_grid);
    }
};

ZENDEFNODE(ZSGridTopoCopy, {/* inputs: */
                            {"Grid", "TopologyGrid"},
                            /* outputs: */
                            {"Grid"},
                            /* params: */
                            {},
                            /* category: */
                            {"Eulerian"}});

struct ZSSparseGridToVDB : INode {
    void apply() override {
        auto zs_grid = get_input<ZenoSparseGrid>("SparseGrid");
        auto attr = get_input2<std::string>("Attribute");
        auto VDBGridClass = get_input2<std::string>("VDBGridClass");

        if (attr.empty())
            attr = "sdf";

        auto &spg = zs_grid->spg;

        auto attrTag = src_tag(zs_grid, attr);

        if (attr == "v") {
            auto vdb_ = zs::convert_sparse_grid_to_float3grid(spg, attrTag);
            auto vdb_grid = std::make_shared<VDBFloat3Grid>();
            vdb_grid->m_grid = vdb_.as<openvdb::Vec3fGrid::Ptr>();

            set_output("VDB", vdb_grid);
        } else {
            zs::u32 gridClass = 0;
            if (VDBGridClass == "UNKNOWN")
                gridClass = 0;
            else if (VDBGridClass == "LEVEL_SET")
                gridClass = 1;
            else if (VDBGridClass == "FOG_VOLUME")
                gridClass = 2;
            else if (VDBGridClass == "STAGGERED")
                gridClass = 3;

            auto vdb_ = zs::convert_sparse_grid_to_floatgrid(spg, attrTag, gridClass);

            auto vdb_grid = std::make_shared<VDBFloatGrid>();
            vdb_grid->m_grid = vdb_.as<openvdb::FloatGrid::Ptr>();

            set_output("VDB", vdb_grid);
        }
    }
};

ZENDEFNODE(ZSSparseGridToVDB, {/* inputs: */
                               {"SparseGrid",
                                {"string", "Attribute", ""},
                                {"enum UNKNOWN LEVEL_SET FOG_VOLUME STAGGERED", "VDBGridClass", "LEVEL_SET"}},
                               /* outputs: */
                               {"VDB"},
                               /* params: */
                               {},
                               /* category: */
                               {"Eulerian"}});

struct ZSVDBToSparseGrid : INode {
    void apply() override {
        auto vdb = get_input<VDBGrid>("VDB");
        auto attr = get_input2<std::string>("Attribute");
        if (attr.empty())
            attr = "sdf";

        if (has_input("SparseGrid")) {
            auto zs_grid = get_input<ZenoSparseGrid>("SparseGrid");
            auto &spg = zs_grid->spg;

            int num_ch;
            if (vdb->getType() == "FloatGrid")
                num_ch = 1;
            else if (vdb->getType() == "Vec3fGrid")
                num_ch = 3;
            else
                throw std::runtime_error("Input VDB must be a FloatGrid or Vec3fGrid!");

            auto attrTag = src_tag(zs_grid, attr);
            if (spg.hasProperty(attrTag)) {
                if (num_ch != spg.getPropertySize(attrTag)) {
                    throw std::runtime_error(fmt::format("The channel number of [{}] doesn't match!", attr));
                }
            } else {
                spg.append_channels(zs::cuda_exec(), {{attrTag, num_ch}});
            }

            if (num_ch == 1) {
                auto vdb_ = std::dynamic_pointer_cast<VDBFloatGrid>(vdb);
                zs::assign_floatgrid_to_sparse_grid(vdb_->m_grid, spg, attrTag);
            } else {
                auto vdb_ = std::dynamic_pointer_cast<VDBFloat3Grid>(vdb);
                zs::assign_float3grid_to_sparse_grid(vdb_->m_grid, spg, attrTag);
            }

            set_output("SparseGrid", zs_grid);
        } else {
            ZenoSparseGrid::spg_t spg;

            auto vdbType = vdb->getType();
            if (vdbType == "FloatGrid") {
                auto vdb_ = std::dynamic_pointer_cast<VDBFloatGrid>(vdb);
                spg =
                    zs::convert_floatgrid_to_sparse_grid(vdb_->m_grid, zs::MemoryHandle{zs::memsrc_e::device, 0}, attr);
            } else if (vdbType == "Vec3fGrid") {
                auto vdb_ = std::dynamic_pointer_cast<VDBFloat3Grid>(vdb);
                spg = zs::convert_float3grid_to_sparse_grid(vdb_->m_grid, zs::MemoryHandle{zs::memsrc_e::device, 0},
                                                            attr);
            } else {
                throw std::runtime_error("Input VDB must be a FloatGrid or Vec3fGrid!");
            }

            auto zsSPG = std::make_shared<ZenoSparseGrid>();
            zsSPG->spg = std::move(spg);

            set_output("SparseGrid", zsSPG);
        }
    }
};

ZENDEFNODE(ZSVDBToSparseGrid, {/* inputs: */
                               {"VDB", "SparseGrid", {"string", "Attribute", ""}},
                               /* outputs: */
                               {"SparseGrid"},
                               /* params: */
                               {},
                               /* category: */
                               {"Eulerian"}});

struct ZSGridVoxelSize : INode {
    void apply() override {
        auto zs_grid = get_input<ZenoSparseGrid>("SparseGrid");

        float dx = zs_grid->getSparseGrid().voxelSize()[0];

        set_output("dx", std::make_shared<NumericObject>(dx));
    }
};

ZENDEFNODE(ZSGridVoxelSize, {/* inputs: */
                             {"SparseGrid"},
                             /* outputs: */
                             {"dx"},
                             /* params: */
                             {},
                             /* category: */
                             {"Eulerian"}});

struct ZSMakeDenseSDF : INode {
    void apply() override {
        float dx = get_input2<float>("dx");
        int nx = get_input2<int>("nx");
        int ny = get_input2<int>("ny");
        int nz = get_input2<int>("nz");

        int nbx = float(nx + 7) / 8.f;
        int nby = float(ny + 7) / 8.f;
        int nbz = float(nz + 7) / 8.f;

        size_t numExpectedBlocks = nbx * nby * nbz;

        auto zsSPG = std::make_shared<ZenoSparseGrid>();
        auto &spg = zsSPG->spg;
        spg = ZenoSparseGrid::spg_t{{{"sdf", 1}}, numExpectedBlocks, zs::memsrc_e::device, 0};
        spg.scale(dx);
        spg._background = dx;

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;
        using ivec3 = zs::vec<int, 3>;

        pol(zs::range(numExpectedBlocks),
            [table = zs::proxy<space>(spg._table), nbx, nby, nbz] __device__(int nb) mutable {
                int i = nb / (nby * nbz);
                nb -= i * (nby * nbz);
                int j = nb / nbz;
                int k = nb - j * nbz;
                table.insert(ivec3{int(i - nbx / 2) * 8, int(j - nby / 2) * 8, int(k - nbz / 2) * 8});
            });

        ivec3 sphere_c{0, 0, 0};
        int sphere_r = 10; // 10*dx

        auto bcnt = spg.numBlocks();
        pol(zs::range(bcnt * 512), [spgv = zs::proxy<space>(spg), sphere_c, sphere_r] __device__(int cellno) mutable {
#if 0            
			int bno = cellno / 512;
            int cno = cellno & 511;
            auto bcoord = spgv._table._activeKeys[bno];
            auto cellid = RM_CVREF_T(spgv)::local_offset_to_coord(cno);
            auto ccoord = bcoord + cellid;
#endif
            auto icoord = spgv.iCoord(cellno);
            auto dx = spgv.voxelSize()[0]; // spgv._transform(0, 0);

            float dist2c = zs::sqrt(float(zs::sqr(icoord[0] - sphere_c[0]) + zs::sqr(icoord[1] - sphere_c[1]) +
                                          zs::sqr(icoord[2] - sphere_c[2])));
            float dist2s = dist2c - sphere_r;

            float init_sdf = dist2s;
            if (dist2s > 2. * dx)
                init_sdf = 2. * dx;
            else if (dist2s < -2. * dx)
                init_sdf = -2. * dx;

            //spgv("sdf", bno, cno) = ;
            spgv("sdf", icoord) = init_sdf;
        });

        // spg.resize(numExpectedBlocks);

        spg.append_channels(pol, {{"v", 3}});

        set_output("Grid", zsSPG);
    }
};

ZENDEFNODE(ZSMakeDenseSDF, {/* inputs: */
                            {{"float", "dx", "1.0"}, {"int", "nx", "128"}, {"int", "ny", "128"}, {"int", "nz", "128"}},
                            /* outputs: */
                            {"Grid"},
                            /* params: */
                            {},
                            /* category: */
                            {"deprecated"}});

} // namespace zeno