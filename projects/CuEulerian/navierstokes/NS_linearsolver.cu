#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/LevelSetUtils.tpp"
#include "zensim/geometry/SparseGrid.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/profile/CppTimers.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

#include <zeno/VDBGrid.h>

#include "../scheme.hpp"
#include "../utils.cuh"

namespace zeno {

struct ZSNSPressureProject : INode {

    template <int level> void clearInit(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid) {
        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->getLevel<level>();
        auto block_cnt = spg.numBlocks();

        // take zero as initial guess
        pol(zs::range(block_cnt * spg.block_size), [spgv = zs::proxy<space>(spg)] __device__(int cellno) mutable {
            auto icoord = spgv.iCoord(cellno);
            spgv("p0", icoord) = 0.f;
        });
    }

    template <int level, typename Ti>
    void PostJacobi(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid, Ti block_cnt, int cur) {
        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;
        auto &spg = NSGrid->getLevel<level>();
        // workaround: write result to "p0" for high level grid
        if (cur == 0)
            return;
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg),
             pSrcTag = zs::SmallString{std::string("p") + std::to_string(cur)}] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);
                spgv("p0", icoord) = spgv(pSrcTag, icoord);
            });
    }

    template <int level> void Jacobi(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid, float rho, int nIter) {
        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->getLevel<level>();
        auto block_cnt = spg.numBlocks();

        auto dx = spg.voxelSize()[0];

        int cur = level == 0 ? NSGrid->readMeta<int>("p_cur") : 0;

        for (int iter = 0; iter < nIter; ++iter) {
            // point Jacobi iteration
            pol(zs::range(block_cnt * spg.block_size),
                [spgv = zs::proxy<space>(spg), dx, rho,
                 pSrcTag = zs::SmallString{std::string("p") + std::to_string(cur)},
                 pDstTag = zs::SmallString{std::string("p") + std::to_string(cur ^ 1)}] __device__(int cellno) mutable {
                    auto icoord = spgv.iCoord(cellno);

                    float div = spgv.value("tmp", icoord);

                    const int stcl = 1; // stencil point in each side
                    float p_x[2 * stcl + 1], p_y[2 * stcl + 1], p_z[2 * stcl + 1];

                    for (int i = -stcl; i <= stcl; ++i) {
                        p_x[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(i, 0, 0));
                        p_y[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, i, 0));
                        p_z[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, 0, i));
                    }

                    float p_this = -(div * dx * dx * rho - (p_x[0] + p_x[2] + p_y[0] + p_y[2] + p_z[0] + p_z[2])) / 6.f;

                    spgv(pDstTag, icoord) = p_this;
                });
            cur ^= 1;
        }
        if constexpr (level == 0)
            NSGrid->setMeta("p_cur", cur);
        else {
            PostJacobi<level>(pol, NSGrid, block_cnt, cur);
        }
    }

    template <int level>
    void coloredSOR(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid, float rho, float sor, int nIter) {
        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->getLevel<level>();
        constexpr int tile_size = zs::math::min((int)RM_CVREF_T(spg)::block_size, (int)32);
        auto block_cnt = spg.numBlocks();

        auto dx = spg.voxelSize()[0];

        for (int iter = 0; iter < nIter; ++iter) {
            for (int clr = 0; clr != 2; ++clr) {

                pol(zs::range(block_cnt * tile_size), [spgv = zs::proxy<space>(spg), dx, rho, clr, sor,
                                                       ts_c = zs::wrapv<tile_size>{}] __device__(int tid) mutable {
                    constexpr int side_length = RM_CVREF_T(spgv)::side_length;
                    constexpr int half_block_size = RM_CVREF_T(spgv)::block_size / 2;
                    constexpr int tile_size = decltype(ts_c)::value;
                    auto tile = zs::cg::tiled_partition<tile_size>(zs::cg::this_thread_block());
                    auto blockno = tid / tile_size;

                    auto bcoord = spgv._table._activeKeys[blockno];
                    if ((((bcoord[0] & side_length) ^ (bcoord[1] & side_length) ^ (bcoord[2] & side_length)) >> 3) ==
                        clr)
                        return;

                    for (int c_clr = 0; c_clr != 2; ++c_clr) {

                        for (int cno = tile.thread_rank(); cno < half_block_size; cno += tile_size) {
                            auto cellno = (cno << 1) | c_clr;

                            auto ccoord = spgv.local_offset_to_coord(cellno);
                            auto icoord = bcoord + ccoord;

                            float div = spgv.value("tmp", blockno, cellno);

                            const int stcl = 1; // stencil point in each side
                            float p_x[2 * stcl + 1], p_y[2 * stcl + 1], p_z[2 * stcl + 1];

                            for (int i = -stcl; i <= stcl; ++i) {
                                p_x[i + stcl] = spgv.value("p0", icoord + zs::vec<int, 3>(i, 0, 0));
                                p_y[i + stcl] = spgv.value("p0", icoord + zs::vec<int, 3>(0, i, 0));
                                p_z[i + stcl] = spgv.value("p0", icoord + zs::vec<int, 3>(0, 0, i));
                            }

                            float p_this =
                                (1.f - sor) * p_x[stcl] +
                                sor * ((p_x[0] + p_x[2] + p_y[0] + p_y[2] + p_z[0] + p_z[2]) - div * dx * dx * rho) /
                                    6.f;

                            spgv("p0", blockno, cellno) = p_this;
                        }
                    }
                });
            }
        }
    }

    template <int level> float residual(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid, float rho) {
        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->getLevel<level>();
        auto block_cnt = spg.numBlocks();

        auto dx = spg.voxelSize()[0];

        int cur = level == 0 ? NSGrid->readMeta<int>("p_cur") : 0;

        // residual
        size_t cell_cnt = block_cnt * spg.block_size;
        zs::Vector<float> res{spg.get_allocator(), count_warps(cell_cnt)};
        res.reset(0);

        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), res = zs::proxy<space>(res), cell_cnt, dx, rho,
             pSrcTag = zs::SmallString{std::string("p") + std::to_string(cur)}] __device__(int blockno,
                                                                                           int cellno) mutable {
                auto icoord = spgv.iCoord(blockno, cellno);

                float div = spgv.value("tmp", icoord);

                const int stcl = 1; // stencil point in each side
                float p_x[2 * stcl + 1], p_y[2 * stcl + 1], p_z[2 * stcl + 1];

                for (int i = -stcl; i <= stcl; ++i) {
                    p_x[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(i, 0, 0));
                    p_y[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, i, 0));
                    p_z[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, 0, i));
                }

                float m_residual = div - (scheme::central_diff_2nd(p_x[0], p_x[1], p_x[2], dx) +
                                          scheme::central_diff_2nd(p_y[0], p_y[1], p_y[2], dx) +
                                          scheme::central_diff_2nd(p_z[0], p_z[1], p_z[2], dx)) /
                                             rho;

                spgv("tmp", 1, blockno, cellno) = m_residual;

                size_t cellno_glb = blockno * spgv.block_size + cellno;

                reduce_max(cellno_glb, cell_cnt, zs::abs(m_residual), res[cellno_glb / 32]);
            });

        float max_residual = reduce(pol, res, thrust::maximum<float>{});
        return max_residual;
    }

    template <int level> void restriction(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid) {
        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg_f = NSGrid->getLevel<level>();
        auto &spg_c = NSGrid->getLevel<level + 1>();

        pol(zs::Collapse{spg_c.numBlocks(), spg_c.block_size},
            [spgv_c = zs::proxy<space>(spg_c), spgv_f = zs::proxy<space>(spg_f)] __device__(int blockno,
                                                                                            int cellno) mutable {
                auto bcoord = spgv_c._table._activeKeys[blockno];
                auto ccoord = spgv_c.local_offset_to_coord(cellno);

                auto icoord_c = bcoord + ccoord;
                auto icoord_f = bcoord * 2 + ccoord * 2;

                float res_sum = 0;
                for (int k = 0; k < 2; ++k)
                    for (int j = 0; j < 2; ++j)
                        for (int i = 0; i < 2; ++i) {
                            res_sum += spgv_f.value("tmp", 1, icoord_f + zs::vec<int, 3>(i, j, k));
                        }

                spgv_c("tmp", icoord_c) = res_sum / 8.f;
            });
    }

    template <int level> void prolongation(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid) {
        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg_f = NSGrid->getLevel<level>();
        auto &spg_c = NSGrid->getLevel<level + 1>();

        int cur = level == 0 ? NSGrid->readMeta<int>("p_cur") : 0;

        pol(zs::Collapse{spg_f.numBlocks(), spg_f.block_size},
            [spgv_f = zs::proxy<space>(spg_f), spgv_c = zs::proxy<space>(spg_c),
             pSrcTag = zs::SmallString{std::string("p") + std::to_string(cur)}] __device__(int blockno,
                                                                                           int cellno) mutable {
                auto bcoord = spgv_f._table._activeKeys[blockno];
                auto ccoord = spgv_f.local_offset_to_coord(cellno);

                auto icoord_f = bcoord + ccoord;
                auto icoord_c = bcoord / 2 + ccoord / 2;

                spgv_f(pSrcTag, icoord_f) += spgv_c("p0", icoord_c);
            });
    }

    template <int level> void multigrid(zs::CudaExecutionPolicy &pol, ZenoSparseGrid *NSGrid, float rho) {
        if constexpr (level == 3) {
            clearInit<level>(pol, NSGrid);
            Jacobi<level>(pol, NSGrid, rho, 100);
            float res = residual<level>(pol, NSGrid, rho);
            printf("MG level %d residual: %e\n", level, res);
        } else {
            if constexpr (level != 0)
                clearInit<level>(pol, NSGrid);
            coloredSOR<level>(pol, NSGrid, rho, 1.2f, 4);
            float res = residual<level>(pol, NSGrid, rho);
            printf("MG level %d residual: %e\n", level, res);
            restriction<level>(pol, NSGrid);

            multigrid<level + 1>(pol, NSGrid, rho);

            prolongation<level>(pol, NSGrid);
            coloredSOR<level>(pol, NSGrid, rho, 1.2f, 4);
        }
        return;
    }

    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto rho = get_input2<float>("Density");
        auto dt = get_input2<float>("dt");
        auto maxIter = get_input2<int>("MaxIterations");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();
        auto dx = spg.voxelSize()[0];

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        // velocity divergence (source term)
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v")] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);

                float u_x[2], u_y[2], u_z[2];
                for (int i = 0; i <= 1; ++i) {
                    u_x[i] = spgv.value(vSrcTag, 0, icoord + zs::vec<int, 3>(i, 0, 0));
                    u_y[i] = spgv.value(vSrcTag, 1, icoord + zs::vec<int, 3>(0, i, 0));
                    u_z[i] = spgv.value(vSrcTag, 2, icoord + zs::vec<int, 3>(0, 0, i));
                }

                float div_term = ((u_x[1] - u_x[0]) + (u_y[1] - u_y[0]) + (u_z[1] - u_z[0])) / dx / dt;

                spgv("tmp", icoord) = div_term;
            });

        // Multi-grid solver with V-Cycle
        const float tolerence = 1e-6;
        printf("========MultiGrid V-cycle Begin========\n");
        for (int iter = 0; iter < maxIter; ++iter) {
            printf("-----%dth V-cycle-----\n", iter);
            multigrid<0>(pol, NSGrid.get(), rho);
            float res = residual<0>(pol, NSGrid.get(), rho);
            if (res < tolerence)
                break;
        }
        printf("========MultiGrid V-cycle End========\n");

        // pressure projection
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt, rho, vSrcTag = src_tag(NSGrid, "v"), vDstTag = dst_tag(NSGrid, "v"),
             pSrcTag = src_tag(NSGrid, "p")] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);
                float p_this = spgv.value(pSrcTag, icoord);

                for (int ch = 0; ch < 3; ++ch) {
                    float u = spgv.value(vSrcTag, ch, icoord);

                    zs::vec<int, 3> offset{0, 0, 0};
                    offset[ch] = -1;

                    float p_m = spgv.value(pSrcTag, icoord + offset);

                    u -= (p_this - p_m) / dx * dt / rho;

                    spgv(vDstTag, ch, icoord) = u;
                }
            });
        update_cur(NSGrid, "v");

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSNSPressureProject, {/* inputs: */
                                 {"NSGrid", "dt", {"float", "Density", "1.0"}, {"int", "MaxIterations", "10"}},
                                 /* outputs: */
                                 {"NSGrid"},
                                 /* params: */
                                 {},
                                 /* category: */
                                 {"Eulerian"}});

} // namespace zeno