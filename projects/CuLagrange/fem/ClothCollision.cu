#include "hip/hip_runtime.h"
#include "Cloth.cuh"
#include "TopoUtils.hpp"
#include "collision_energy/vertex_face_sqrt_collision.hpp"
#include "zensim/Logger.hpp"
#include "zensim/geometry/Distance.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include <zeno/core/INode.h>
#include <zeno/types/ListObject.h>
#include <zeno/utils/log.h>
#include <zeno/zeno.h>

namespace zeno {

void ClothSystem::findCollisionConstraints(zs::CudaExecutionPolicy &pol, T dHat) {
    // nPP.setVal(0);
    // nPE.setVal(0);
    // nPT.setVal(0);
    // nEE.setVal(0);

    ncsPT.setVal(0);
    ncsEE.setVal(0);

    zs::CppTimer timer;
    timer.tick();
    if (enableContactSelf) {
        auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, 0);
        stBvh.refit(pol, triBvs);
        auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", seInds, zs::wrapv<2>{}, 0);
        seBvh.refit(pol, edgeBvs);
        findCollisionConstraintsImpl(pol, dHat, false);
    }

    if (coVerts)
        if (coVerts->size()) {
            auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, coOffset);
            bouStBvh.refit(pol, triBvs);
            auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, coOffset);
            bouSeBvh.refit(pol, edgeBvs);
#if 1
            findCollisionConstraintsImpl(pol, dHat, true);
#else
            findBoundaryCellCollisionConstraints(pol, dHat);
#endif
        }
    auto [npt, nee] = getCollisionCnts();
    timer.tock(fmt::format("dcd broad phase [pt, ee]({}, {})", npt, nee));

    frontManageRequired = false;
}

#define PROFILE_CD 0

void ClothSystem::findBoundaryCellCollisionConstraints(zs::CudaExecutionPolicy &pol, T dHat) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    update_surface_cell_normals(pol, vtemp, "xn", coOffset, *coEles, "nrm", *coEdges, "nrm");

    pol.profile(PROFILE_CD);
    /// pt
    const auto &stbvh = bouStBvh;
    auto &stfront = boundaryStFront;
    pol(Collapse{stfront.size()},
        [svInds = proxy<space>({}, svInds), eles = proxy<space>({}, *coEles), bouSes = proxy<space>({}, *coEdges),
         exclTris = proxy<space>(exclBouSts), vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(stbvh),
         front = proxy<space>(stfront), csPT = proxy<space>(csPT), ncsPT = proxy<space>(ncsPT), dHat2 = dHat * dHat,
         out_collisionEps = dHat, in_collisionEps = dHat * 10, voffset = coOffset,
         frontManageRequired = frontManageRequired] __device__(int i) mutable {
            auto vi = front.prim(i);
            vi = reinterpret_bits<int>(svInds("inds", vi));
            auto p = vtemp.pack(dim_c<3>, "xn", vi);
            auto bv = bv_t{get_bounding_box(p - in_collisionEps, p + in_collisionEps)};
            auto f = [&](int stI) {
                if (exclTris[stI])
                    return;
                auto tri = eles.pack(dim_c<3>, "inds", stI).reinterpret_bits(int_c) + voffset;
                if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                    return;

                auto triNrm = eles.pack(dim_c<3>, "nrm", stI);
                const zs::vec<float, 3> ts[3] = {vtemp.pack(dim_c<3>, "xn", tri[0]), vtemp.pack(dim_c<3>, "xn", tri[1]),
                                                 vtemp.pack(dim_c<3>, "xn", tri[2])};
                const auto &t0 = ts[0];
                const auto &t1 = ts[1];
                const auto &t2 = ts[2];

                auto seg = p - t0;
                auto e01 = (t0 - t1).norm();
                auto e02 = (t0 - t2).norm();
                auto e12 = (t1 - t2).norm();
                float barySum = (float)1.0;
                float distance = COLLISION_UTILS::pointTriangleDistance(t0, t1, t2, p, barySum);

                auto collisionEps = seg.dot(triNrm) > 0 ? out_collisionEps : in_collisionEps;

                if (barySum > 2)
                    return;
                if (distance > collisionEps)
                    return;

                // if the triangle cell is too degenerate
                auto get_bisector_orient = [&](int j) {
                    auto ej = reinterpret_bits<int>(eles("fe_inds", j, stI));
                    auto line = bouSes.pack(dim_c<2>, "inds", ej).reinterpret_bits(int_c) + voffset;
                    auto tline = zs::vec<int, 2>{tri[j], tri[(j + 1) % 3]};
                    auto biNrm = bouSes.pack(dim_c<3>, "nrm", ej);
                    if (tline[1] == line[0] && tline[0] == line[1])
                        biNrm *= -1;
                    return biNrm;
                };
                if (!COLLISION_UTILS::pointProjectsInsideTriangle(t0, t1, t2, p))
                    for (int i = 0; i != 3; ++i) {
                        auto bisector_normal = get_bisector_orient(i);
                        if (bisector_normal.dot(seg) < 0)
                            return;
                        if (i != 2)
                            seg = p - ts[i + 1];
                    }
                // now the points is inside the cell
                csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};

                // dist = seg.dot(triNrm);
            };
            if (frontManageRequired)
                bvh.iter_neighbors(bv, i, front, f);
            else
                bvh.iter_neighbors(bv, front.node(i), f);
        });
    if (frontManageRequired)
        stfront.reorder(pol);
    /// ee
    if (enableContactEE) {
        const auto &sebvh = bouSeBvh;
        auto &sefront = boundarySeFront;
        pol(Collapse{sefront.size()},
            [seInds = proxy<space>({}, seInds), sedges = proxy<space>({}, *coEdges), exclSes = proxy<space>(exclSes),
             vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(sebvh), front = proxy<space>(sefront),
             csEE = proxy<space>(csEE), ncsEE = proxy<space>(ncsEE), dHat2 = dHat * dHat, thickness = dHat,
             voffset = coOffset, frontManageRequired = frontManageRequired] __device__(int i) mutable {
                auto sei = front.prim(i);
                if (exclSes[sei])
                    return;
                auto eiInds = seInds.pack(dim_c<2>, "inds", sei).reinterpret_bits(int_c);
                auto v0 = vtemp.pack(dim_c<3>, "xn", eiInds[0]);
                auto v1 = vtemp.pack(dim_c<3>, "xn", eiInds[1]);
                auto [mi, ma] = get_bounding_box(v0, v1);
                auto bv = bv_t{mi - thickness, ma + thickness};
                auto f = [&](int sej) {
                    if (voffset == 0 && sei < sej) // only check this for self intersection
                        return;
                    auto ejInds = sedges.pack(dim_c<2>, "inds", sej).reinterpret_bits(int_c) + voffset;
                    if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] || eiInds[1] == ejInds[0] ||
                        eiInds[1] == ejInds[1])
                        return;
                    auto v2 = vtemp.pack(dim_c<3>, "xn", ejInds[0]);
                    auto v3 = vtemp.pack(dim_c<3>, "xn", ejInds[1]);

                    // ee_distance_type(v0, v1, v2, v3);
                    // csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                };
                if (frontManageRequired)
                    bvh.iter_neighbors(bv, i, front, f);
                else
                    bvh.iter_neighbors(bv, front.node(i), f);
            });
        if (frontManageRequired)
            sefront.reorder(pol);
    }
    pol.profile(false);
}

void ClothSystem::findCollisionConstraintsImpl(zs::CudaExecutionPolicy &pol, T dHat, bool withBoundary) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    pol.profile(PROFILE_CD);
    /// pt
    const auto &stbvh = withBoundary ? bouStBvh : stBvh;
    auto &stfront = withBoundary ? boundaryStFront : selfStFront;
    pol(Collapse{stfront.size()},
        [svInds = proxy<space>({}, svInds), eles = proxy<space>({}, withBoundary ? *coEles : stInds),
         exclTris = withBoundary ? proxy<space>(exclBouSts) : proxy<space>(exclSts), vtemp = proxy<space>({}, vtemp),
         bvh = proxy<space>(stbvh), front = proxy<space>(stfront),
         // PP = proxy<space>(PP), nPP = proxy<space>(nPP), PE = proxy<space>(PE),
         // nPE = proxy<space>(nPE), PT = proxy<space>(PT), nPT = proxy<space>(nPT),
         csPT = proxy<space>(csPT), ncsPT = proxy<space>(ncsPT), dHat2 = dHat * dHat, thickness = dHat,
         voffset = withBoundary ? coOffset : 0, frontManageRequired = frontManageRequired] __device__(int i) mutable {
            auto vi = front.prim(i);
            vi = reinterpret_bits<int>(svInds("inds", vi));
            auto p = vtemp.pack(dim_c<3>, "xn", vi);
            auto bv = bv_t{get_bounding_box(p - thickness, p + thickness)};
            auto f = [&](int stI) {
                if (exclTris[stI])
                    return;
                auto tri = eles.pack(dim_c<3>, "inds", stI).reinterpret_bits(int_c) + voffset;
                if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                    return;
                auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
                auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
                auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);

                switch (pt_distance_type(p, t0, t1, t2)) {
#if 0
                case 0: {
                    if (auto d2 = dist2_pp(p, t0); d2 < dHat2) {
                        //auto no = atomic_add(exec_cuda, &nPP[0], 1);
                        //PP[no] = pair_t{vi, tri[0]};
                        csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};
                    }
                    break;
                }
                case 1: {
                    if (auto d2 = dist2_pp(p, t1); d2 < dHat2) {
                        //auto no = atomic_add(exec_cuda, &nPP[0], 1);
                        //PP[no] = pair_t{vi, tri[1]};
                        csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};
                    }
                    break;
                }
                case 2: {
                    if (auto d2 = dist2_pp(p, t2); d2 < dHat2) {
                        //auto no = atomic_add(exec_cuda, &nPP[0], 1);
                        //PP[no] = pair_t{vi, tri[2]};
                        csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};
                    }
                    break;
                }
                case 3: {
                    if (auto d2 = dist2_pe(p, t0, t1); d2 < dHat2) {
                        //auto no = atomic_add(exec_cuda, &nPE[0], 1);
                        //PE[no] = pair3_t{vi, tri[0], tri[1]};
                        csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};
                    }
                    break;
                }
                case 4: {
                    if (auto d2 = dist2_pe(p, t1, t2); d2 < dHat2) {
                        //auto no = atomic_add(exec_cuda, &nPE[0], 1);
                        //PE[no] = pair3_t{vi, tri[1], tri[2]};
                        csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};
                    }
                    break;
                }
                case 5: {
                    if (auto d2 = dist2_pe(p, t2, t0); d2 < dHat2) {
                        //auto no = atomic_add(exec_cuda, &nPE[0], 1);
                        //PE[no] = pair3_t{vi, tri[2], tri[0]};
                        csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};
                    }
                    break;
                }
#endif
                case 6: {
                    if (auto d2 = dist2_pt(p, t0, t1, t2); d2 < dHat2) {
                        //auto no = atomic_add(exec_cuda, &nPT[0], 1);
                        //PT[no] = pair4_t{vi, tri[0], tri[1], tri[2]};
                        csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] = pair4_t{vi, tri[0], tri[1], tri[2]};
                    }
                    break;
                }
                default: break;
                }
            };
            if (frontManageRequired)
                bvh.iter_neighbors(bv, i, front, f);
            else
                bvh.iter_neighbors(bv, front.node(i), f);
        });
    if (frontManageRequired)
        stfront.reorder(pol);
    /// ee
    if (enableContactEE) {
        const auto &sebvh = withBoundary ? bouSeBvh : seBvh;
        auto &sefront = withBoundary ? boundarySeFront : selfSeFront;
        pol(Collapse{sefront.size()}, [seInds = proxy<space>({}, seInds),
                                       sedges = proxy<space>({}, withBoundary ? *coEdges : seInds),
                                       exclSes = proxy<space>(exclSes), vtemp = proxy<space>({}, vtemp),
                                       bvh = proxy<space>(sebvh), front = proxy<space>(sefront),
                                       // PP = proxy<space>(PP), nPP = proxy<space>(nPP), PE = proxy<space>(PE),
                                       // nPE = proxy<space>(nPE), EE = proxy<space>(EE), nEE = proxy<space>(nEE),
                                       //
                                       csEE = proxy<space>(csEE), ncsEE = proxy<space>(ncsEE), dHat2 = dHat * dHat,
                                       thickness = dHat, voffset = withBoundary ? coOffset : 0,
                                       frontManageRequired = frontManageRequired] __device__(int i) mutable {
            auto sei = front.prim(i);
            if (exclSes[sei])
                return;
            auto eiInds = seInds.pack(dim_c<2>, "inds", sei).reinterpret_bits(int_c);
            auto v0 = vtemp.pack(dim_c<3>, "xn", eiInds[0]);
            auto v1 = vtemp.pack(dim_c<3>, "xn", eiInds[1]);
            auto [mi, ma] = get_bounding_box(v0, v1);
            auto bv = bv_t{mi - thickness, ma + thickness};
            auto f = [&](int sej) {
                if (voffset == 0 && sei < sej) // only check this for self intersection
                    return;
                auto ejInds = sedges.pack(dim_c<2>, "inds", sej).reinterpret_bits(int_c) + voffset;
                if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] || eiInds[1] == ejInds[0] ||
                    eiInds[1] == ejInds[1])
                    return;
                auto v2 = vtemp.pack(dim_c<3>, "xn", ejInds[0]);
                auto v3 = vtemp.pack(dim_c<3>, "xn", ejInds[1]);

                switch (ee_distance_type(v0, v1, v2, v3)) {
                case 0: {
                    if (auto d2 = dist2_pp(v0, v2); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPP[0], 1);
                        //   PP[no] = pair_t{eiInds[0], ejInds[0]};
                    }
                    break;
                }
                case 1: {
                    if (auto d2 = dist2_pp(v0, v3); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPP[0], 1);
                        //   PP[no] = pair_t{eiInds[0], ejInds[1]};
                    }
                    break;
                }
                case 2: {
                    if (auto d2 = dist2_pe(v0, v2, v3); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPE[0], 1);
                        //   PE[no] = pair3_t{eiInds[0], ejInds[0], ejInds[1]};
                    }
                    break;
                }
                case 3: {
                    if (auto d2 = dist2_pp(v1, v2); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPP[0], 1);
                        //   PP[no] = pair_t{eiInds[1], ejInds[0]};
                    }
                    break;
                }
                case 4: {
                    if (auto d2 = dist2_pp(v1, v3); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPP[0], 1);
                        //   PP[no] = pair_t{eiInds[1], ejInds[1]};
                    }
                    break;
                }
                case 5: {
                    if (auto d2 = dist2_pe(v1, v2, v3); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPE[0], 1);
                        // PE[no] = pair3_t{eiInds[1], ejInds[0], ejInds[1]};
                    }
                    break;
                }
                case 6: {
                    if (auto d2 = dist2_pe(v2, v0, v1); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPE[0], 1);
                        // PE[no] = pair3_t{ejInds[0], eiInds[0], eiInds[1]};
                    }
                    break;
                }
                case 7: {
                    if (auto d2 = dist2_pe(v3, v0, v1); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nPE[0], 1);
                        // PE[no] = pair3_t{ejInds[1], eiInds[0], eiInds[1]};
                    }
                    break;
                }
                case 8: {
                    if (auto d2 = dist2_ee(v0, v1, v2, v3); d2 < dHat2) {
                        csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                        // auto no = atomic_add(exec_cuda, &nEE[0], 1);
                        // EE[no] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                    }
                    break;
                }
                default: break;
                }
            };
            if (frontManageRequired)
                bvh.iter_neighbors(bv, i, front, f);
            else
                bvh.iter_neighbors(bv, front.node(i), f);
        });
        if (frontManageRequired)
            sefront.reorder(pol);
    }
    pol.profile(false);
}

void ClothSystem::computeCollisionGradientAndHessian(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    auto numPT = ncsPT.getVal();
    // group of size-4 tiles?
    pol(range(numPT), [vtemp = proxy<space>({}, vtemp), tempPT = proxy<space>({}, tempPT), csPT = proxy<space>(csPT),
                       gradOffset = vtemp.getPropertyOffset("grad"), thickness = dHat, maxMu = maxMu,
                       maxLam = maxLam] __device__(int i) mutable {
        auto pt = csPT[i];
        zs::vec<T, 3> vs[4] = {vtemp.pack(dim_c<3>, "xn", pt[0]), vtemp.pack(dim_c<3>, "xn", pt[1]),
                               vtemp.pack(dim_c<3>, "xn", pt[2]), vtemp.pack(dim_c<3>, "xn", pt[3])};
        auto grad = VERTEX_FACE_SQRT_COLLISION::gradient(vs, maxMu, maxLam, thickness);
        auto hess = VERTEX_FACE_SQRT_COLLISION::hessian(vs, maxMu, maxLam, thickness);
        // gradient
        for (int d = 0; d != 3; ++d) {
            atomic_add(exec_cuda, &vtemp(gradOffset + d, pt[0]), -grad(0 + d));
            atomic_add(exec_cuda, &vtemp(gradOffset + d, pt[1]), -grad(3 + d));
            atomic_add(exec_cuda, &vtemp(gradOffset + d, pt[2]), -grad(6 + d));
            atomic_add(exec_cuda, &vtemp(gradOffset + d, pt[3]), -grad(9 + d));
        }
        // hessian
        tempPT.tuple(dim_c<12, 12>, "H", i) = hess;
    });
    if (enableContactEE) {
        ;
    }
}

} // namespace zeno