#include "hip/hip_runtime.h"
#include "Cloth.cuh"
#include "Structures.hpp"
#include "TopoUtils.hpp"
#include "zensim/geometry/Distance.hpp"
#include <zeno/types/ListObject.h>

#define RETRIEVE_OBJECT_PTRS(T, STR)                                                  \
    ([this](const std::string_view str) {                                             \
        std::vector<T *> objPtrs{};                                                   \
        if (has_input<T>(str.data()))                                                 \
            objPtrs.push_back(get_input<T>(str.data()).get());                        \
        else if (has_input<zeno::ListObject>(str.data())) {                           \
            auto &objSharedPtrLists = *get_input<zeno::ListObject>(str.data());       \
            for (auto &&objSharedPtr : objSharedPtrLists.get())                       \
                if (auto ptr = dynamic_cast<T *>(objSharedPtr.get()); ptr != nullptr) \
                    objPtrs.push_back(ptr);                                           \
        }                                                                             \
        return objPtrs;                                                               \
    })(STR);

namespace zeno {

ClothSystem::PrimitiveHandle::PrimitiveHandle(std::shared_ptr<tiles_t> elesPtr_, ZenoParticles::category_e category)
    : zsprimPtr{}, modelsPtr{}, vertsPtr{}, elesPtr{elesPtr_},
      etemp{elesPtr_->get_allocator(), {{"He", 6 * 6}}, elesPtr_->size()}, surfTrisPtr{}, surfEdgesPtr{},
      surfVertsPtr{}, svtemp{}, vOffset{0}, sfOffset{0}, seOffset{0}, svOffset{0}, category{category} {
    ;
}
ClothSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, Ti &vOffset, Ti &sfOffset, Ti &seOffset,
                                              Ti &svOffset, zs::wrapv<2>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}}, vertsPtr{&zsprim.getParticles(),
                                                                                                [](void *) {}},
      elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, etemp{zsprim.getQuadraturePoints().get_allocator(),
                                                                   {{"He", 6 * 6}},
                                                                   zsprim.numElements()},
      surfTrisPtr{&zsprim.getQuadraturePoints(), [](void *) {}},  // this is fake!
      surfEdgesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, // all elements are surface edges
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::curve)
        throw std::runtime_error("dimension of 2 but is not curve");
    vOffset += getVerts().size();
    // sfOffset += 0; // no surface triangles
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
ClothSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, Ti &vOffset, Ti &sfOffset, Ti &seOffset,
                                              Ti &svOffset, zs::wrapv<3>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}}, vertsPtr{&zsprim.getParticles(),
                                                                                                [](void *) {}},
      elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, etemp{zsprim.getQuadraturePoints().get_allocator(),
                                                                   {{"He", 9 * 9}},
                                                                   zsprim.numElements()},
      surfTrisPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, surfEdgesPtr{&zsprim[ZenoParticles::s_surfEdgeTag],
                                                                              [](void *) {}},
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::surface)
        throw std::runtime_error("dimension of 3 but is not surface");
    vOffset += getVerts().size();
    sfOffset += getSurfTris().size();
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
ClothSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, Ti &vOffset, Ti &sfOffset, Ti &seOffset,
                                              Ti &svOffset, zs::wrapv<4>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}}, vertsPtr{&zsprim.getParticles(),
                                                                                                [](void *) {}},
      elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, etemp{zsprim.getQuadraturePoints().get_allocator(),
                                                                   {{"He", 12 * 12}},
                                                                   zsprim.numElements()},
      surfTrisPtr{&zsprim[ZenoParticles::s_surfTriTag], [](void *) {}},
      surfEdgesPtr{&zsprim[ZenoParticles::s_surfEdgeTag], [](void *) {}},
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::tet)
        throw std::runtime_error("dimension of 4 but is not tetrahedra");
    vOffset += getVerts().size();
    sfOffset += getSurfTris().size();
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
typename ClothSystem::T ClothSystem::PrimitiveHandle::averageNodalMass(zs::CudaExecutionPolicy &pol) const {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    if (zsprimPtr->hasMeta(s_meanMassTag))
        return zsprimPtr->readMeta(s_meanMassTag, zs::wrapt<T>{});
    auto &verts = getVerts();
    Vector<T> masses{verts.get_allocator(), verts.size()};
    pol(Collapse{verts.size()}, [verts = proxy<space>({}, verts), masses = proxy<space>(masses)] ZS_LAMBDA(
                                    int vi) mutable { masses[vi] = verts("m", vi); });
    auto tmp = reduce(pol, masses) / masses.size();
    zsprimPtr->setMeta(s_meanMassTag, tmp);
    return tmp;
}

/// ClothSystem
typename ClothSystem::T ClothSystem::averageNodalMass(zs::CudaExecutionPolicy &pol) {
    using T = typename ClothSystem::T;
    T sumNodalMass = 0;
    int sumNodes = 0;
    for (auto &&primHandle : prims) {
        if (primHandle.isBoundary())
            continue;
        auto numNodes = primHandle.getVerts().size();
        sumNodes += numNodes;
        sumNodalMass += primHandle.averageNodalMass(pol) * numNodes;
    }
    if (sumNodes)
        return sumNodalMass / sumNodes;
    else
        return 0;
}
void ClothSystem::updateWholeBoundingBoxSize(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    bv_t bv = seBvh.getTotalBox(pol);
    if (coVerts)
        if (coVerts->size()) {
            auto bouBv = bouSeBvh.getTotalBox(pol);
            merge(bv, bouBv._min);
            merge(bv, bouBv._max);
        }
    boxDiagSize2 = (bv._max - bv._min).l2NormSqr();
}

void ClothSystem::markSelfIntersectionPrimitives(zs::CudaExecutionPolicy &pol) {
    //exclSes, exclSts, stInds, seInds, seBvh
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    exclSes.reset(0);
    exclSts.reset(0);
    exclBouSes.reset(0);
    exclBouSts.reset(0);

    Vector<int> cnt{vtemp.get_allocator(), 1};
    cnt.setVal(0);

    auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", seInds, wrapv<2>{}, 0);
    seBvh.refit(pol, edgeBvs);
    pol(range(stInds.size()), [vtemp = proxy<space>({}, vtemp), stInds = proxy<space>({}, stInds),
                               seInds = proxy<space>({}, seInds), exclSes = proxy<space>(exclSes),
                               exclSts = proxy<space>(exclSts), bvh = proxy<space>(seBvh), cnt = proxy<space>(cnt),
                               dHat = dHat] __device__(int sti) mutable {
        auto tri = stInds.pack(dim_c<3>, "inds", sti).reinterpret_bits(int_c);
        auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
        auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
        auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);
        auto bv = bv_t{get_bounding_box(t0, t1)};
        merge(bv, t2);
        bool triIntersected = false;
        bvh.iter_neighbors(bv, [&](int sei) {
            auto line = seInds.pack(dim_c<2>, "inds", sei).reinterpret_bits(int_c);
            if (tri[0] == line[0] || tri[0] == line[1] || tri[1] == line[0] || tri[1] == line[1] || tri[2] == line[0] ||
                tri[2] == line[1])
                return;
            if (et_intersected(vtemp.pack(dim_c<3>, "xn", line[0]), vtemp.pack(dim_c<3>, "xn", line[1]), t0, t1, t2)) {
                triIntersected = true;
                exclSes[sei] = 1;

                atomic_add(exec_cuda, &cnt[0], 1);
            }
        });
        if (triIntersected)
            exclSts[sti] = 1;
    });
    zeno::log_info("{} self et intersections\n", cnt.getVal());

    if (coEdges) {
        cnt.setVal(0);
        edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, coOffset);
        bouSeBvh.refit(pol, edgeBvs);
        pol(range(stInds.size()),
            [vtemp = proxy<space>({}, vtemp), stInds = proxy<space>({}, stInds), seInds = proxy<space>({}, *coEdges),
             exclBouSes = proxy<space>(exclBouSes), exclSts = proxy<space>(exclSts), bvh = proxy<space>(bouSeBvh),
             cnt = proxy<space>(cnt), dHat = dHat, voffset = coOffset] __device__(int sti) mutable {
                auto tri = stInds.pack(dim_c<3>, "inds", sti).reinterpret_bits(int_c);
                auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
                auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
                auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);
                auto bv = bv_t{get_bounding_box(t0, t1)};
                merge(bv, t2);
                bool triIntersected = false;
                bvh.iter_neighbors(bv, [&](int sei) {
                    auto line = seInds.pack(dim_c<2>, "inds", sei).reinterpret_bits(int_c) + voffset;
                    // no need to check common vertices here
                    if (et_intersected(vtemp.pack(dim_c<3>, "xn", line[0]), vtemp.pack(dim_c<3>, "xn", line[1]), t0, t1,
                                       t2)) {
                        triIntersected = true;
                        exclBouSes[sei] = 1;

                        atomic_add(exec_cuda, &cnt[0], 1);
                    }
                });
                if (triIntersected)
                    exclSts[sti] = 1;
            });

        auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, coOffset);
        bouStBvh.refit(pol, triBvs);
        pol(range(seInds.size()),
            [vtemp = proxy<space>({}, vtemp), seInds = proxy<space>({}, seInds), coTris = proxy<space>({}, *coEles),
             exclBouSts = proxy<space>(exclBouSts), exclSes = proxy<space>(exclSes), bvh = proxy<space>(bouStBvh),
             cnt = proxy<space>(cnt), dHat = dHat, voffset = coOffset] __device__(int sei) mutable {
                auto line = seInds.pack(dim_c<2>, "inds", sei).reinterpret_bits(int_c);
                auto e0 = vtemp.pack(dim_c<3>, "xn", line[0]);
                auto e1 = vtemp.pack(dim_c<3>, "xn", line[1]);
                auto bv = bv_t{get_bounding_box(e0, e1)};
                bool edgeIntersected = false;
                bvh.iter_neighbors(bv, [&](int sti) {
                    auto tri = coTris.pack(dim_c<3>, "inds", sti).reinterpret_bits(int_c) + voffset;
                    // no need to check common vertices here
                    if (et_intersected(e0, e1, vtemp.pack(dim_c<3>, "xn", tri[0]), vtemp.pack(dim_c<3>, "xn", tri[1]),
                                       vtemp.pack(dim_c<3>, "xn", tri[2]))) {
                        edgeIntersected = true;
                        exclBouSts[sti] = 1;

                        atomic_add(exec_cuda, &cnt[0], 1);
                    }
                });
                if (edgeIntersected)
                    exclSes[sei] = 1;
            });
        zeno::log_info("{} boundary et intersections\n", cnt.getVal());
    }
    return;
}

void ClothSystem::initialize(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    stInds = tiles_t{vtemp.get_allocator(), {{"inds", 3}}, (std::size_t)sfOffset};
    seInds = tiles_t{vtemp.get_allocator(), {{"inds", 2}}, (std::size_t)seOffset};
    svInds = tiles_t{vtemp.get_allocator(), {{"inds", 1}}, (std::size_t)svOffset};
    exclSes = Vector<u8>{vtemp.get_allocator(), (std::size_t)seOffset};
    exclSts = Vector<u8>{vtemp.get_allocator(), (std::size_t)sfOffset};
    std::size_t nBouSes = 0, nBouSts = 0;
    if (coEdges) {
        nBouSes = coEdges->size();
        nBouSts = coEles->size();
    }
    exclBouSes = Vector<u8>{vtemp.get_allocator(), nBouSes};
    exclBouSts = Vector<u8>{vtemp.get_allocator(), nBouSts};

    auto deduce_node_cnt = [](std::size_t numLeaves) {
        if (numLeaves <= 2)
            return numLeaves;
        return numLeaves * 2 - 1;
    };
    selfStFront = bvfront_t{(int)deduce_node_cnt(stInds.size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
    selfSeFront = bvfront_t{(int)deduce_node_cnt(seInds.size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
    if (coVerts) {
        boundaryStFront =
            bvfront_t{(int)deduce_node_cnt(coEles->size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
        boundarySeFront =
            bvfront_t{(int)deduce_node_cnt(coEdges->size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
    }

    avgNodeMass = averageNodalMass(pol);

    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        const auto &verts = primHandle.getVerts();
        // record surface (tri) indices
        if (primHandle.category != ZenoParticles::category_e::curve) {
            auto &tris = primHandle.getSurfTris();
            pol(Collapse(tris.size()), [stInds = proxy<space>({}, stInds), tris = proxy<space>({}, tris),
                                        voffset = primHandle.vOffset,
                                        sfoffset = primHandle.sfOffset] __device__(int i) mutable {
                stInds.tuple(dim_c<3>, "inds", sfoffset + i) =
                    (tris.pack(dim_c<3>, "inds", i).reinterpret_bits(int_c) + (int)voffset).reinterpret_bits(float_c);
            });
        }
        const auto &edges = primHandle.getSurfEdges();
        pol(Collapse(edges.size()),
            [seInds = proxy<space>({}, seInds), edges = proxy<space>({}, edges), voffset = primHandle.vOffset,
             seoffset = primHandle.seOffset] __device__(int i) mutable {
                seInds.tuple(dim_c<2>, "inds", seoffset + i) =
                    (edges.pack(dim_c<2>, "inds", i).reinterpret_bits(int_c) + (int)voffset).reinterpret_bits(float_c);
            });
        const auto &points = primHandle.getSurfVerts();
        pol(Collapse(points.size()),
            [svInds = proxy<space>({}, svInds), points = proxy<space>({}, points), voffset = primHandle.vOffset,
             svoffset = primHandle.svOffset] __device__(int i) mutable {
                svInds("inds", svoffset + i) =
                    reinterpret_bits<float>(reinterpret_bits<int>(points("inds", i)) + (int)voffset);
            });
    }
    // initialize vtemp & spatial accel
    reinitialize(pol, dt);
}

void ClothSystem::reinitialize(zs::CudaExecutionPolicy &pol, T framedt) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    dt = framedt;
    this->framedt = framedt;
    curRatio = 0;

    substep = -1;
    projectDBC = false;

    if (enableContact) {
        nPP.setVal(0);
        nPE.setVal(0);
        nPT.setVal(0);
        nEE.setVal(0);

        ncsPT.setVal(0);
        ncsEE.setVal(0);
    }

    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        auto &verts = primHandle.getVerts();
        // initialize BC info
        // predict pos, initialize augmented lagrangian, constrain weights
        pol(Collapse(verts.size()),
            [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts), voffset = primHandle.vOffset, dt = dt,
             avgNodeMass = avgNodeMass] __device__(int i) mutable {
                auto x = verts.pack<3>("x", i);
                auto v = verts.pack<3>("v", i);

                vtemp("ws", voffset + i) = verts("m", i);
                vtemp.tuple<3>("xtilde", voffset + i) = x + v * dt;
                vtemp.tuple<3>("xn", voffset + i) = x;
                vtemp.tuple<3>("vn", voffset + i) = v;
                vtemp.tuple<3>("xhat", voffset + i) = x;
            });
    }
    if (coVerts)
        if (auto coSize = coVerts->size(); coSize) {
            pol(Collapse(coSize),
                [vtemp = proxy<space>({}, vtemp), coverts = proxy<space>({}, *coVerts), coOffset = coOffset, dt = dt,
                 augLagCoeff = augLagCoeff, avgNodeMass = avgNodeMass] __device__(int i) mutable {
                    auto x = coverts.pack<3>("x", i);
                    auto v = coverts.pack<3>("v", i);
                    auto newX = x + v * dt;

                    vtemp("ws", coOffset + i) = avgNodeMass * augLagCoeff;
                    vtemp.tuple<3>("xtilde", coOffset + i) = newX;
                    vtemp.tuple<3>("xn", coOffset + i) = x;
                    // vtemp.tuple<3>("vn", coOffset + i) = v;
                    // vtemp.tuple<3>("xhat", coOffset + i) = x;
                });
        }

    // spatial accel structs
    frontManageRequired = true;
#define init_front(sInds, front)                                                                                 \
    {                                                                                                            \
        auto numNodes = front.numNodes();                                                                        \
        if (numNodes <= 2) {                                                                                     \
            front.reserve(sInds.size() * numNodes);                                                              \
            front.setCounter(sInds.size() * numNodes);                                                           \
            pol(Collapse{sInds.size()}, [front = proxy<space>(selfStFront), numNodes] ZS_LAMBDA(int i) mutable { \
                for (int j = 0; j != numNodes; ++j)                                                              \
                    front.assign(i *numNodes + j, i, j);                                                         \
            });                                                                                                  \
        } else {                                                                                                 \
            front.reserve(sInds.size());                                                                         \
            front.setCounter(sInds.size());                                                                      \
            pol(Collapse{sInds.size()},                                                                          \
                [front = proxy<space>(front)] ZS_LAMBDA(int i) mutable { front.assign(i, i, 0); });              \
        }                                                                                                        \
    }
    {
        auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, 0);
        stBvh.build(pol, triBvs);
        init_front(svInds, selfStFront);

        auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", seInds, zs::wrapv<2>{}, 0);
        seBvh.build(pol, edgeBvs);
        init_front(seInds, selfSeFront);
    }
    if (coVerts)
        if (coVerts->size()) {
            auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, coOffset);
            bouStBvh.build(pol, triBvs);
            init_front(svInds, boundaryStFront);

            auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, coOffset);
            bouSeBvh.build(pol, edgeBvs);
            init_front(seInds, boundarySeFront);
        }

    updateWholeBoundingBoxSize(pol);
    /// update grad pn residual tolerance
    targetGRes = pnRel * std::sqrt(boxDiagSize2);
}

ClothSystem::ClothSystem(std::vector<ZenoParticles *> zsprims, tiles_t *coVerts, tiles_t *coEdges, tiles_t *coEles,
                         T dt, std::size_t estNumCps, bool withContact, T augLagCoeff, T pnRel, T cgRel, int PNCap,
                         int CGCap, T dHat_, T gravity)
    : coVerts{coVerts}, coEdges{coEdges}, coEles{coEles}, PP{estNumCps, zs::memsrc_e::um, 0},
      nPP{zsprims[0]->getParticles().get_allocator(), 1}, tempPP{{{"H", 36}}, estNumCps, zs::memsrc_e::um, 0},
      PE{estNumCps, zs::memsrc_e::um, 0}, nPE{zsprims[0]->getParticles().get_allocator(), 1},
      tempPE{{{"H", 81}}, estNumCps, zs::memsrc_e::um, 0}, PT{estNumCps, zs::memsrc_e::um, 0},
      nPT{zsprims[0]->getParticles().get_allocator(), 1}, tempPT{{{"H", 144}}, estNumCps, zs::memsrc_e::um, 0},
      EE{estNumCps, zs::memsrc_e::um, 0}, nEE{zsprims[0]->getParticles().get_allocator(), 1}, tempEE{{{"H", 144}},
                                                                                                     estNumCps,
                                                                                                     zs::memsrc_e::um,
                                                                                                     0},
      //
      temp{estNumCps, zs::memsrc_e::um, 0}, csPT{estNumCps, zs::memsrc_e::um, 0}, csEE{estNumCps, zs::memsrc_e::um, 0},
      ncsPT{zsprims[0]->getParticles().get_allocator(), 1}, ncsEE{zsprims[0]->getParticles().get_allocator(), 1},
      //
      dt{dt}, framedt{dt}, curRatio{0}, estNumCps{estNumCps}, enableContact{withContact}, augLagCoeff{augLagCoeff},
      pnRel{pnRel}, cgRel{cgRel}, PNCap{PNCap}, CGCap{CGCap}, dHat{dHat_}, extAccel{0, gravity, 0} {
    coOffset = sfOffset = seOffset = svOffset = 0;
    for (auto primPtr : zsprims) {
        if (primPtr->category == ZenoParticles::category_e::curve) {
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<2>{});
        } else if (primPtr->category == ZenoParticles::category_e::surface)
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<3>{});
        else if (primPtr->category == ZenoParticles::category_e::tet)
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<4>{});
    }
    numDofs = coOffset;
    if (coVerts)
        numDofs += coVerts->size();
    numBouDofs = numDofs - coOffset;

    fmt::print("num total obj <verts, bouVerts, surfV, surfE, surfT>: {}, {}, {}, {}, {}\n", coOffset, numBouDofs,
               svOffset, seOffset, sfOffset);

    vtemp = tiles_t{zsprims[0]->getParticles().get_allocator(),
                    {{"grad", 3},
                     {"P", 9},
                     {"ws", 1}, // also as constraint jacobian
                     {"cons", 3},

                     {"dir", 3},
                     {"xn", 3},
                     {"vn", 3},
                     {"xtilde", 3},
                     {"xhat", 3}, // initial positions at the current substep (constraint,
                                  // extAccel)
                     {"temp", 3},
                     {"r", 3},
                     {"p", 3},
                     {"q", 3}},
                    (std::size_t)numDofs};

    auto cudaPol = zs::cuda_exec();
    // average edge length (for CCD filtering)
    initialize(cudaPol); // update vtemp, bvh, boxsize, targetGRes
                         // adaptive dhat, targetGRes, kappa
    // dHat (static)
    this->dHat = dHat_ * std::sqrt(boxDiagSize2);

    auto [mu_, lam_] = largestLameParams();
    maxMu = mu_;
    maxLam = lam_;

    // check initial self intersections including proximity pairs, do once
    markSelfIntersectionPrimitives(cudaPol);
}

void ClothSystem::advanceSubstep(zs::CudaExecutionPolicy &pol, T ratio) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    // setup substep dt
    ++substep;
    dt = framedt * ratio;
    curRatio += ratio;

    projectDBC = false;
    pol(Collapse(coOffset), [vtemp = proxy<space>({}, vtemp), coOffset = coOffset, dt = dt] __device__(int vi) mutable {
        auto xn = vtemp.pack(dim_c<3>, "xn", vi);
        vtemp.tuple(dim_c<3>, "xhat", vi) = xn;
        auto newX = xn + vtemp.pack(dim_c<3>, "vn", vi) * dt;
        vtemp.tuple(dim_c<3>, "xtilde", vi) = newX;
    });
    if (coVerts)
        if (auto coSize = coVerts->size(); coSize)
            pol(Collapse(coSize), [vtemp = proxy<space>({}, vtemp), coverts = proxy<space>({}, *coVerts),
                                   coOffset = coOffset, dt = dt] __device__(int i) mutable {
                auto xn = vtemp.pack(dim_c<3>, "xn", coOffset + i);
                vtemp.tuple(dim_c<3>, "xhat", coOffset + i) = xn;
                auto newX = xn + coverts.pack(dim_c<3>, "v", i) * dt;
                vtemp.tuple(dim_c<3>, "xtilde", coOffset + i) = newX;
            });
    for (auto &primHandle : auxPrims) {
        /// @note hard constraint
        if (primHandle.category == ZenoParticles::category_e::tracker) {
            const auto &eles = primHandle.getEles();
            pol(Collapse(eles.size()), [vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
                                        framedt = framedt, curRatio = curRatio] __device__(int ei) mutable {
                auto inds = eles.pack(dim_c<2>, "inds", ei).reinterpret_bits(int_c);
                // retrieve motion from the associated boundary vert
                auto deltaX = vtemp.pack(dim_c<3>, "xtilde", inds[1]) - vtemp.pack(dim_c<3>, "xhat", inds[1]);
                auto xn = vtemp.pack(dim_c<3>, "xn", inds[0]);
                vtemp.tuple(dim_c<3>, "xtilde", inds[0]) = xn + deltaX;
            });
        }
    }
}

void ClothSystem::updateVelocities(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    pol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp), dt = dt] __device__(int vi) mutable {
        auto newX = vtemp.pack<3>("xn", vi);
        auto dv = (newX - vtemp.pack<3>("xtilde", vi)) / dt;
        auto vn = vtemp.pack<3>("vn", vi);
        vn += dv;
        vtemp.tuple<3>("vn", vi) = vn;
    });
}

void ClothSystem::writebackPositionsAndVelocities(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        auto &verts = primHandle.getVerts();
        // update velocity and positions
        pol(zs::range(verts.size()),
            [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts), dt = dt, vOffset = primHandle.vOffset,
             asBoundary = primHandle.isBoundary()] __device__(int vi) mutable {
                verts.tuple<3>("x", vi) = vtemp.pack<3>("xn", vOffset + vi);
                if (!asBoundary)
                    verts.tuple<3>("v", vi) = vtemp.pack<3>("vn", vOffset + vi);
            });
    }
    if (coVerts)
        if (auto coSize = coVerts->size(); coSize)
            pol(Collapse(coSize),
                [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, *const_cast<tiles_t *>(coVerts)),
                 coOffset = coOffset] ZS_LAMBDA(int vi) mutable {
                    verts.tuple(dim_c<3>, "x", vi) = vtemp.pack(dim_c<3>, "xn", coOffset + vi);
                    // no need to update v here. positions are moved accordingly
                    // also, boundary velocies are set elsewhere
                });
}

struct MakeClothSystem : INode {
    void apply() override {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        auto zsprims = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");
        std::shared_ptr<ZenoParticles> zsboundary;
        if (has_input<ZenoParticles>("ZSBoundaryPrimitives"))
            zsboundary = get_input<ZenoParticles>("ZSBoundaryPrimitives");

        auto cudaPol = zs::cuda_exec();

        typename ClothSystem::tiles_t *coVerts = zsboundary ? &zsboundary->getParticles() : nullptr;
        typename ClothSystem::tiles_t *coEdges = zsboundary ? &(*zsboundary)[ZenoParticles::s_surfEdgeTag] : nullptr;
        typename ClothSystem::tiles_t *coEles = zsboundary ? &zsboundary->getQuadraturePoints() : nullptr;
#if 0
        const typename ClothSystem::tiles_t *coSvs =
            zsboundary ? &(*zsboundary)[ZenoParticles::s_surfVertTag] : nullptr;
#endif

        if (zsboundary) {
            auto pol = cuda_exec();
            compute_surface_neighbors(pol, *coEles, *coEdges, (*zsboundary)[ZenoParticles::s_surfVertTag]);
            coEles->append_channels(pol, {{"nrm", 3}});
            coEdges->append_channels(pol, {{"nrm", 3}});
        }

        /// solver parameters
        auto input_est_num_cps = get_input2<int>("est_num_cps");
        auto input_withContact = get_input2<bool>("with_contact");
        auto input_contactEE = get_input2<bool>("contact_with_ee");
        auto input_contactSelf = get_input2<bool>("contact_with_self");
        auto input_dHat = get_input2<float>("dHat");
        auto input_aug_coeff = get_input2<float>("aug_coeff");
        auto input_pn_rel = get_input2<float>("pn_rel");
        auto input_cg_rel = get_input2<float>("cg_rel");
        auto input_pn_cap = get_input2<int>("pn_iter_cap");
        auto input_cg_cap = get_input2<int>("cg_iter_cap");
        auto input_gravity = get_input2<float>("gravity");
        auto dt = get_input2<float>("dt");

        auto A = std::make_shared<ClothSystem>(zsprims, coVerts, coEdges, coEles, dt,
                                               (std::size_t)(input_est_num_cps ? input_est_num_cps : 1000000),
                                               input_withContact, input_aug_coeff, input_pn_rel, input_cg_rel,
                                               input_pn_cap, input_cg_cap, input_dHat, input_gravity);
        A->enableContactEE = input_contactEE;
        A->enableContactSelf = input_contactSelf;

        set_output("ZSClothSystem", A);
    }
};

ZENDEFNODE(MakeClothSystem, {{
                                 "ZSParticles",
                                 "ZSBoundaryPrimitives",
                                 {"int", "est_num_cps", "1000000"},
                                 {"bool", "with_contact", "1"},
                                 {"bool", "contact_with_ee", "1"},
                                 {"bool", "contact_with_self", "1"},
                                 {"float", "dt", "0.01"},
                                 {"float", "dHat", "0.001"},
                                 {"float", "aug_coeff", "1e2"},
                                 {"float", "pn_rel", "0.01"},
                                 {"float", "cg_rel", "0.001"},
                                 {"int", "pn_iter_cap", "1000"},
                                 {"int", "cg_iter_cap", "1000"},
                                 {"float", "gravity", "-9.8"},
                             },
                             {"ZSClothSystem"},
                             {},
                             {"FEM"}});

} // namespace zeno