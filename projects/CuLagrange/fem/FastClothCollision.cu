#include "hip/hip_runtime.h"
#include "FastCloth.cuh"
#include "TopoUtils.hpp"
#include "collision_energy/vertex_face_sqrt_collision.hpp"
#include "zensim/Logger.hpp"
#include "zensim/geometry/Distance.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include <zeno/core/INode.h>
#include <zeno/types/ListObject.h>
#include <zeno/utils/log.h>
#include <zeno/zeno.h>

namespace zeno {

void FastClothSystem::initialStepping(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    /// @brief Xinit
    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), D = D] ZS_LAMBDA(int i) mutable {
        auto xk = vtemp.pack(dim_c<3>, "xn", i);
        auto ykp1 = vtemp.pack(dim_c<3>, "yn", i);
        auto diff = ykp1 - xk;
        T coeff = 1;
        if (auto len2 = diff.l2NormSqr(); len2 > limits<T>::epsilon() * 10)
            coeff = zs::min(D / zs::sqrt(len2), (T)1);
        vtemp.tuple(dim_c<3>, "xinit", i) = xk + coeff * diff;
    });
}

void FastClothSystem::findConstraints(zs::CudaExecutionPolicy &pol, T dHat, const zs::SmallString &tag) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    // zs::CppTimer timer;
    if (enableContact) {
        nPP.setVal(0);
        if (enableContactSelf) {
            auto pBvs = retrieve_bounding_volumes(pol, vtemp, tag, svInds, zs::wrapv<1>{}, 0);
            svBvh.refit(pol, pBvs);
            /// @note all cloth edge lower-bound constraints inheritly included
            findCollisionConstraints(pol, dHat, false);
        }
        if (hasBoundary()) {
            auto pBvs = retrieve_bounding_volumes(pol, vtemp, tag, *coPoints, zs::wrapv<1>{}, coOffset);
            bouSvBvh.refit(pol, pBvs);
            findCollisionConstraints(pol, dHat, true);
            // for repulsion
            // findBoundaryCellCollisionConstraints(pol, dHat);
        }
        frontManageRequired = false;
    }
    /// @note check upper-bound constraints for cloth edges
    nE.setVal(0);
    for (auto &primHandle : prims) {
        auto &ses = primHandle.getSurfEdges();
        pol(Collapse{ses.size()},
            [ses = proxy<space>({}, ses), vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), nE = proxy<space>(nE),
             threshold = L * L - epsSlack, vOffset = primHandle.vOffset] __device__(int sei) mutable {
                const auto vij = ses.pack(dim_c<2>, "inds", sei).reinterpret_bits(int_c) + vOffset;
                const auto &vi = vij[0];
                const auto &vj = vij[1];
                auto pi = vtemp.pack(dim_c<3>, "xn", vi);
                auto pj = vtemp.pack(dim_c<3>, "xn", vj);
                if (auto d2 = dist2_pp(pi, pj); d2 > threshold) {
                    auto no = atomic_add(exec_cuda, &nE[0], 1);
                    E[no] = vij;
                }
            });
    }
}

#define PROFILE_CD 0

void FastClothSystem::findCollisionConstraints(zs::CudaExecutionPolicy &pol, T dHat, bool withBoundary) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    pol.profile(PROFILE_CD);
    /// pt
    const auto &svbvh = withBoundary ? bouSvBvh : svBvh;
    auto &svfront = withBoundary ? boundarySvFront : selfSvFront;
    pol(Collapse{svfront.size()},
        [svInds = proxy<space>({}, svInds), eles = proxy<space>({}, withBoundary ? *coPoints : svInds),
         eTab = proxy<space>(eTab), 
         // exclTris = withBoundary ? proxy<space>(exclBouSts) : proxy<space>(exclSts),
         vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(svbvh), front = proxy<space>(svfront),
         PP = proxy<space>(PP), nPP = proxy<space>(nPP), dHat2 = dHat * dHat, thickness = dHat,
         voffset = withBoundary ? coOffset : 0, frontManageRequired = frontManageRequired] __device__(int i) mutable {
            auto vi = front.prim(i);
            vi = reinterpret_bits<int>(svInds("inds", vi));
            auto pi = vtemp.pack(dim_c<3>, "xn", vi);
            auto bv = bv_t{get_bounding_box(pi - thickness, pi + thickness)};
            auto f = [&](int svI) {
                // if (exclTris[stI]) return;
                auto vj = reinterpret_bits<int>(eles("inds", svI)) + voffset;
                if (vi > vj)
                    return;
                auto pj = vtemp.pack(dim_c<3>, "xn", vj);
                // edge or not
                if (eTab.single_query(ivec2 {vi, vj}) >= 0 || eTab.single_query(ivec2 {vj, vi}) >= 0)
                    return; 
                if (auto d2 = dist2_pp(pi, pj); d2 < dHat2) {
                    auto no = atomic_add(exec_cuda, &nPP[0], 1);
                    PP[no] = pair_t{vi, vj};
                }
            };
            if (frontManageRequired)
                bvh.iter_neighbors(bv, i, front, f);
            else
                bvh.iter_neighbors(bv, front.node(i), f);
        });
    if (frontManageRequired)
        svfront.reorder(pol);
    pol.profile(false);
}

bool FastClothSystem::collisionStep(zs::CudaExecutionPolicy &pol, bool enableHardPhase) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    auto [npp_, ne_] = getConstraintCnt();
    npp = npp_;
    ne = ne_;
    fmt::print("collision stepping [pp, edge constraints]: {}, {}\n", npp, ne);

    ///
    /// @brief soft phase for constraints
    ///
    pol(range(numDofs), [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
        auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            vtemp("xn", d, i) = xinit(d); // soft phase optimization starts from xinit
        }
    });
    for (int l = 0; l != ISoft; ++l) {
        softPhase(pol);
    }

    ///
    /// @brief check whether constraints satisfied
    ///
    findConstraints(pol, dHat); 
    if (constraintSatisfied(pol))
    {
        fmt::print(fg(fmt::color::yellow),"\tsoft phase finished successfully!\n"); 
        return true;
    }
    fmt::print(fg(fmt::color::red),"\tsoft phase failed!\n"); 
    if (!enableHardPhase)
        return false;

    ///
    /// @brief hard phase for constraints
    ///
    fmt::print(fg(fmt::color::light_golden_rod_yellow), "entering hard phase.\n");
    /// @note start from collision-free state x^k
    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
        vtemp.tuple(dim_c<3>, "xn", i) = vtemp.pack(dim_c<3>, "xk", i);
    });
    for (int l = 0; l != IHard; ++l) {
        /// @note "xk" will be used for backtracking in hardphase
        hardPhase(pol);
    }

    return constraintSatisfied(pol);
}
void FastClothSystem::softPhase(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    T descentStepsize = 0.1f; 
    /// @note shape matching
    pol(range(numDofs), [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
        auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
        auto xn = vtemp.pack(dim_c<3>, "xn", i);
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            vtemp("dir", d, i) = 2.0f * (xinit(d) - xn(d)); // minus grad of ||x-xinit||^2
        }
    });
    /// @note constraints
    pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), rho = rho, dHat2 = dHat * dHat] __device__(int i) mutable {
        auto pp = PP[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]); 
        // ||v0 - v1||^2 >= (B + Bt)^2 + epsSlack 
        // c(x) = ||v0 - v1||^2 - (B + Bt)^2
        if ((x0 - x1).l2NormSqr() >= dHat2)
            return; 
        auto grad0 = - rho * (T)2.0 * (x0 - x1);
#pragma unroll 3
        for (int d = 0; d < 3; d++) {
            atomic_add(exec_cuda, &vtemp("dir", d, pp[0]), -grad0(d)); 
            atomic_add(exec_cuda, &vtemp("dir", d, pp[1]), grad0(d)); 
        } 
    }); 

    pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), rho = rho, 
        maxLen2 = L * L - epsSlack] __device__(int i) mutable {
        auto e = E[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]); 
        // ||v0 - v1||^2 <= L^2 - epsSlack 
        // i.e. L^2 - ||v0 - v1||^2 >= epsSlack
        // c(x) = L^2 - ||v0 - v1||^2
        if ((x0 - x1).l2NormSqr() <= maxLen2)
            return; 
        auto grad0 = rho * (T)2.0 * (x0 - x1);
#pragma unroll 3
        for (int d = 0; d < 3; d++) {
            atomic_add(exec_cuda, &vtemp("dir", d, e[0]), -grad0(d)); 
            atomic_add(exec_cuda, &vtemp("dir", d, e[1]), grad0(d)); 
        }
    });
    pol(range(numDofs), [vtemp = proxy<space>({}, vtemp), 
            descentStepsize] __device__(int i) mutable {
        auto dir = vtemp.pack(dim_c<3>, "dir", i);
        auto xn = vtemp.pack(dim_c<3>, "xn", i); 
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            atomic_add(exec_cuda, &vtemp("xn", d, i), descentStepsize * dir(d));
        }
    });
}
void FastClothSystem::hardPhase(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    /// @note shape matching (reset included)
    pol(range(numDofs), [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
        auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
        auto xn = vtemp.pack(dim_c<3>, "xn", i);
#pragma unroll 3
        for (int d = 0; d < 3; ++d)
            vtemp("dir", d, i) = 2 * (xinit(d) - xn(d));
    });
    /// @note constraints
    pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), mu = mu,
                     Btot2 = (B + Btight) * (B + Btight), eps = epsSlack, dHat2 = dHat * dHat] __device__(int i) mutable {
        auto pp = PP[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]); 
        if ((x0 - x1).l2NormSqr() >= dHat2)
            return; 
        zs::vec<T, 3> vs[2] = {x0, x1};
        const auto &a = vs[0];
        const auto &b = vs[1];
        const auto t2 = a[0] * 2;
        const auto t3 = a[1] * 2;
        const auto t4 = a[2] * 2;
        const auto t5 = b[0] * 2;
        const auto t6 = b[1] * 2;
        const auto t7 = b[2] * 2;

        auto t8 = -Btot2;
        auto t9 = -b[0];
        auto t11 = -b[1];
        auto t13 = -b[2];
        auto t15 = 1 / eps;
        auto t10 = -t5;
        auto t12 = -t6;
        auto t14 = -t7;
        auto t16 = t15 * t15;
        auto t17 = a[0] + t9;
        auto t18 = a[1] + t11;
        auto t19 = a[2] + t13;
        auto t20 = t2 + t10;
        auto t21 = t3 + t12;
        auto t22 = t4 + t14;
        auto t23 = t17 * t17;
        auto t24 = t18 * t18;
        auto t25 = t19 * t10;
        auto t26 = t8 + t23 + t24 + t25;
        auto t27 = t26 * t26;
        auto t28 = t26 * t26 * t26;
        auto t32 = t15 * t20 * t26 * 2;
        auto t33 = t15 * t21 * t26 * 2;
        auto t34 = t15 * t22 * t26 * 2;
        auto t29 = t15 * t27;
        auto t30 = t16 * t28;
        auto t35 = t16 * t20 * t27 * 3;
        auto t36 = t16 * t21 * t27 * 3;
        auto t37 = t16 * t22 * t27 * 3;
        auto t31 = -t30;
        auto t38 = -t35;
        auto t39 = -t36;
        auto t40 = -t37;
        auto t41 = t20 + t32 + t38;
        auto t42 = t21 + t33 + t39;
        auto t43 = t22 + t34 + t40;
        auto t44 = t26 + t29 + t31;
        auto t45 = 1 / t44;
        auto t46 = mu * t41 * t45;
        auto t47 = mu * t42 * t45;
        auto t48 = mu * t43 * t45;
        auto grad = zs::vec<T, 6>{-t46, -t47, -t48, t46, t47, t48};
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            atomic_add(exec_cuda, &vtemp("dir", d, pp[0]), -grad(d));
            atomic_add(exec_cuda, &vtemp("dir", d, pp[1]), -grad(3 + d));
        }
    });
    pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), mu = mu, L2 = L * L,
                    eps = epsSlack, maxLen2 = L * L - epsSlack] __device__(int i) mutable {
        auto e = E[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]); 
        if ((x0 - x1).l2NormSqr() <= maxLen2)
            return; 
        zs::vec<T, 3> vs[2] = {x0, x1};
        const auto &a = vs[0];
        const auto &b = vs[1];
        const auto t2 = a[0] * 2;
        const auto t3 = a[1] * 2;
        const auto t4 = a[2] * 2;
        const auto t5 = b[0] * 2;
        const auto t6 = b[1] * 2;
        const auto t7 = b[2] * 2;
        auto t8 = -L2;
        auto t12 = -b[0];
        auto t14 = -b[1];
        auto t16 = -b[2];
        auto t18 = 1 / eps;
        auto t9 = -t2;
        auto t10 = -t3;
        auto t11 = -t4;
        auto t13 = -t5;
        auto t15 = -t6;
        auto t17 = -t7;
        auto t19 = t18 * t18;
        auto t20 = a[0] + t12;
        auto t21 = a[1] + t14;
        auto t22 = a[2] + t16;
        auto t23 = t2 + t13;
        auto t24 = t3 + t15;
        auto t25 = t4 + t17;
        auto t26 = t20 * t20;
        auto t27 = t21 * t21;
        auto t28 = t22 * t22;
        auto t29 = -t26;
        auto t30 = -t27;
        auto t31 = -t28;
        auto t32 = t8 + t26 + t27 + t28;
        auto t33 = t32 * t32;
        auto t34 = t32 * t32 * t32;
        auto t37 = t18 * t23 * t32 * 2;
        auto t38 = t18 * t24 * t32 * 2;
        auto t39 = t18 * t25 * t32 * 2;
        auto t35 = t18 * t33;
        auto t36 = t19 * t34;
        auto t40 = t19 * t23 * t33 * 3;
        auto t41 = t19 * t24 * t33 * 3;
        auto t42 = t19 * t25 * t33 * 3;
        auto t43 = t5 + t9 + t37 + t40;
        auto t44 = t6 + t10 + t38 + t41;
        auto t45 = t7 + t11 + t39 + t42;
        auto t46 = L2 + t29 + t30 + t31 + t35 + t36;
        auto t47 = 1 / t46;
        auto t48 = mu * t43 * t47;
        auto t49 = mu * t44 * t47;
        auto t50 = mu * t45 * t47;
        auto grad = zs::vec<T, 6>{-t48, -t49, -t50, t48, t49, t50};
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            atomic_add(exec_cuda, &vtemp("dir", d, e[0]), -grad(d));
            atomic_add(exec_cuda, &vtemp("dir", d, e[1]), -grad(3 + d));
        }
    });
    /// @brief compute appropriate step size that does not violates constraints
    auto alpha = (T)0.1;
    /// @note vertex displacement constraint. ref 4.2.2, item 3
    auto displacement = infNorm(pol); // "dir"
    if (auto v = std::sqrt((B + Btight) * (B + Btight) - B * B) / displacement; v < alpha)
        alpha = v;

    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
        vtemp.tuple(dim_c<3>, "xn0", i) = vtemp.pack(dim_c<3>, "xn", i);
    });
    auto E0 = constraintEnergy(pol); // "xn"
    auto c1m = armijoParam * dot(pol, "dir", "dir");
    fmt::print(fg(fmt::color::white), "c1m : {}\n", c1m);
    do {
        pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), alpha] ZS_LAMBDA(int i) mutable {
            vtemp.tuple(dim_c<3>, "xn", i) = vtemp.pack(dim_c<3>, "xn0", i) + alpha * vtemp.pack(dim_c<3>, "dir", i);
        });

        ///
        /// @note check c_ij(x^{l+1}). ref 4.2.2, item 1
        ///
        temp.setVal(0);
        pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), mark = proxy<space>(temp),
                         threshold = (B + Btight) * (B + Btight)] __device__(int i) mutable { // no constraints margin here according to paper 4.2.2
            auto pp = PP[i];
            auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]);
            auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]);
            if (auto d2 = dist2_pp(x0, x1); d2 < threshold)
                mark[0] = 1;
        });
        if (temp.getVal() == 0) {
            pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), mark = proxy<space>(temp),
                            threshold = L * L] __device__(int i) mutable { // no constraints margin here according to paper 4.2.2
                auto e = E[i];
                auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]);
                auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]);
                if (auto d2 = dist2_pp(x0, x1); d2 > threshold)
                    mark[0] = 1;
            });
        }

        /// @brief backtracking if discrete constraints violated
        if (temp.getVal() == 1) {
            alpha /= 2;
            fmt::print("\t[back-tracing] alpha: {} constraint not satisfied\n", alpha); 
            continue;
        }

        ///
        /// @note objective decreases adequately. ref 4.2.2, item 2
        ///
        auto E = constraintEnergy(pol);
        if (E <= E0 + alpha * c1m)
        {
            fmt::print("\t[back-tracing] alpha: {} line search finished!\n", alpha);
            break;
        }
        alpha /= 2;
    } while (true);
    fmt::print(fg(fmt::color::antique_white), "alpha_l^hard: {}\n", alpha);
}

bool FastClothSystem::constraintSatisfied(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    temp.setVal(0);
    pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), mark = proxy<space>(temp),
                     threshold = (B + Btight) * (B + Btight) + epsCond] __device__(int i) mutable { // epsCond: paper 4.2.2
        auto pp = PP[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]);
        auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]);
        auto x0k = vtemp.pack(dim_c<3>, "xk", pp[0]); 
        auto x1k = vtemp.pack(dim_c<3>, "xk", pp[1]); 
        auto ek = x1k - x0k, ek1 = x1 - x0; 
        auto dir = ek1 - ek; 
        auto de2 = dir.l2NormSqr(); 
        if (de2 > limits<T>::epsilon()) // check continuous constraints 4.2.1 & 4.1
        {
            auto numerator = -ek.dot(dir); 
            auto t = numerator / de2; 
            if (t > 0 && t < 1)
            {
                auto et = t * dir + ek;
                printf("t: %f, et.l2NormSqr: %f, threshold: %f\n", 
                    (float)t, (float)(et.l2NormSqr()), (float)threshold); 
                if (et.l2NormSqr() < threshold)
                {
                    // printf("et.l2NormSqr: %f\n", (float)(et.l2NormSqr())); 
                    mark[0] = 1; 
                    return; 
                }
            }
        } else {
            printf("\t\tcontinuous constraints met small edge displacement^2: %f(*1e-3)\n", 
                (float)(1e3f * de2)); 
        }
        if (auto d2 = dist2_pp(x0, x1); d2 < threshold)
            mark[0] = 1;
    });
    if (temp.getVal() == 0) {
        pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), mark = proxy<space>(temp),
                        threshold = L * L - epsCond] __device__(int i) mutable { // epsCond: paper 4.2.2
            auto e = E[i];
            auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]);
            auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]);
            if (auto d2 = dist2_pp(x0, x1); d2 > threshold)
                mark[0] = 1;
        });
    }
    // all constraints satisfied if temp.getVal() == 0
    return temp.getVal() == 0;
}

typename FastClothSystem::T FastClothSystem::constraintEnergy(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    temp.setVal(0);
    pol(range(numDofs),
        [vtemp = proxy<space>({}, vtemp), energy = proxy<space>(temp), n = numDofs] __device__(int i) mutable {
            auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
            auto xn = vtemp.pack(dim_c<3>, "xn", i);
            reduce_to(i, n, (xinit - xn).l2NormSqr(), energy[0]);
        });
    pol(range(npp),
        [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), energy = proxy<space>(temp), n = npp, mu = mu,
         Btot2 = (B + Btight) * (B + Btight), eps = epsSlack, a3 = a3, a2 = a2] __device__(int i) mutable {
            auto pp = PP[i];
            zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", pp[0]), vtemp.pack(dim_c<3>, "xn", pp[1])};
            T cij = (vs[1] - vs[0]).l2NormSqr() - Btot2;
            T f = eps;
            if (cij <= 0)
                printf("\n\n\nthis should not happen! pp constraint <%d, %d> cij: %f\n", (int)pp[0], (int)pp[1], cij);
            if (cij <= eps) {
                auto x2 = cij * cij;
                f = a3 * x2 * cij + a2 * x2 + cij;
            }
            T E = -mu * zs::log(f);
            reduce_to(i, n, E, energy[0]);
        });
    pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), energy = proxy<space>(temp), n = ne, mu = mu,
                    L2 = L * L, eps = epsSlack, a3 = a3, a2 = a2] __device__(int i) mutable {
        auto e = E[i];
        zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", e[0]), vtemp.pack(dim_c<3>, "xn", e[1])};
        T cij = L2 - (vs[1] - vs[0]).l2NormSqr();
        T f = eps;
        if (cij <= 0)
            printf("\n\n\nthis should not happen! edge constraint <%d, %d> cij: %f\n", (int)e[0], (int)e[1], cij);
        if (cij <= eps) {
            auto x2 = cij * cij;
            f = a3 * x2 * cij + a2 * x2 + cij;
        }
        T E = -mu * zs::log(f);
        reduce_to(i, n, E, energy[0]);
    });
    return temp.getVal();
}

#if 0
void FastClothSystem::computeConstraintGradients(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    auto [npp, ne] = getConstraintCnt();
    fmt::print("dcd broad phase [pp, edge constraints]: {}, {}", npp, ne);
    pol(range(npp),
        [vtemp = proxy<space>({}, vtemp), tempPP = proxy<space>({}, tempPP), PP = proxy<space>(PP), rho = rho, mu = mu,
         Btot2 = (B + Btight) * (B + Btight), eps = epsSlack] __device__(int i) mutable {
            auto pp = PP[i];
            zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", pp[0]), vtemp.pack(dim_c<3>, "xn", pp[1])};
            const auto &a = vs[0];
            const auto &b = vs[1];
            const auto t2 = a[0] * 2;
            const auto t3 = a[1] * 2;
            const auto t4 = a[2] * 2;
            const auto t5 = b[0] * 2;
            const auto t6 = b[1] * 2;
            const auto t7 = b[2] * 2;
            {

                const auto t8 = -t5;
                const auto t9 = -t6;
                const auto t10 = -t7;
                const auto t11 = t2 + t8;
                const auto t12 = t3 + t9;
                const auto t13 = t4 + t10;
                const auto t14 = rho * t11;
                const auto t15 = rho * t12;
                const auto t16 = rho * t13;
                auto grad = zs::vec<T, 6>{-t14, -t15, -t16, t14, t15, t16};
                tempPP.tuple(dim_c<6>, "softG", i) = grad;
            }
            {
                auto t8 = -Btot2;
                auto t9 = -b[0];
                auto t11 = -b[1];
                auto t13 = -b[2];
                auto t15 = 1 / eps;
                auto t10 = -t5;
                auto t12 = -t6;
                auto t14 = -t7;
                auto t16 = t15 * t15;
                auto t17 = a[0] + t9;
                auto t18 = a[1] + t11;
                auto t19 = a[2] + t13;
                auto t20 = t2 + t10;
                auto t21 = t3 + t12;
                auto t22 = t4 + t14;
                auto t23 = t17 * t17;
                auto t24 = t18 * t18;
                auto t25 = t19 * t10;
                auto t26 = t8 + t23 + t24 + t25;
                auto t27 = t26 * t26;
                auto t28 = t26 * t26 * t26;
                auto t32 = t15 * t20 * t26 * 2;
                auto t33 = t15 * t21 * t26 * 2;
                auto t34 = t15 * t22 * t26 * 2;
                auto t29 = t15 * t27;
                auto t30 = t16 * t28;
                auto t35 = t16 * t20 * t27 * 3;
                auto t36 = t16 * t21 * t27 * 3;
                auto t37 = t16 * t22 * t27 * 3;
                auto t31 = -t30;
                auto t38 = -t35;
                auto t39 = -t36;
                auto t40 = -t37;
                auto t41 = t20 + t32 + t38;
                auto t42 = t21 + t33 + t39;
                auto t43 = t22 + t34 + t40;
                auto t44 = t26 + t29 + t31;
                auto t45 = 1 / t44;
                auto t46 = mu * t41 * t45;
                auto t47 = mu * t42 * t45;
                auto t48 = mu * t43 * t45;
                auto grad = zs::vec<T, 6>{-t46, -t47, -t48, t46, t47, t48};
                tempPP.tuple(dim_c<6>, "hardG", i) = grad;
            }
        });

    pol(range(ne), [vtemp = proxy<space>({}, vtemp), tempE = proxy<space>({}, tempE), E = proxy<space>(E), rho = rho,
                    mu = mu, L2 = L * L, eps = epsSlack] __device__(int i) mutable {
        auto e = E[i];
        zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", e[0]), vtemp.pack(dim_c<3>, "xn", e[1])};
        const auto &a = vs[0];
        const auto &b = vs[1];
        const auto t2 = a[0] * 2;
        const auto t3 = a[1] * 2;
        const auto t4 = a[2] * 2;
        const auto t5 = b[0] * 2;
        const auto t6 = b[1] * 2;
        const auto t7 = b[2] * 2;
        {
            const auto t8 = -t5;
            const auto t9 = -t6;
            const auto t10 = -t7;
            const auto t11 = t2 + t8;
            const auto t12 = t3 + t9;
            const auto t13 = t4 + t10;
            const auto t14 = rho * t11;
            const auto t15 = rho * t12;
            const auto t16 = rho * t13;
            auto grad = zs::vec<T, 6>{t14, t15, t16, -t14, -t15, -t16};
            tempE.tuple(dim_c<6>, "softG", i) = grad;
        }
        {
            auto t8 = -L2;
            auto t12 = -b[0];
            auto t14 = -b[1];
            auto t16 = -b[2];
            auto t18 = 1 / eps;
            auto t9 = -t2;
            auto t10 = -t3;
            auto t11 = -t4;
            auto t13 = -t5;
            auto t15 = -t6;
            auto t17 = -t7;
            auto t19 = t18 * t18;
            auto t20 = a[0] + t12;
            auto t21 = a[1] + t14;
            auto t22 = a[2] + t16;
            auto t23 = t2 + t13;
            auto t24 = t3 + t15;
            auto t25 = t4 + t17;
            auto t26 = t20 * t20;
            auto t27 = t21 * t21;
            auto t28 = t22 * t22;
            auto t29 = -t26;
            auto t30 = -t27;
            auto t31 = -t28;
            auto t32 = t8 + t26 + t27 + t28;
            auto t33 = t32 * t32;
            auto t34 = t32 * t32 * t32;
            auto t37 = t18 * t23 * t32 * 2;
            auto t38 = t18 * t24 * t32 * 2;
            auto t39 = t18 * t25 * t32 * 2;
            auto t35 = t18 * t33;
            auto t36 = t19 * t34;
            auto t40 = t19 * t23 * t33 * 3;
            auto t41 = t19 * t24 * t33 * 3;
            auto t42 = t19 * t25 * t33 * 3;
            auto t43 = t5 + t9 + t37 + t40;
            auto t44 = t6 + t10 + t38 + t41;
            auto t45 = t7 + t11 + t39 + t42;
            auto t46 = L2 + t29 + t30 + t31 + t35 + t36;
            auto t47 = 1 / t46;
            auto t48 = mu * t43 * t47;
            auto t49 = mu * t44 * t47;
            auto t50 = mu * t45 * t47;
            auto grad = zs::vec<T, 6>{-t48, -t49, -t50, t48, t49, t50};
            tempE.tuple(dim_c<6>, "hardG", i) = grad;
        }
    });
}
#endif

} // namespace zeno