#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/Logger.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/geometry/PoissonDisk.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

#include "../geometry/linear_system/mfcg.hpp"

#include "../geometry/kernel/calculate_facet_normal.hpp"
#include "../geometry/kernel/topology.hpp"
#include "../geometry/kernel/compute_characteristic_length.hpp"
#include "../geometry/kernel/calculate_bisector_normal.hpp"

#include "../geometry/kernel/tiled_vector_ops.hpp"
#include "../geometry/kernel/geo_math.hpp"

#include "../geometry/kernel/calculate_edge_normal.hpp"

#include "zensim/container/Bvh.hpp"
#include "zensim/container/Bvs.hpp"
#include "zensim/container/Bvtt.hpp"

#include "collision_energy/vertex_face_sqrt_collision.hpp"
#include "collision_energy/vertex_face_collision.hpp"
#include "collision_energy/edge_edge_sqrt_collision.hpp"
#include "collision_energy/edge_edge_collision.hpp"




#include "collision_energy/evaluate_collision.hpp"

#define DEBUG_FLESH_DYN_STEPPING 1

namespace zeno {

// TODO : boundary force
// TODO : fixed points
// Anisotropic Cardiac

#define MAX_FP_COLLISION_PAIRS 6

struct FleshDynamicStepping : INode {

    using T = float;
    using Ti = int;
    using dtiles_t = zs::TileVector<T,32>;
    using tiles_t = typename ZenoParticles::particles_t;
    using vec3 = zs::vec<T, 3>;
    using mat3 = zs::vec<T, 3, 3>;
    using mat9 = zs::vec<T,9,9>;
    using mat12 = zs::vec<T,12,12>;

    using bvh_t = zs::LBvh<3,int,T>;
    using bv_t = zs::AABBBox<3, T>;

    using pair3_t = zs::vec<Ti,3>;
    using pair4_t = zs::vec<Ti,4>;

    // currently only backward euler integrator is supported
    // topology evaluation should be called before applying this node
    struct FEMDynamicSteppingSystem {

        constexpr auto dFAdF(const mat3& A) {
            mat9 M{};
            M(0,0) = M(1,1) = M(2,2) = A(0,0);
            M(3,0) = M(4,1) = M(5,2) = A(0,1);
            M(6,0) = M(7,1) = M(8,2) = A(0,2);

            M(0,3) = M(1,4) = M(2,5) = A(1,0);
            M(3,3) = M(4,4) = M(5,5) = A(1,1);
            M(6,3) = M(7,4) = M(8,5) = A(1,2);

            M(0,6) = M(1,7) = M(2,8) = A(2,0);
            M(3,6) = M(4,7) = M(5,8) = A(2,1);
            M(6,6) = M(7,7) = M(8,8) = A(2,2);

            return M;        
        }

        template <typename Model>
        void computeCollisionEnergy(zs::CudaExecutionPolicy& cudaPol,const Model& model,
                dtiles_t& vtemp,
                dtiles_t& etemp,
                dtiles_t& sttemp,
                dtiles_t& setemp,
                dtiles_t& cptemp,
                // const bvh_t& stBvh,
                // const bvh_t& seBvh,
                const T& thickness) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            T lambda = model.lam;
            T mu = model.mu;


        }


        template <typename Model>
        void computeCollisionGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,const Model& model,
                            dtiles_t& vtemp,
                            dtiles_t& etemp,
                            dtiles_t& sttemp,
                            dtiles_t& setemp,
                            dtiles_t& cptemp,
                            // const bvh_t& stBvh,
                            // const bvh_t& seBvh,
                            const T& thickness,
                            bool explicit_collision = false,
                            bool neglect_inverted = true) {
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            T lambda = model.lam;
            T mu = model.mu; 

            #if DEBUG_FLESH_DYN_STEPPING
                if(!vtemp.hasProperty("grad"))
                    fmt::print(fg(fmt::color::red),"the vtemp has no 'grad' channel\n");
                if(!vtemp.hasProperty("xn"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'xn' channel\n");
                if(!vtemp.hasProperty("xp"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'xn' channel\n");
                if(!vtemp.hasProperty("is_inverted"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'is_inverted' channel\n");
                if(!vtemp.hasProperty("vp"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'vp' channel\n");

                if(!etemp.hasProperty("H"))
                    fmt::print(fg(fmt::color::red),"the etemp has no 'H' channel\n");
                if(!etemp.hasProperty("ActInv"))
                    fmt::print(fg(fmt::color::red),"the etemp has no 'ActInv' channel\n");
                
                if(!verts.hasProperty("m"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'm' channel\n");

                if(!eles.hasProperty("inds"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'IB' channel\n");        
                if(!eles.hasProperty("IB"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'IB' channel\n");
                if(!eles.hasProperty("m"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'm' channel\n");
                if(!eles.hasProperty("vol"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'vol' channel\n");

                // fmt::print(fg(fmt::color::blue),"the size of tris : {}\n",tris.size());
                if(!tris.hasProperty("inds"))
                    fmt::print(fg(fmt::color::red),"the tris has no 'inds' channel\n");
                if(!tris.hasProperty("area"))
                    fmt::print(fg(fmt::color::red),"the tris has no 'area' channel\n");
                if(!points.hasProperty("area"))
                    fmt::print(fg(fmt::color::red),"the points has no 'area' channel\n");

            #endif            

            auto xtag = zs::SmallString("xn");
            if(explicit_collision)
                xtag = zs::SmallString("xp");


            if(neglect_inverted) {
            // // figure out all the vertices which is incident to an inverted tet
                TILEVEC_OPS::fill(cudaPol,vtemp,"is_inverted",reinterpret_bits<T>((int)0));  
                cudaPol(zs::range(eles.size()),
                    [vtemp = proxy<space>({},vtemp),quads = proxy<space>({},eles),xtag] ZS_LAMBDA(int ei) mutable {
                        auto DmInv = quads.template pack<3,3>("IB",ei);
                        auto inds = quads.template pack<4>("inds",ei).reinterpret_bits(int_c);
                        vec3 x1[4] = {vtemp.template pack<3>(xtag, inds[0]),
                                vtemp.template pack<3>(xtag, inds[1]),
                                vtemp.template pack<3>(xtag, inds[2]),
                                vtemp.template pack<3>(xtag, inds[3])};   

                        mat3 F{};
                        {
                            auto x1x0 = x1[1] - x1[0];
                            auto x2x0 = x1[2] - x1[0];
                            auto x3x0 = x1[3] - x1[0];
                            auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                            x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                            F = Ds * DmInv;
                        } 
                        if(zs::determinant(F) < 0.0)
                            for(int i = 0;i < 4;++i)
                                vtemp("is_inverted",inds[i]) = reinterpret_bits<T>((int)1);                  
                });

            }


#if 0
            TILEVEC_OPS::fill<4>(cudaPol,cptemp,"inds",zs::vec<int,4>::uniform(-1).template reinterpret_bits<T>());
            // TILEVEC_OPS::fill<12*12>(cudaPol,cptemp,"H",zs::vec<T,12*12>::zeros());

            // compute vertex facet contact pairs
            cudaPol(zs::range(points.size()),[lambda = lambda,mu = mu,collisionStiffness = collisionStiffness,
                            in_collisionEps = in_collisionEps,out_collisionEps = out_collisionEps,
                            vtemp = proxy<space>({},vtemp),
                            etemp = proxy<space>({},etemp),
                            sttemp = proxy<space>({},sttemp),
                            setemp = proxy<space>({},setemp),
                            cptemp = proxy<space>({},cptemp),
                            points = proxy<space>({},points),
                            lines = proxy<space>({},lines),
                            tris = proxy<space>({},tris),
                            stbvh = proxy<space>(stBvh),thickness = thickness,
                            neglect_inverted = neglect_inverted,xtag] ZS_LAMBDA(int svi) mutable {
                // if(svi == 0)    {
                //     if(tris.hasProperty("inds"))
                //         printf("compare size : %d %d %d\n",(int)vtemp.size(),(int)tris.size(),(int)tris.propertySize("inds"));
                //     else
                //         printf("the tris has no inds channel!!!\n"); 
                // }


                auto vi = reinterpret_bits<int>(points("inds",svi));

                if(neglect_inverted)   {
                    auto is_vertex_inverted = reinterpret_bits<int>(vtemp("is_inverted",vi));
                    if(is_vertex_inverted)
                        return;
                }

                auto p = vtemp.template pack<3>(xtag,vi);
                auto bv = bv_t{get_bounding_box(p - thickness, p + thickness)};


                // check whether there is collision happening, and if so, apply the collision force and addup the collision hessian
                int nm_collision_pairs = 0;
                auto process_vertex_face_collision_pairs = [&](int stI) {

                    if(nm_collision_pairs >= MAX_FP_COLLISION_PAIRS)     
                        return;   

                    auto tri = tris.pack(dim_c<3>, "inds",stI).reinterpret_bits(int_c);
                    if(tri[0] == vi || tri[1] == vi || tri[2] == vi)
                        return;

                    auto t0 = vtemp.template pack<3>(xtag,tri[0]);
                    auto t1 = vtemp.template pack<3>(xtag,tri[1]);
                    auto t2 = vtemp.template pack<3>(xtag,tri[2]);
                    // check whether the triangle is degenerate
                    auto restArea = tris("area",stI);
                    // skip the triangle too small at rest configuration
                    // if(restArea < (T)1e-6)
                    //     return;

                    const auto e10 = t1 - t0;
                    const auto e20 = t2 - t0;
                    auto deformedArea = (T)0.5 * e10.cross(e20).norm();
                    const T degeneracyEps = 1e-4;
                    // skip the degenerate triangles
                    const T relativeArea = deformedArea / (restArea + (T)1e-6);
                    if(relativeArea < degeneracyEps)
                        return;

                    bool collide = false;

                    if(COLLISION_UTILS::is_inside_the_cell(vtemp,xtag,
                            lines,tris,
                            sttemp,"nrm",
                            setemp,"nrm",
                            stI,p,in_collisionEps,out_collisionEps)){
                        // printf("find collision facet-vertex collision in-cell pair : %d %d\n",stI,svi);
                        collide = true;
                    }

                    if(!collide)
                        return;

                    // now there is collision, build the "collision tets"
                    // if(!vtemp.hasProperty("oneRingArea"))
                    //     printf("vtemp has no oneRingArea");

                    cptemp.template tuple<4>("inds",svi * MAX_FP_COLLISION_PAIRS + nm_collision_pairs) = zs::vec<int,4>(vi,tri[0],tri[1],tri[2]).template reinterpret_bits<T>();

                    auto vertexFaceCollisionAreas = restArea + points("area",svi);
                    
                    vec3 collision_verts[4] = {};
                    collision_verts[0] = p;
                    collision_verts[1] = t0;
                    collision_verts[1] = t1;
                    collision_verts[1] = t2;

                    auto collisionEps = in_collisionEps;

                    auto grad = collisionStiffness * VERTEX_FACE_SQRT_COLLISION::gradient(collision_verts,mu,lambda,collisionEps) * vertexFaceCollisionAreas;
                    auto hessian = collisionStiffness * VERTEX_FACE_SQRT_COLLISION::hessian(collision_verts,mu,lambda,collisionEps) * vertexFaceCollisionAreas;
                    cptemp.template tuple<12*12>("H",svi * MAX_FP_COLLISION_PAIRS + nm_collision_pairs) = hessian;

                    for(int i = 0;i != 4;++i) {
                        auto g_vi = i == 0 ? vi : tri[i-1];
                        for (int d = 0; d != 3; ++d)
                            atomic_add(exec_cuda, &vtemp("grad", d, g_vi), grad(i * 3 + d));
                    }
                    nm_collision_pairs++;

                };
                stbvh.iter_neighbors(bv,process_vertex_face_collision_pairs);
            });

#else

        COLLISION_UTILS::do_facet_point_collision_detection<MAX_FP_COLLISION_PAIRS>(cudaPol,
            vtemp,"xn",
            points,
            lines,
            tris,
            sttemp,
            setemp,
            cptemp,
            // stBvh,
            in_collisionEps,out_collisionEps);


        // output all the collision pairs
        // cudaPol(zs::range(cptemp.size()),
        //     [cptemp = proxy<space>({},cptemp)] ZS_LAMBDA(int cpi) mutable {
        //         auto inds = cptemp.template pack<4>("inds",cpi).reinterpret_bits(int_c);
        //         bool collide = true;
        //         for(int i = 0;i != 4;++i)
        //             if(inds[i] < 0)
        //                 collide = false;
        //         if(collide)
        //             printf("collision_pair[%d] : %d %d %d %d\n",
        //                 cpi,inds[0],inds[1],inds[2],inds[3]);
        // });

        COLLISION_UTILS::evaluate_collision_grad_and_hessian<MAX_FP_COLLISION_PAIRS>(cudaPol,
            vtemp,"xn",
            cptemp,
            in_collisionEps,out_collisionEps,
            (T)collisionStiffness,
            (T)mu,(T)lambda);



        // project out all the neglect verts
        if(neglect_inverted) {
            cudaPol(zs::range(cptemp.size()),
                [cptemp = proxy<space>({},cptemp),vtemp = proxy<space>({},vtemp)] ZS_LAMBDA(int cpi) {
                    auto inds = cptemp.template pack<4>("inds",cpi).reinterpret_bits(int_c);
                    for(int i = 0;i != 4;++i)
                        if(inds[i] < 0)
                            return;

                    bool is_inverted = false;
                    for(int i = 0;i != 4;++i){
                        auto vi = inds[i];
                        auto is_vertex_inverted = reinterpret_bits<int>(vtemp("is_inverted",vi));
                        if(is_vertex_inverted)
                            is_inverted = true;
                    }

                    if(is_inverted){
                        cptemp.template tuple<12*12>("H",cpi) = zs::vec<T,12,12>::zeros();
                        cptemp.template tuple<12>("grad",cpi) = zs::vec<T,12>::zeros();
                    }
            });    
        }

        // auto gradN = TILEVEC_OPS::inf_norm<12>(cudaPol,cptemp,"grad");
        // fmt::print(fg(fmt::color::red),"collision gradN = {}\n",gradN);
        // TILEVEC_OPS::fill<12*12>(cudaPol,cptemp,"H",zs::vec<T,12*12>::zeros());

        TILEVEC_OPS::assemble<3,4>(cudaPol,cptemp,"grad",vtemp,"grad");


#endif


        }


        template <typename Model>
        void computeGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
                            const Model& model,
                            dtiles_t& vtemp,
                            dtiles_t& etemp) {        
            using namespace zs;
            constexpr auto space = execspace_e::cuda;

            #if DEBUG_FLESH_DYN_STEPPING
                // std::cout << "CHECK THE PROPERTY CHANNEL" << std::endl;
                if(!vtemp.hasProperty("grad"))
                    fmt::print(fg(fmt::color::red),"the vtemp has no 'grad' channel\n");
                if(!vtemp.hasProperty("xn"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'xn' channel\n");
                if(!vtemp.hasProperty("xp"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'xp' channel\n");
                if(!vtemp.hasProperty("vp"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'vp' channel\n");

                if(!etemp.hasProperty("H"))
                    fmt::print(fg(fmt::color::red),"the etemp has no 'H' channel\n");
                if(!etemp.hasProperty("ActInv"))
                    fmt::print(fg(fmt::color::red),"the etemp has no 'ActInv' channel\n");
                
                if(!verts.hasProperty("m"))
                    fmt::print(fg(fmt::color::red),"the verts has no 'm' channel\n");

                if(!eles.hasProperty("IB"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'IB' channel\n");
                if(!eles.hasProperty("m"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'm' channel\n");
                if(!eles.hasProperty("vol"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'vol' channel\n");
                if(!eles.hasProperty("inds"))
                    fmt::print(fg(fmt::color::red),"the eles has no 'inds' channel\n");
            #endif

            TILEVEC_OPS::fill<3>(cudaPol,vtemp,"grad",zs::vec<T,3>::zeros());
            TILEVEC_OPS::fill<144>(cudaPol,etemp,"H",zs::vec<T,144>::zeros());         
            
            // eval the inertia term gradient
            cudaPol(zs::range(vtemp.size()), [dt2 = dt2,
                        vtemp = proxy<space>({},vtemp),
                        verts = proxy<space>({},verts),
                        dt = dt] ZS_LAMBDA(int vi) mutable {
                auto m = verts("m",vi);// nodal mass
                auto x1 = vtemp.pack<3>("xn",vi);
                auto x0 = vtemp.pack<3>("xp",vi);
                auto v0 = vtemp.pack<3>("vp",vi);
                vtemp.template tuple<3>("grad",vi) = -m * (x1 - x0 - v0 * dt) / dt2;                    
            });

            cudaPol(zs::range(eles.size()), [this,dt2 = dt2,
                            vtemp = proxy<space>({}, vtemp),
                            etemp = proxy<space>({}, etemp),
                            bcws = proxy<space>({},b_bcws),
                            b_verts = proxy<space>({},b_verts),
                            verts = proxy<space>({}, verts),
                            eles = proxy<space>({}, eles),
                            model, volf = volf] ZS_LAMBDA (int ei) mutable {
                    auto DmInv = eles.template pack<3,3>("IB",ei);
                    auto dFdX = dFdXMatrix(DmInv);
                    auto inds = eles.template pack<4>("inds",ei).reinterpret_bits(int_c);
                    vec3 x1[4] = {vtemp.template pack<3>("xn", inds[0]),
                            vtemp.template pack<3>("xn", inds[1]),
                            vtemp.template pack<3>("xn", inds[2]),
                            vtemp.template pack<3>("xn", inds[3])};   

                    mat3 FAct{};
                    {
                        auto x1x0 = x1[1] - x1[0];
                        auto x2x0 = x1[2] - x1[0];
                        auto x3x0 = x1[3] - x1[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                        x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        FAct = Ds * DmInv;

                        FAct = FAct * etemp.template pack<3,3>("ActInv",ei);
                    } 
                    auto dFActdF = dFAdF(etemp.template pack<3,3>("ActInv",ei));

                    // add the force term in gradient
                    auto P = model.first_piola(FAct);
                    auto vole = eles("vol", ei);
                    auto vecP = flatten(P);
                    vecP = dFActdF.transpose() * vecP;
                    auto dFdXT = dFdX.transpose();
                    auto vf = -vole * (dFdXT * vecP);     

                    auto mg = volf * vole / 4;
                    for (int i = 0; i != 4; ++i) {
                        auto vi = inds[i];
                        for (int d = 0; d != 3; ++d)
                            atomic_add(exec_cuda, &vtemp("grad", d, vi), vf(i * 3 + d) + mg(d));
                    }

                    // assemble element-wise hessian matrix
                    auto Hq = model.first_piola_derivative(FAct, true_c);
                    auto dFdAct_dFdX = dFActdF * dFdX; 
                    // dFdAct_dFdX = dFdX; 
                    auto H = dFdAct_dFdX.transpose() * Hq * dFdAct_dFdX * vole;
                    etemp.template tuple<12 * 12>("H", ei) = H;

                    // add inertia hessian term
                    auto m = eles("m",ei);// element-wise mass
                    for(int i = 0;i < 12;++i){
                        // Mass(i,i) = 1;
                        etemp("H",i * 12 + i,ei) += m /dt2/4;
                    }


            });
        // Bone Driven Potential Energy
            T lambda = model.lam;
            T mu = model.mu;
            auto nmEmbedVerts = b_verts.size();
            cudaPol(zs::range(nmEmbedVerts), [this,
                    bcws = proxy<space>({},b_bcws),b_verts = proxy<space>({},b_verts),vtemp = proxy<space>({},vtemp),etemp = proxy<space>({},etemp),
                    eles = proxy<space>({},eles),lambda,mu,bone_driven_weight = bone_driven_weight] ZS_LAMBDA(int vi) mutable {
                        auto ei = reinterpret_bits<int>(bcws("inds",vi));
                        if(ei < 0)
                            return;
                        auto inds = eles.pack<4>("inds",ei).reinterpret_bits<int>();
                        auto w = bcws.pack<4>("w",vi);
                        auto tpos = vec3::zeros();
                        for(size_t i = 0;i != 4;++i)
                            tpos += w[i] * vtemp.pack<3>("xn",inds[i]);
                        auto pdiff = tpos - b_verts.pack<3>("x",vi);

                        T stiffness = 2.0066 * mu + 1.0122 * lambda;

                        for(size_t i = 0;i != 4;++i){
                            auto tmp = pdiff * (-stiffness * bcws("cnorm",vi) * bone_driven_weight * w[i] * eles("vol",ei)); 
                            // tmp = pdiff * (-lambda * bcws("cnorm",vi) * bone_driven_weight * w[i]);
                            for(size_t d = 0;d != 3;++d)
                                atomic_add(exec_cuda,&vtemp("grad",d,inds[i]),(T)tmp[d]);
                        }
                        for(int i = 0;i != 4;++i)
                            for(int j = 0;j != 4;++j){
                                T alpha = stiffness * bone_driven_weight * w[i] * w[j] * bcws("cnorm",vi) * eles("vol",ei);
                                for(int d = 0;d != 3;++d){
                                    atomic_add(exec_cuda,&etemp("H",(i * 3 + d) * 12 + j * 3 + d,ei),alpha);
                                }
                            }

            });

        }


        FEMDynamicSteppingSystem(const tiles_t &verts, const tiles_t &eles,
                const tiles_t& points,const tiles_t& lines,const tiles_t& tris,
                T in_collisionEps,T out_collisionEps,
                const tiles_t &b_bcws, const tiles_t& b_verts,T bone_driven_weight,
                vec3 volf,const T& _dt,const T& collisionStiffness)
            : verts{verts}, eles{eles},points{points}, lines{lines}, tris{tris},
                    in_collisionEps{in_collisionEps},out_collisionEps{out_collisionEps},
                    b_bcws{b_bcws}, b_verts{b_verts}, bone_driven_weight{bone_driven_weight},
                    volf{volf},
                    dt{_dt}, dt2{dt * dt},collisionStiffness{collisionStiffness},use_edge_edge_collision{true}, use_vertex_facet_collision{true} {}

        const tiles_t &verts;
        const tiles_t &eles;
        const tiles_t &points;
        const tiles_t &lines;
        const tiles_t &tris;
        const tiles_t &b_bcws;  // the barycentric interpolation of embeded bones 
        const tiles_t &b_verts; // the position of embeded bones

        T bone_driven_weight;
        vec3 volf;
        T dt;
        T dt2;
        T in_collisionEps;
        T out_collisionEps;

        T collisionStiffness;

        bool bvh_initialized;
        bool use_edge_edge_collision;
        bool use_vertex_facet_collision;

        // int default_muscle_id;
        // zs::vec<T,3> default_muscle_dir;
        // T default_act;

        // T inset;
        // T outset;
    };



    void apply() override {
        using namespace zs;
        auto zsparticles = get_input<ZenoParticles>("ZSParticles");
        auto gravity = zeno::vec<3,T>(0);
        if(has_input("gravity"))
            gravity = get_input<zeno::NumericObject>("gravity")->get<zeno::vec<3,T>>();
        T armijo = (T)1e-4;
        T wolfe = (T)0.9;
        // T cg_res = (T)0.001;
        T cg_res = (T)0.0001;
        T btl_res = (T)0.1;
        auto models = zsparticles->getModel();
        auto& verts = zsparticles->getParticles();
        auto& eles = zsparticles->getQuadraturePoints();

        if(eles.getChannelSize("inds") != 4)
            throw std::runtime_error("the input zsparticles is not a tetrahedra mesh");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfTriTag))
            throw std::runtime_error("the input zsparticles has no surface tris");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfEdgeTag))
            throw std::runtime_error("the input zsparticles has no surface lines");
        if(!zsparticles->hasAuxData(ZenoParticles::s_surfVertTag)) 
            throw std::runtime_error("the input zsparticles has no surface points");
        // if(!zsparticles->hasBvh(ZenoParticles::s_surfTriTag)) {
        //     throw std::runtime_error("the input zsparticles has no surface tris's spacial structure");
        // }
        // if(!zsparticles->hasBvh(ZenoParticles::s_surfEdgeTag)) {
        //     throw std::runtime_error("the input zsparticles has no surface edge's spacial structure");
        // }
        // if(!zsparticles->hasBvh(ZenoParticles::s_surfVertTag))  {
        //     throw std::runtime_error("the input zsparticles has no surface vert's spacial structure");
        // }

        auto& tris  = (*zsparticles)[ZenoParticles::s_surfTriTag];
        auto& lines = (*zsparticles)[ZenoParticles::s_surfEdgeTag];
        auto& points = (*zsparticles)[ZenoParticles::s_surfVertTag];

        // auto& stBvh = zsparticles->bvh(ZenoParticles::s_surfTriTag);
        // auto& seBvh = zsparticles->bvh(ZenoParticles::s_surfEdgeTag);


        auto zsbones = get_input<ZenoParticles>("driven_boudary");
        auto driven_tag = get_input2<std::string>("driven_tag");
        auto bone_driven_weight = get_input2<float>("driven_weight");
        auto muscle_id_tag = get_input2<std::string>("muscle_id_tag");
        // auto bone_driven_weight = (T)0.02;



        auto newton_res = (T)0.01;

        auto dt = get_input2<float>("dt");

        auto volf = vec3::from_array(gravity * models.density);

        std::vector<float> act_;    
        std::size_t nm_acts = 0;

        if(has_input("Acts")) {
            act_ = get_input<zeno::ListObject>("Acts")->getLiterial<float>();
            nm_acts = act_.size();
        }

        constexpr auto host_space = zs::execspace_e::openmp;
        auto ompExec = zs::omp_exec();
        auto act_buffer = dtiles_t{{{"act",1}},nm_acts,zs::memsrc_e::host};
        ompExec(zs::range(act_buffer.size()),
            [act_buffer = proxy<host_space>({},act_buffer),act_] (int i) mutable {
                act_buffer("act",i) = act_[i];
        });
        act_buffer = act_buffer.clone({zs::memsrc_e::device, 0});

        // the temp buffer only store the data that will change every iterations or every frame
        static dtiles_t vtemp{verts.get_allocator(),
                            {
                                {"grad", 3},
                                {"P", 9},
                                {"bou_tag",1},
                                {"dir", 3},
                                {"xn", 3},
                                {"xp",3},
                                {"vp",3},
                                {"is_inverted",1},
                                {"active",1}
                            },verts.size()};

        // auto max_collision_pairs = tris.size() / 10; 
        static dtiles_t etemp{eles.get_allocator(), {
                {"H", 12 * 12},
                {"inds",4},
                {"ActInv",3*3},
                // {"muscle_ID",1},
                // {"fiber",3}
                }, eles.size()};

                // {{tags}, cnt, memsrc_e::um, 0}
        static dtiles_t sttemp(tris.get_allocator(),
            {
                {"nrm",3}
            },tris.size()
        );
        static dtiles_t setemp(lines.get_allocator(),
            {
                {"nrm",3}
            },lines.size()
        );

        static dtiles_t cptemp(points.get_allocator(),{
            {"inds",4},
            {"area",1},
            {"grad",12},
            {"inverted",1},
            {"H",12 * 12}
        },points.size() * MAX_FP_COLLISION_PAIRS);


        constexpr auto space = execspace_e::cuda;
        auto cudaPol = cuda_exec();
    

        // TILEVEC_OPS::fill<4>(cudaPol,etemp,"inds",zs::vec<int,4>::uniform(-1).template reinterpret_bits<T>())
        TILEVEC_OPS::copy<4>(cudaPol,eles,"inds",etemp,"inds");

        auto avgl = compute_average_edge_length(cudaPol,verts,"x",tris);
        // auto avgl = (T)1.0;

        auto collisionStiffness = get_input2<float>("cstiffness");


        // auto inset_ratio = get_input2<float>("collision_inset");
        // auto outset_ratio = get_input2<float>("collision_outset");    

        auto in_collisionEps = get_input2<float>("in_collisionEps");
        auto out_collisionEps = get_input2<float>("out_collisionEps");

        FEMDynamicSteppingSystem A{
            verts,eles,
            points,lines,tris,
            (T)in_collisionEps,(T)out_collisionEps,
            (*zsparticles)[driven_tag],zsbones->getParticles(),bone_driven_weight,
            volf,dt,collisionStiffness};


        // TILEVEC_OPS::fill<9>(cudaPol,etemp,"ActInv",zs::vec<T,9>{1.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,1.0});
        // apply muscle activation
        cudaPol(zs::range(eles.size()),
            [etemp = proxy<space>({},etemp),eles = proxy<space>({},eles),
                act_buffer = proxy<space>({},act_buffer),muscle_id_tag = SmallString(muscle_id_tag),nm_acts,avgl] ZS_LAMBDA(int ei) mutable {
                // auto act = eles.template pack<3>("act",ei);
                // auto fiber = etemp.template pack<3>("fiber",ei);
                zs::vec<T,3> fiber{};
                if(!eles.hasProperty("fiber"))
                    fiber = eles.template pack<3>("fiber",ei);
                else 
                    fiber = zs::vec<T,3>(1.0,0.0,0.0);
                vec3 act{1.0,1.0,1.0};


                auto nfiber = fiber.norm();
                // auto ID = etemp("muscle_ID",ei);
                int ID = -1;
                if(eles.hasProperty(muscle_id_tag))
                    ID = (int)eles(muscle_id_tag,ei);
                
                if(nm_acts > 0 && ID > -1){
                    float a = 1. - act_buffer("act",ID);
                    act = vec3{1,zs::sqrt(1./a),zs::sqrt(1./a)};
                }

                vec3 dir[3];
                dir[0] = fiber;
                auto tmp = vec3{0.0,1.0,0.0};
                dir[1] = dir[0].cross(tmp);
                if(dir[1].length() < 1e-3) {
                    tmp = vec3{0.0,0.0,1.0};
                    dir[1] = dir[0].cross(tmp);
                }

                dir[1] = dir[1] / dir[1].length();
                dir[2] = dir[0].cross(dir[1]);
                dir[2] = dir[2] / dir[2].length();

                auto R = mat3{};
                for(int i = 0;i < 3;++i)
                    for(int j = 0;j < 3;++j)
                        R(i,j) = dir[j][i];

                auto Act = mat3::zeros();
                Act(0,0) = act[0];
                Act(1,1) = act[1];
                Act(2,2) = act[2];

                Act = R * Act * R.transpose();

                // if(ei == 0) {
                //     printf("Act[0]:\n%f %f %f\n%f %f %f\n%f %f %f\n",
                //         (float)Act(0,0),(float)Act(0,1),(float)Act(0,2),
                //         (float)Act(1,0),(float)Act(1,1),(float)Act(1,2),
                //         (float)Act(2,0),(float)Act(2,1),(float)Act(2,2));
                // }

                etemp.template tuple<9>("ActInv",ei) = zs::inverse(Act);
        });
        // std::cout << "set initial guess" << std::endl;
        // setup initial guess
        TILEVEC_OPS::copy<3>(cudaPol,verts,"x",vtemp,"xp");
        TILEVEC_OPS::copy<3>(cudaPol,verts,"v",vtemp,"vp");
        TILEVEC_OPS::copy(cudaPol,verts,"active",vtemp,"active");
        if(verts.hasProperty("init_x"))
            TILEVEC_OPS::copy<3>(cudaPol,verts,"init_x",vtemp,"xn");   
        else {
            // TILEVEC_OPS::add<3>(cudaPol,vtemp,"xp",1.0,"vp",dt,"xn");  
            TILEVEC_OPS::add<3>(cudaPol,vtemp,"xp",1.0,"vp",(T)0.0,"xn");  
        }
        TILEVEC_OPS::fill(cudaPol,vtemp,"bou_tag",(T)0.0);


        auto bvh_thickness = 5 * avgl;

        int max_newton_iterations = 5;
        int nm_iters = 0;

        // make sure, at least one baraf simi-implicit step will be taken
        auto res0 = 1e10;

        while(nm_iters < max_newton_iterations) {

            match([&](auto &elasticModel) {
                A.computeGradientAndHessian(cudaPol, elasticModel,vtemp,etemp);
            })(models.getElasticModel());

            bool include_collision = true;
            if(include_collision) {

                // if(!calculate_facet_normal(cudaPol,vtemp,"xn",tris,sttemp,"nrm")){
                //         throw std::runtime_error("fail updating facet normal");
                // }

                // if(!COLLISION_UTILS::calculate_cell_bisector_normal(cudaPol,
                //     vtemp,"xn",
                //     lines,
                //     tris,
                //     sttemp,"nrm",
                //     setemp,"nrm")){
                //         throw std::runtime_error("fail calculate cell bisector normal");
                // }

                // auto stbvs = retrieve_bounding_volumes(cudaPol,vtemp,tris,wrapv<3>{},(T)0.0,"xn");
                // auto sebvs = retrieve_bounding_volumes(cudaPol,vtemp,lines,wrapv<2>{},(T)0.0,"xn");
                // stBvh.refit(cudaPol,stbvs);
                // seBvh.refit(cudaPol,sebvs);

                match([&](auto &elasticModel) {
                    A.computeCollisionGradientAndHessian(cudaPol,elasticModel,
                        vtemp,
                        etemp,
                        sttemp,
                        setemp,
                        cptemp,
                        // stBvh,
                        // seBvh,
                        bvh_thickness);
                })(models.getElasticModel());

            }

            PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",etemp,cptemp,"P",vtemp);
            // PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"H",etemp,"P",vtemp);
            // if the grad is too small, return the result
            // Solve equation using PCG
            TILEVEC_OPS::fill<3>(cudaPol,vtemp,"dir",zs::vec<T,3>::zeros());
            // std::cout << "solve using pcg" << std::endl;
            PCG::pcg_with_fixed_sol_solve<3,4>(cudaPol,vtemp,etemp,cptemp,"dir","bou_tag","grad","P","inds","H",cg_res,1000,50);
            // PCG::pcg_with_fixed_sol_solve<3,4>(cudaPol,vtemp,etemp,"dir","bou_tag","grad","P","inds","H",cg_res,1000,50);
            // std::cout << "finish solve pcg" << std::endl;
            PCG::project<3>(cudaPol,vtemp,"dir","bou_tag");
            T alpha = 1.;
            cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),alpha] __device__(int i) mutable {
                vtemp.tuple<3>("xn", i) =
                    vtemp.pack<3>("xn", i) + alpha * vtemp.pack<3>("dir", i);
            });

            T res = TILEVEC_OPS::inf_norm<3>(cudaPol, vtemp, "dir");// this norm is independent of descriterization
            std::cout << "res[" << nm_iters << "] : " << res << std::endl;
            if(res < 1e-3)
                break;

            // keep dropping, to avoid explosion
            if(res < res0)
                res0 = res;
            else {
                // reverse 
                cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),alpha] __device__(int i) mutable {
                    vtemp.tuple<3>("xn", i) =
                        vtemp.pack<3>("xn", i) - alpha * vtemp.pack<3>("dir", i);
                });

                break;
            }
            nm_iters++;
        }



        cudaPol(zs::range(verts.size()),
                [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),dt] __device__(int vi) mutable {
                    auto newX = vtemp.pack<3>("xn", vi);
                    verts.tuple<3>("x", vi) = newX;
                    verts.tuple<3>("v",vi) = (vtemp.pack<3>("xn",vi) - vtemp.pack<3>("xp",vi))/dt;
                });

        dtiles_t nodalForceVis(verts.get_allocator(),
            {
                {"x",3},
                {"dir",3},
            },verts.size());



        // TILEVEC_OPS::copy<3>(cudaPol,vtemp,"xn",nodalForceVis,"x");
        // TILEVEC_OPS::fill<3>(cudaPol,nodalForceVis,"dir",zs::vec<T,3>::zeros());
        // TILEVEC_OPS::assemble<3,4>(cudaPol,cptemp,"grad",nodalForceVis,"dir");





        set_output("ZSParticles", zsparticles);
    }


};

ZENDEFNODE(FleshDynamicStepping, {{"ZSParticles",
                                    "gravity","Acts",
                                    "driven_boudary",
                                    {"string","driven_tag","bone_bw"},
                                    {"float","driven_weight","0.02"},
                                    {"string","muscle_id_tag","ms_id_tag"},
                                    {"float","cstiffness","0.0"},
                                    {"float","in_collisionEps","0.01"},
                                    {"float","out_collisionEps","0.01"},
                                    {"float","dt","0.5"}
                                    },
                                  {"ZSParticles"},
                                  {
                                  },
                                  {"FEM"}});



};