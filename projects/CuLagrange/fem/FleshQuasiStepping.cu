#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/Logger.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/geometry/PoissonDisk.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

#include "../geometry/linear_system/mfcg.hpp"

namespace zeno {


struct FleshQuasiStaticStepping : INode {
  using T = float;
  using dtiles_t = zs::TileVector<T,32>;
  using tiles_t = typename ZenoParticles::particles_t;
  using vec3 = zs::vec<T, 3>;
  using mat3 = zs::vec<T, 3, 3>;
  struct FEMQuasiStaticSystem {

    constexpr auto dFAdF(const mat3& A) {
        zs::vec<T,9,9> M{};
        M(0,0) = M(1,1) = M(2,2) = A(0,0);
        M(3,0) = M(4,1) = M(5,2) = A(0,1);
        M(6,0) = M(7,1) = M(8,2) = A(0,2);

        M(0,3) = M(1,4) = M(2,5) = A(1,0);
        M(3,3) = M(4,4) = M(5,5) = A(1,1);
        M(6,3) = M(7,4) = M(8,5) = A(1,2);

        M(0,6) = M(1,7) = M(2,8) = A(2,0);
        M(3,6) = M(4,7) = M(5,8) = A(2,1);
        M(6,6) = M(7,7) = M(8,8) = A(2,2);

        return M;        
    }


    template <typename Pol, typename Model>
    T energy(Pol &pol, const Model &model, const zs::SmallString tag, dtiles_t& vtemp,dtiles_t& etemp) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      Vector<T> res{verts.get_allocator(), 1};
      res.setVal(0);
      bool shouldSync = pol.shouldSync();
      pol.sync(true);
    //   elastic potential
      pol(range(eles.size()), [verts = proxy<space>({}, verts),
                               eles = proxy<space>({}, eles),
                               vtemp = proxy<space>({}, vtemp),
                               etemp = proxy<space>({},etemp),
                               res = proxy<space>(res), tag, model = model,volf = volf] 
                               ZS_LAMBDA (int ei) mutable {
        auto DmInv = eles.template pack<3, 3>("IB", ei);
        auto inds = eles.template pack<4>("inds", ei).template reinterpret_bits<int>();
        vec3 xs[4] = {vtemp.pack<3>(tag, inds[0]), vtemp.pack<3>(tag, inds[1]),
                      vtemp.pack<3>(tag, inds[2]), vtemp.pack<3>(tag, inds[3])};
        mat3 FAct{};
        {
          auto x1x0 = xs[1] - xs[0];
          auto x2x0 = xs[2] - xs[0];
          auto x3x0 = xs[3] - xs[0];
          auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                         x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
          FAct = Ds * DmInv;

          FAct = FAct * etemp.template pack<3,3>("ActInv",ei);

        //   if(ei == 0) {
        //     printf("FAct in energy : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
        //         (float)FAct(0,0),(float)FAct(0,1),(float)FAct(0,2),
        //         (float)FAct(1,0),(float)FAct(1,1),(float)FAct(1,2),
        //         (float)FAct(2,0),(float)FAct(2,1),(float)FAct(2,2));
        //   }
        }

        auto psi = model.psi(FAct);
        auto vole = eles("vol", ei);

        T gpsi = 0;
        for(int i = 0;i != 4;++i)
            gpsi += (-volf.dot(xs[i])/4); 

        atomic_add(exec_cuda, &res[0], (T)(vole * (psi + gpsi)));
      });
// Bone Driven Potential Energy
      T lambda = model.lam;
      T mu = model.mu;
      auto nmEmbedVerts = b_verts.size();
      if(b_bcws.size() != b_verts.size()){
          fmt::print("B_BCWS_SIZE = {}\t B_VERTS_SIZE = {}\n",b_bcws.size(),b_verts.size());
          throw std::runtime_error("B_BCWS SIZE AND B_VERTS SIZE NOT MATCH");
      }
      pol(range(nmEmbedVerts), [vtemp = proxy<space>({},vtemp),
          eles = proxy<space>({},eles),
          b_verts = proxy<space>({},b_verts),
          bcws = proxy<space>({},b_bcws),lambda,mu,tag,res = proxy<space>(res),bone_driven_weight = bone_driven_weight]
          ZS_LAMBDA(int vi) mutable {
              auto ei = reinterpret_bits<int>(bcws("inds",vi));
              if(ei < 0)
                  return;
              auto inds = eles.pack<4>("inds",ei).reinterpret_bits<int>();
              auto w = bcws.pack<4>("w",vi);

              auto tpos = vec3::zeros();
              for(size_t i = 0;i != 4;++i)
                  tpos += w[i] * vtemp.pack<3>(tag,inds[i]);
              auto pdiff = tpos - b_verts.pack<3>("x",vi);

              T stiffness = 2.0066 * mu + 1.0122 * lambda;
              // if(eles("vol",ei) < 0)
              //     printf("WARNING INVERT TET DETECTED<%d> %f\n",ei,(float)eles("vol",ei));
              T bpsi = (0.5 * bcws("cnorm",vi) * stiffness * bone_driven_weight * eles("vol",ei)) * pdiff.l2NormSqr();
                // bpsi = (0.5 * bcws("cnorm",vi) * lambda * bone_driven_weight) * pdiff.dot(pdiff);
                // the cnorm here should be the allocated volume of point in embeded tet 
              atomic_add(exec_cuda, &res[0], (T)bpsi);
      });
      pol.sync(shouldSync);
      return res.getVal();
    }

    template <typename Model>
    void computeGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
                                            const Model& model,
                                            const zs::SmallString tag, 
                                            dtiles_t& vtemp,
                                            dtiles_t& etemp) {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        // fmt::print("check here 0");
        TILEVEC_OPS::fill<3>(cudaPol,vtemp,"grad",zs::vec<T,3>::zeros());
        TILEVEC_OPS::fill<144>(cudaPol,etemp,"He",zs::vec<T,144>::zeros());
        cudaPol(zs::range(eles.size()), [this,
                                        vtemp = proxy<space>({}, vtemp),
                                        etemp = proxy<space>({}, etemp),
                                        bcws = proxy<space>({},b_bcws),
                                        b_verts = proxy<space>({},b_verts),
                                        verts = proxy<space>({}, verts),
                                        eles = proxy<space>({}, eles),tag, model, volf = volf] ZS_LAMBDA (int ei) mutable {
            auto DmInv = eles.template pack<3, 3>("IB", ei);
            auto dFdX = dFdXMatrix(DmInv);
            auto inds = eles.template pack<4>("inds", ei).template reinterpret_bits<int>();
            vec3 xs[4] = {vtemp.pack<3>(tag, inds[0]), vtemp.pack<3>(tag, inds[1]),
                            vtemp.pack<3>(tag, inds[2]), vtemp.pack<3>(tag, inds[3])};
            mat3 FAct{};
            {
                auto x1x0 = xs[1] - xs[0];
                auto x2x0 = xs[2] - xs[0];
                auto x3x0 = xs[3] - xs[0];
                auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                            x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                FAct = Ds * DmInv;

                FAct = FAct * etemp.template pack<3,3>("ActInv",ei);

                // if(ei == 0) {
                //     printf("FAct in gH : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
                //         (float)FAct(0,0),(float)FAct(0,1),(float)FAct(0,2),
                //         (float)FAct(1,0),(float)FAct(1,1),(float)FAct(1,2),
                //         (float)FAct(2,0),(float)FAct(2,1),(float)FAct(2,2));
                    
                //     auto Act =  etemp.template pack<3,3>("ActInv",ei);

                //     printf("Act in gH : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
                //         (float)Act(0,0),(float)Act(0,1),(float)Act(0,2),
                //         (float)Act(1,0),(float)Act(1,1),(float)Act(1,2),
                //         (float)Act(2,0),(float)Act(2,1),(float)Act(2,2));                        
                // }

                // auto ActInv_check = etemp.template pack<3,3>("ActInv",ei);
                // for(int i = 0;i != 3;++i)
                //     ActInv_check(i,i) -= 1.0;
                // if(ActInv_check.norm() > 1){
                //     auto ActInv = etemp.template pack<3,3>("ActInv",ei);
                //     printf("wierd ActInv<%d> in gH : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",ei,
                //         (float)ActInv(0,0),(float)ActInv(0,1),(float)ActInv(0,2),
                //         (float)ActInv(1,0),(float)ActInv(1,1),(float)ActInv(1,2),
                //         (float)ActInv(2,0),(float)ActInv(2,1),(float)ActInv(2,2));  
                // }

            }

            auto dFActdF = dFAdF(etemp.template pack<3,3>("ActInv",ei));

            auto P = model.first_piola(FAct);
            auto vole = eles("vol", ei);
            auto vecP = flatten(P);
            vecP = dFActdF.transpose() * vecP;
            auto dFdXT = dFdX.transpose();
            auto vf = -vole * (dFdXT * vecP);

            auto mg = volf * vole / 4;
            for (int i = 0; i != 4; ++i) {
                auto vi = inds[i];
                for (int d = 0; d != 3; ++d)
                    atomic_add(exec_cuda, &vtemp("grad", d, vi), vf(i * 3 + d) + mg(d));
            }

            auto Hq = model.first_piola_derivative(FAct, true_c);
            auto dFdAct_dFdX = dFActdF * dFdX; 
            // dFdAct_dFdX = dFdX; 
            auto H = dFdAct_dFdX.transpose() * Hq * dFdAct_dFdX * vole;

            etemp.tuple<12 * 12>("He", ei) = H;


            // auto Hn = H.norm();
            // if(isnan(Hn)){
            //     auto Hqn = Hq.norm();
            //     auto dFdXn = dFdAct_dFdX.norm();
            //     printf("elm<%d>_Hn : %f %f %f\n",ei,(float)Hn,(float)dFdXn,(float)Hqn);
            //     printf("FAct<%d> in gH : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",ei,
            //         (float)FAct(0,0),(float)FAct(0,1),(float)FAct(0,2),
            //         (float)FAct(1,0),(float)FAct(1,1),(float)FAct(1,2),
            //         (float)FAct(2,0),(float)FAct(2,1),(float)FAct(2,2));

            //     auto Act = etemp.template pack<3,3>("ActInv",ei);
            //     printf("Act<%d> in gH : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",ei,
            //         (float)Act(0,0),(float)Act(0,1),(float)Act(0,2),
            //         (float)Act(1,0),(float)Act(1,1),(float)Act(1,2),
            //         (float)Act(2,0),(float)Act(2,1),(float)Act(2,2));                
            // }

        });


        // fmt::print("check here 1\n");
        T lambda = model.lam;
        T mu = model.mu;
        if(b_bcws.size() != b_verts.size()){
            fmt::print("B_BCWS_SIZE = {}\t B_VERTS_SIZE = {}\n",b_bcws.size(),b_verts.size());
            throw std::runtime_error("B_BCWS SIZE AND B_VERTS SIZE NOT MATCH");
        }

        // fmt::print("check here 2\n");

        auto nmEmbedVerts = b_verts.size();
        cudaPol(zs::range(nmEmbedVerts),
            [bcws = proxy<space>({},b_bcws),b_verts = proxy<space>({},b_verts),vtemp = proxy<space>({},vtemp),etemp = proxy<space>({},etemp),
                eles = proxy<space>({},eles),lambda,mu,tag,bone_driven_weight = bone_driven_weight] ZS_LAMBDA(int vi) mutable {
                    auto ei = reinterpret_bits<int>(bcws("inds",vi));
                    if(ei < 0)
                        return;
                    auto inds = eles.pack<4>("inds",ei).reinterpret_bits<int>();
                    auto w = bcws.pack<4>("w",vi);
                    auto tpos = vec3::zeros();
                    for(size_t i = 0;i != 4;++i)
                        tpos += w[i] * vtemp.pack<3>(tag,inds[i]);
                    auto pdiff = tpos - b_verts.pack<3>("x",vi);

                    T stiffness = 2.0066 * mu + 1.0122 * lambda;

                    for(size_t i = 0;i != 4;++i){
                        auto tmp = pdiff * (-stiffness * bcws("cnorm",vi) * bone_driven_weight * w[i] * eles("vol",ei)); 
                        // tmp = pdiff * (-lambda * bcws("cnorm",vi) * bone_driven_weight * w[i]);
                        for(size_t d = 0;d != 3;++d)
                            atomic_add(exec_cuda,&vtemp("grad",d,inds[i]),(T)tmp[d]);
                    }
                    for(int i = 0;i != 4;++i)
                        for(int j = 0;j != 4;++j){
                            T alpha = stiffness * bone_driven_weight * w[i] * w[j] * bcws("cnorm",vi) * eles("vol",ei);
                            // alpha = lambda * bone_driven_weight * w[i] * w[j] * bcws("cnorm",vi);
                            // if(ei == 11221)
                            //   if(i == 3 && j == 3)
                            //     printf("alpha : %f\n",alpha);
                            for(int d = 0;d != 3;++d){
                                // etemp("He",(i * 3 + d) * 12 + j * 3 + d,ei) += alpha;
                                // if(isnan(alpha)){
                                //     printf("nan alpha<%d,%d,%d> %f %f %f %f %f\n",vi,i,j,(float)lambda,(float)bone_driven_weight,(float)w[i],(float)w[j],(float)bcws("cnorm",vi));
                                // }
                                atomic_add(exec_cuda,&etemp("He",(i * 3 + d) * 12 + j * 3 + d,ei),alpha);
                            }
                        }

        });

    }

    FEMQuasiStaticSystem(const tiles_t &verts, const tiles_t &eles, const tiles_t &b_bcws, const tiles_t& b_verts,T bone_driven_weight,vec3 volf)
        : verts{verts}, eles{eles}, b_bcws{b_bcws}, b_verts{b_verts}, bone_driven_weight{bone_driven_weight},volf{volf}{}

    const tiles_t &verts;
    const tiles_t &eles;
    const tiles_t &b_bcws;  // the barycentric interpolation of embeded bones 
    const tiles_t &b_verts; // the position of embeded bones

    T bone_driven_weight;
    vec3 volf;
  };

  void apply() override {
    using namespace zs;
    auto zstets = get_input<ZenoParticles>("ZSParticles");
    auto gravity = zeno::vec<3,T>(0);
    if(has_input("gravity"))
      gravity = get_input<zeno::NumericObject>("gravity")->get<zeno::vec<3,T>>();
    auto armijo = get_param<float>("armijo");
    auto curvature = get_param<float>("wolfe");
    auto cg_res = get_param<float>("cg_res");
    auto btl_res = get_param<float>("btl_res");
    auto models = zstets->getModel();
    auto& verts = zstets->getParticles();
    auto& eles = zstets->getQuadraturePoints();
    auto zsbones = get_input<ZenoParticles>("driven_bones");
    auto tag = get_param<std::string>("driven_tag");
    auto muscle_id_tag = get_param<std::string>("muscle_id_tag");
    auto bone_driven_weight = get_param<float>("bone_driven_weight");
    auto newton_res = get_param<float>("newton_res");

    auto volf = vec3::from_array(gravity * models.density);

    // auto nm_acts = get_input<zeno::ListObject>("Acts")->arr.size();
    // fmt::print("number of activations : {}\n",nm_acts);

    std::vector<float> act_;    
    std::size_t nm_acts = 0;
    // auto nm_acts_ = zstets->get().get("NM_MUSCLES");
    // std::cout << "nm_acts_ : " << std::endl;

    if(has_input("Acts")) {
      act_ = get_input<zeno::ListObject>("Acts")->getLiterial<float>();
      nm_acts = act_.size();
    }
    // auto act_ = get_input<zeno::ListObject>("Acts")->getLiterial<float>();
    // initialize on host qs[i] = qs_[i]->get<zeno::vec4f>();

    constexpr auto host_space = zs::execspace_e::openmp;
    auto ompExec = zs::omp_exec();
    auto act_buffer = dtiles_t{{{"act",1}},nm_acts,zs::memsrc_e::host};
    ompExec(range(act_buffer.size()),
        [act_buffer = proxy<host_space>({},act_buffer),act_] (int i) mutable{
            act_buffer("act",i) = act_[i];
            // fmt::print("act<{}> : {}\n",i,act_buffer("act",i));
    });
    act_buffer = act_buffer.clone({zs::memsrc_e::device, 0});

    static dtiles_t vtemp{verts.get_allocator(),
                          {{"grad", 3},
                           {"P", 9},
                           {"bou_tag",1},
                           {"dir", 3},
                           {"xn", 3},
                           {"xn0", 3},
                           {"temp", 3},
                           {"r", 3},
                           {"p", 3},
                           {"q", 3}},
                          verts.size()};
    static dtiles_t etemp{eles.get_allocator(), {{"He", 12 * 12},{"inds",4},{"ActInv",3*3},{"muscle_ID",1},{"fiber",3}}, eles.size()};
    vtemp.resize(verts.size());
    etemp.resize(eles.size());

    FEMQuasiStaticSystem A{verts,eles,(*zstets)[tag],zsbones->getParticles(),bone_driven_weight,volf};

    constexpr auto space = execspace_e::cuda;
    auto cudaPol = cuda_exec().sync(false);

    TILEVEC_OPS::copy<4>(cudaPol,eles,"inds",etemp,"inds");


    if(!eles.hasProperty("fiber")){
        // fmt::print("The input flesh have no fiber orientations, use the default setting\n");
        TILEVEC_OPS::fill<3>(cudaPol,etemp,"fiber",{1.,0.,0.});
        // throw std::runtime_error("The input flesh should have fiber orientations");

    }else {
      if(eles.getChannelSize("fiber") != 3){
          fmt::print("The input fiber  has wrong channel size\n");
          throw std::runtime_error("The input fiber has wrong channel size");
      }
      TILEVEC_OPS::copy<3>(cudaPol,eles,"fiber",etemp,"fiber");
    }
    if(!eles.hasProperty(muscle_id_tag)) {
      // if((!eles.hasProperty(muscle_id_tag)) || (eles.getChannelSize(muscle_id_tag) != 1)){
      //     fmt::print("the quadrature has no muscle id tag : {} {}\n",muscle_id_tag,eles.getChannelSize(muscle_id_tag));
      //     throw std::runtime_error("the quadrature has no muscle id tag");
      // }
      // fmt::print("The input flesh have no mosucle_id specified, use the default setting");
      TILEVEC_OPS::fill(cudaPol,etemp,"muscle_ID",-1);
    }else {
      TILEVEC_OPS::copy(cudaPol,eles,muscle_id_tag,etemp,"muscle_ID");
    }

    // apply muscle activation
    cudaPol(range(etemp.size()),
        [etemp = proxy<space>({},etemp),act_buffer = proxy<space>({},act_buffer),muscle_id_tag = SmallString(muscle_id_tag),nm_acts] ZS_LAMBDA(int ei) mutable {
            // auto act = eles.template pack<3>("act",ei);
            auto fiber = etemp.template pack<3>("fiber",ei);
              
            vec3 act{0};

            auto nfiber = fiber.norm();
            auto ID = etemp("muscle_ID",ei);
            if(nfiber < 0.5 || ID < -1e-6 || nm_acts == 0){ // if there is no local fiber orientaion, use the default act and fiber
                fiber = vec3{1.0,0.0,0.0};
                act = vec3{1.0,1.0,1.0};
            }else{
                // a test
                int id = (int)ID;
                float a = 1. - act_buffer("act",id);
                act = vec3{1,zs::sqrt(1./a),zs::sqrt(1./a)};
                fiber /= nfiber;// in case there is some floating-point error

                // printf("use act[%d] : %f\n",id,(float)a);
            }

            vec3 dir[3];
            dir[0] = fiber;
            auto tmp = vec3{1.0,0.0,0.0};
            dir[1] = dir[0].cross(tmp);
            if(dir[1].length() < 1e-3) {
                tmp = vec3{0.0,1.0,0.0};
                dir[1] = dir[0].cross(tmp);
            }

            dir[1] = dir[1] / dir[1].length();
            dir[2] = dir[0].cross(dir[1]);

            auto R = mat3{};
            for(int i = 0;i < 3;++i)
                for(int j = 0;j < 3;++j)
                    R(i,j) = dir[j][i];

            auto Act = mat3::zeros();
            Act(0,0) = act[0];
            Act(1,1) = act[1];
            Act(2,2) = act[2];

            Act = R * Act * R.transpose();

            // if(ei == 0) {
            //     printf("Act : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
            //         (float)Act(0,0),(float)Act(0,1),(float)Act(0,2),
            //         (float)Act(1,0),(float)Act(1,1),(float)Act(1,2),
            //         (float)Act(2,0),(float)Act(2,1),(float)Act(2,2));                        
            // }


            etemp.template tuple<9>("ActInv",ei) = zs::inverse(Act);

            // if(ei == 0) {
            //     Act = etemp.template pack<3,3>("ActInv",ei);
            //     printf("Act : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
            //         (float)Act(0,0),(float)Act(0,1),(float)Act(0,2),
            //         (float)Act(1,0),(float)Act(1,1),(float)Act(1,2),
            //         (float)Act(2,0),(float)Act(2,1),(float)Act(2,2));  

            //     // auto dFActdF = dFAdF(eles.template pack<3,3>("ActInv",ei));
            //     // printf("dFActdF : \n%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\")

            // }

    });

    // setup initial guess
    TILEVEC_OPS::copy<3>(cudaPol,verts,verts.hasProperty("init_x") ? "init_x" : "x",vtemp,"xn");    
    TILEVEC_OPS::fill<1>(cudaPol,vtemp,"bou_tag",zs::vec<T,1>::zeros());

    for(int newtonIter = 0;newtonIter != 1000;++newtonIter){
      match([&](auto &elasticModel) {
        A.computeGradientAndHessian(cudaPol, elasticModel,"xn",vtemp,etemp);
      })(models.getElasticModel());

    // auto Hn = TILEVEC_OPS::dot<144>(cudaPol,etemp,"He","He");
    // fmt::print("Hn : {}\n",(float)Hn);    

    // break;

    //  Prepare Preconditioning
      PCG::prepare_block_diagonal_preconditioner<4,3>(cudaPol,"He",etemp,"P",vtemp);

      // if the grad is too small, return the result
      // Solve equation using PCG
      TILEVEC_OPS::fill<3>(cudaPol,vtemp,"dir",zs::vec<T,3>::zeros());
      PCG::pcg_with_fixed_sol_solve<3,4>(cudaPol,vtemp,etemp,"dir","bou_tag","grad","P","inds","He",cg_res,1000,50);
      PCG::project<3>(cudaPol,vtemp,"dir","bou_tag");
      PCG::project<3>(cudaPol,vtemp,"grad","bou_tag");
      T res = TILEVEC_OPS::inf_norm<3>(cudaPol, vtemp, "dir");// this norm is independent of descriterization

      if (res < newton_res) {
        fmt::print("\t# newton optimizer reach desired resolution in {} iters with residual {}\n",
                   newtonIter, res);
        break;
      }
      T dg = TILEVEC_OPS::dot<3>(cudaPol,vtemp,"grad","dir");
      if(fabs(dg) < btl_res){
        // fmt::print("\t# newton optimizer reach stagnation point in {} iters with residual {}\n",newtonIter, res);
        break;
      }
      if(dg < 0){
          T gradn = std::sqrt(TILEVEC_OPS::dot<3>(cudaPol,vtemp,"grad","grad"));
          T dirn = std::sqrt(TILEVEC_OPS::dot<3>(cudaPol,vtemp,"dir","dir"));
          fmt::print("invalid dg = {} grad = {} dir = {}\n",dg,gradn,dirn);
          throw std::runtime_error("INVALID DESCENT DIRECTION");
      }
      T alpha = 1.;
      TILEVEC_OPS::copy<3>(cudaPol,vtemp,"xn",vtemp,"xn0");
      T E0;
      match([&](auto &elasticModel) {
        E0 = A.energy(cudaPol, elasticModel, "xn0",vtemp,etemp);
      })(models.getElasticModel());

      dg = -dg;

      T E{E0};
    //   Backtracking Linesearch
      int max_line_search = 10;
      int line_search = 0;
      std::vector<T> armijo_buffer(max_line_search);
      do {
        TILEVEC_OPS::add<3>(cudaPol,vtemp,"xn0",(T)1.0,"dir",alpha,"xn");
        match([&](auto &elasticModel) {
          E = A.energy(cudaPol, elasticModel, "xn",vtemp,etemp);
        })(models.getElasticModel());
        armijo_buffer[line_search] = (E - E0)/alpha;
        // test Armojo condition
        if (((double)E - (double)E0) < (double)armijo * (double)dg * (double)alpha)
          break;
        alpha /= 2;
        ++line_search;
      } while (line_search < max_line_search);
      if(line_search == max_line_search){
          fmt::print("LINE_SEARCH_EXCEED: %f\n",dg);
          // for(size_t i = 0;i != max_line_search;++i)
          //   fmt::print("AB[{}]\t = {} dg = {}\n",i,armijo_buffer[i],dg);
      }

      cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),
                                        alpha] __device__(int i) mutable {
        vtemp.tuple<3>("xn", i) =
            vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
      });
    
    }

    cudaPol(zs::range(verts.size()),
            [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts)] __device__(int vi) mutable {
              auto newX = vtemp.pack<3>("xn", vi);
              verts.tuple<3>("x", vi) = newX;
            });

    cudaPol.syncCtx();

    // write back muscle activation
    auto output_act = get_param<int>("output_act");
    if(output_act) {
      auto ActTag = get_param<std::string>("actTag");
      if(!eles.hasProperty(ActTag))
        eles.append_channels(cudaPol,{{ActTag,1}});
      TILEVEC_OPS::fill(cudaPol,eles,ActTag,0);
      if(nm_acts > 0) {
        cudaPol(zs::range(eles.size()),
          [eles = proxy<space>({},eles),muscle_id_tag = zs::SmallString{muscle_id_tag},
              act_buffer = proxy<space>({},act_buffer),ActTag = zs::SmallString{ActTag}] __device__(int ei) mutable {
            auto ID = eles(muscle_id_tag,ei);
            int id = (int)ID;
            eles(ActTag,ei) = id > -1 ? act_buffer("act",id) : 0;
            // eles(ActTag,ei) = id > -1 ? 0.5 : 0;
        });
      }
    }

    cudaPol.syncCtx();

    set_output("ZSParticles", zstets);
  }
};

ZENDEFNODE(FleshQuasiStaticStepping, {{"ZSParticles","driven_bones","gravity","Acts"},
                                  {"ZSParticles"},
                                  {{"float","armijo","0.1"},{"float","wolfe","0.9"},
                                    {"float","cg_res","0.1"},{"float","btl_res","0.0001"},{"float","newton_res","0.001"},
                                    {"string","driven_tag","bone_bw"},{"float","bone_driven_weight","0.0"},
                                    {"string","muscle_id_tag","ms_id_tag"},{"int","output_act","0"},{"string","actTag","Act"}  
                                  },
                                  {"FEM"}});

}