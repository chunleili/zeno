#include "hip/hip_runtime.h"
#include "Solver.cuh"
#include "Utils.hpp"
#include <zeno/types/NumericObject.h>

namespace zeno {

IPCSystem::PrimitiveHandle::PrimitiveHandle(std::shared_ptr<tiles_t> elesPtr_, ZenoParticles::category_e category)
    : zsprimPtr{}, modelsPtr{}, vertsPtr{}, elesPtr{elesPtr_},
      etemp{elesPtr_->get_allocator(), {{"He", 6 * 6}}, elesPtr_->size()}, surfTrisPtr{}, surfEdgesPtr{},
      surfVertsPtr{}, svtemp{}, vOffset{0}, sfOffset{0}, seOffset{0}, svOffset{0}, category{category} {
    ;
}
IPCSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, std::size_t &vOffset, std::size_t &sfOffset,
                                            std::size_t &seOffset, std::size_t &svOffset, zs::wrapv<2>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}},
      vertsPtr{&zsprim.getParticles<true>(), [](void *) {}}, elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}},
      etemp{zsprim.getQuadraturePoints().get_allocator(), {{"He", 6 * 6}}, zsprim.numElements()},
      surfTrisPtr{&zsprim.getQuadraturePoints(), [](void *) {}},  // this is fake!
      surfEdgesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, // all elements are surface edges
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::curve)
        throw std::runtime_error("dimension of 2 but is not curve");
    vOffset += getVerts().size();
    // sfOffset += 0; // no surface triangles
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
IPCSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, std::size_t &vOffset, std::size_t &sfOffset,
                                            std::size_t &seOffset, std::size_t &svOffset, zs::wrapv<3>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}},
      vertsPtr{&zsprim.getParticles<true>(), [](void *) {}}, elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}},
      etemp{zsprim.getQuadraturePoints().get_allocator(), {{"He", 9 * 9}}, zsprim.numElements()},
      surfTrisPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, surfEdgesPtr{&zsprim[ZenoParticles::s_surfEdgeTag],
                                                                              [](void *) {}},
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::surface)
        throw std::runtime_error("dimension of 3 but is not surface");
    vOffset += getVerts().size();
    sfOffset += getSurfTris().size();
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
IPCSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, std::size_t &vOffset, std::size_t &sfOffset,
                                            std::size_t &seOffset, std::size_t &svOffset, zs::wrapv<4>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}},
      vertsPtr{&zsprim.getParticles<true>(), [](void *) {}}, elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}},
      etemp{zsprim.getQuadraturePoints().get_allocator(), {{"He", 12 * 12}}, zsprim.numElements()},
      surfTrisPtr{&zsprim[ZenoParticles::s_surfTriTag], [](void *) {}},
      surfEdgesPtr{&zsprim[ZenoParticles::s_surfEdgeTag], [](void *) {}},
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::tet)
        throw std::runtime_error("dimension of 4 but is not tetrahedra");
    vOffset += getVerts().size();
    sfOffset += getSurfTris().size();
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
typename IPCSystem::T IPCSystem::PrimitiveHandle::averageNodalMass(zs::CudaExecutionPolicy &pol) const {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    if (zsprimPtr->hasMeta(s_meanMassTag))
        return zsprimPtr->readMeta(s_meanMassTag, zs::wrapt<T>{});
    auto &verts = getVerts();
    Vector<T> masses{verts.get_allocator(), verts.size()};
    pol(Collapse{verts.size()}, [verts = proxy<space>({}, verts), masses = proxy<space>(masses)] ZS_LAMBDA(
                                    int vi) mutable { masses[vi] = verts("m", vi); });
    auto tmp = reduce(pol, masses) / masses.size();
    zsprimPtr->setMeta(s_meanMassTag, tmp);
    return tmp;
}
typename IPCSystem::T IPCSystem::PrimitiveHandle::averageSurfEdgeLength(zs::CudaExecutionPolicy &pol) const {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    if (zsprimPtr->hasMeta(s_meanSurfEdgeLengthTag))
        return zsprimPtr->readMeta(s_meanSurfEdgeLengthTag, zs::wrapt<T>{});
    auto &verts = getVerts();
    auto &edges = getSurfEdges();
    Vector<T> edgeLengths{edges.get_allocator(), edges.size()};
    pol(Collapse{edges.size()}, [edges = proxy<space>({}, edges), verts = proxy<space>({}, verts),
                                 edgeLengths = proxy<space>(edgeLengths)] ZS_LAMBDA(int ei) mutable {
        auto inds = edges.pack(dim_c<2>, "inds", ei).template reinterpret_bits<int>();
        edgeLengths[ei] = (verts.pack<3>("x0", inds[0]) - verts.pack<3>("x0", inds[1])).norm();
    });
    auto tmp = reduce(pol, edgeLengths) / edges.size();
    zsprimPtr->setMeta(s_meanSurfEdgeLengthTag, tmp);
    return tmp;
}
typename IPCSystem::T IPCSystem::PrimitiveHandle::averageSurfArea(zs::CudaExecutionPolicy &pol) const {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    if (zsprimPtr->category == ZenoParticles::curve)
        return (T)0;
    if (zsprimPtr->hasMeta(s_meanSurfAreaTag))
        return zsprimPtr->readMeta(s_meanSurfAreaTag, zs::wrapt<T>{});
    auto &verts = getVerts();
    auto &tris = getSurfTris();
    Vector<T> surfAreas{tris.get_allocator(), tris.size()};
    pol(Collapse{surfAreas.size()}, [tris = proxy<space>({}, tris), verts = proxy<space>({}, verts),
                                     surfAreas = proxy<space>(surfAreas)] ZS_LAMBDA(int ei) mutable {
        auto inds = tris.pack(dim_c<3>, "inds", ei).template reinterpret_bits<int>();
        surfAreas[ei] = (verts.pack<3>("x0", inds[1]) - verts.pack<3>("x0", inds[0]))
                            .cross(verts.pack<3>("x0", inds[2]) - verts.pack<3>("x0", inds[0]))
                            .norm() /
                        2;
    });
    auto tmp = reduce(pol, surfAreas) / tris.size();
    zsprimPtr->setMeta(s_meanSurfAreaTag, tmp);
    return tmp;
}

/// IPCSystem
typename IPCSystem::T IPCSystem::averageNodalMass(zs::CudaExecutionPolicy &pol) {
    using T = typename IPCSystem::T;
    T sumNodalMass = 0;
    std::size_t sumNodes = 0;
    for (auto &&primHandle : prims) {
        if (primHandle.isBoundary())
            continue;
        auto numNodes = primHandle.getVerts().size();
        sumNodes += numNodes;
        sumNodalMass += primHandle.averageNodalMass(pol) * numNodes;
    }
    if (sumNodes)
        return sumNodalMass / sumNodes;
    else
        return 0;
}
typename IPCSystem::T IPCSystem::averageSurfEdgeLength(zs::CudaExecutionPolicy &pol) {
    using T = typename IPCSystem::T;
    T sumSurfEdgeLengths = 0;
    std::size_t sumSE = 0;
    for (auto &&primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        auto numSE = primHandle.getSurfEdges().size();
        sumSE += numSE;
        sumSurfEdgeLengths += primHandle.averageSurfEdgeLength(pol) * numSE;
    }
    if (sumSE)
        return sumSurfEdgeLengths / sumSE;
    else
        return 0;
}
typename IPCSystem::T IPCSystem::averageSurfArea(zs::CudaExecutionPolicy &pol) {
    using T = typename IPCSystem::T;
    T sumSurfArea = 0;
    std::size_t sumSF = 0;
    for (auto &&primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        if (primHandle.category == ZenoParticles::curve)
            continue;
        auto numSF = primHandle.getSurfTris().size();
        sumSF += numSF;
        sumSurfArea += primHandle.averageSurfArea(pol) * numSF;
    }
    if (sumSF)
        return sumSurfArea / sumSF;
    else
        return 0;
}
void IPCSystem::updateWholeBoundingBoxSize(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    bv_t bv = seBvh.getTotalBox(pol);
    if (coVerts)
        if (coVerts->size()) {
            auto bouBv = bouSeBvh.getTotalBox(pol);
            merge(bv, bouBv._min);
            merge(bv, bouBv._max);
        }
    boxDiagSize2 = (bv._max - bv._min).l2NormSqr();
}
void IPCSystem::initKappa(zs::CudaExecutionPolicy &pol) {
    // should be called after dHat set
    if (!enableContact)
        return;
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
        vtemp.tuple<3>("p", i) = vec3::zeros();
        vtemp.tuple<3>("q", i) = vec3::zeros();
    });
    // inertial + elasticity
    computeInertialPotentialGradient(pol, "p");
    computeElasticGradientAndHessian(pol, "p", false);
    // contacts
    findCollisionConstraints(pol, dHat, xi);
    auto prevKappa = kappa;
    kappa = 1;
    computeBarrierGradientAndHessian(pol, "q", false);
    // computeBoundaryBarrierGradientAndHessian(pol, "q", false);
    kappa = prevKappa;
    auto gsum = dot(pol, vtemp, "p", "q");
    auto gsnorm = dot(pol, vtemp, "q", "q");
    if (gsnorm < limits<T>::epsilon() * 10)
        kappaMin = 0;
    else
        kappaMin = -gsum / gsnorm;
    // zeno::log_info("kappaMin: {}, gsum: {}, gsnorm: {}\n", kappaMin, gsum, gsnorm);
}

void IPCSystem::initialize(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    stInds = tiles_t{vtemp.get_allocator(), {{"inds", 3}}, sfOffset};
    seInds = tiles_t{vtemp.get_allocator(), {{"inds", 2}}, seOffset};
    svInds = tiles_t{vtemp.get_allocator(), {{"inds", 1}}, svOffset};
    exclSes = Vector<u8>{vtemp.get_allocator(), seOffset};
    exclSts = Vector<u8>{vtemp.get_allocator(), sfOffset};
    std::size_t nBouSes = 0, nBouSts = 0;
    if (coEdges) {
        nBouSes = coEdges->size();
        nBouSts = coEles->size();
    }
    exclBouSes = Vector<u8>{vtemp.get_allocator(), nBouSes};
    exclBouSts = Vector<u8>{vtemp.get_allocator(), nBouSts};

    auto deduce_node_cnt = [](std::size_t numLeaves) {
        if (numLeaves <= 2)
            return numLeaves;
        return numLeaves * 2 - 1;
    };
    selfStFront = bvfront_t{(int)deduce_node_cnt(stInds.size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
    selfSeFront = bvfront_t{(int)deduce_node_cnt(seInds.size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
    if (coVerts) {
        boundaryStFront =
            bvfront_t{(int)deduce_node_cnt(coEles->size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
        boundarySeFront =
            bvfront_t{(int)deduce_node_cnt(coEdges->size()), (int)estNumCps, zs::memsrc_e::um, vtemp.devid()};
    }

    meanEdgeLength = averageSurfEdgeLength(pol);
    meanSurfaceArea = averageSurfArea(pol);
    avgNodeMass = averageNodalMass(pol);
    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        auto &verts = primHandle.getVerts();
        // record surface (tri) indices
        if (primHandle.category != ZenoParticles::category_e::curve) {
            auto &tris = primHandle.getSurfTris();
            pol(Collapse(tris.size()),
                [stInds = proxy<space>({}, stInds), tris = proxy<space>({}, tris), voffset = primHandle.vOffset,
                 sfoffset = primHandle.sfOffset] __device__(int i) mutable {
                    stInds.template tuple<3>("inds", sfoffset + i) =
                        (tris.pack(dim_c<3>, "inds", i).template reinterpret_bits<int>() + (int)voffset)
                            .template reinterpret_bits<float>();
                });
        }
        auto &edges = primHandle.getSurfEdges();
        pol(Collapse(edges.size()),
            [seInds = proxy<space>({}, seInds), edges = proxy<space>({}, edges), voffset = primHandle.vOffset,
             seoffset = primHandle.seOffset] __device__(int i) mutable {
                seInds.template tuple<2>("inds", seoffset + i) =
                    (edges.pack(dim_c<2>, "inds", i).template reinterpret_bits<int>() + (int)voffset)
                        .template reinterpret_bits<float>();
            });
        auto &points = primHandle.getSurfVerts();
        pol(Collapse(points.size()),
            [svInds = proxy<space>({}, svInds), points = proxy<space>({}, points), voffset = primHandle.vOffset,
             svoffset = primHandle.svOffset] __device__(int i) mutable {
                svInds("inds", svoffset + i) =
                    reinterpret_bits<float>(reinterpret_bits<int>(points("inds", i)) + (int)voffset);
            });
    }
    // initialize vtemp & spatial accel
    reinitialize(pol, dt);
}

IPCSystem::IPCSystem(std::vector<ZenoParticles *> zsprims, const typename IPCSystem::dtiles_t *coVerts,
                     const typename IPCSystem::tiles_t *coLowResVerts, const typename IPCSystem::tiles_t *coEdges,
                     const tiles_t *coEles, T dt, std::size_t estNumCps, bool withGround, bool withContact,
                     bool withMollification, T augLagCoeff, T pnRel, T cgRel, int PNCap, int CGCap, int CCDCap,
                     T kappa0, T fricMu, T dHat_, T epsv_, zeno::vec3f gn, T gravity)
    : coVerts{coVerts}, coLowResVerts{coLowResVerts}, coEdges{coEdges}, coEles{coEles},
      PP{estNumCps, zs::memsrc_e::um, 0}, nPP{zsprims[0]->getParticles<true>().get_allocator(), 1},
      tempPP{{{"H", 36}}, estNumCps, zs::memsrc_e::um, 0}, PE{estNumCps, zs::memsrc_e::um, 0},
      nPE{zsprims[0]->getParticles<true>().get_allocator(), 1}, tempPE{{{"H", 81}}, estNumCps, zs::memsrc_e::um, 0},
      PT{estNumCps, zs::memsrc_e::um, 0}, nPT{zsprims[0]->getParticles<true>().get_allocator(), 1},
      tempPT{{{"H", 144}}, estNumCps, zs::memsrc_e::um, 0}, EE{estNumCps, zs::memsrc_e::um, 0},
      nEE{zsprims[0]->getParticles<true>().get_allocator(), 1}, tempEE{{{"H", 144}}, estNumCps, zs::memsrc_e::um, 0},
      // mollify
      PPM{estNumCps, zs::memsrc_e::um, 0}, nPPM{zsprims[0]->getParticles<true>().get_allocator(), 1},
      tempPPM{{{"H", 144}}, estNumCps, zs::memsrc_e::um, 0}, PEM{estNumCps, zs::memsrc_e::um, 0},
      nPEM{zsprims[0]->getParticles<true>().get_allocator(), 1}, tempPEM{{{"H", 144}}, estNumCps, zs::memsrc_e::um, 0},
      EEM{estNumCps, zs::memsrc_e::um, 0}, nEEM{zsprims[0]->getParticles<true>().get_allocator(), 1},
      tempEEM{{{"H", 144}}, estNumCps, zs::memsrc_e::um, 0},
      // friction
      FPP{estNumCps, zs::memsrc_e::um, 0}, nFPP{zsprims[0]->getParticles<true>().get_allocator(), 1},
      fricPP{{{"H", 36}, {"basis", 6}, {"fn", 1}}, estNumCps, zs::memsrc_e::um, 0}, FPE{estNumCps, zs::memsrc_e::um, 0},
      nFPE{zsprims[0]->getParticles<true>().get_allocator(), 1},
      fricPE{{{"H", 81}, {"basis", 6}, {"fn", 1}, {"yita", 1}}, estNumCps, zs::memsrc_e::um, 0},
      FPT{estNumCps, zs::memsrc_e::um, 0}, nFPT{zsprims[0]->getParticles<true>().get_allocator(), 1},
      fricPT{{{"H", 144}, {"basis", 6}, {"fn", 1}, {"beta", 2}}, estNumCps, zs::memsrc_e::um, 0},
      FEE{estNumCps, zs::memsrc_e::um, 0}, nFEE{zsprims[0]->getParticles<true>().get_allocator(), 1},
      fricEE{{{"H", 144}, {"basis", 6}, {"fn", 1}, {"gamma", 2}}, estNumCps, zs::memsrc_e::um, 0},
      //
      temp{estNumCps, zs::memsrc_e::um, zsprims[0]->getParticles<true>().devid()}, csPT{estNumCps, zs::memsrc_e::um, 0},
      csEE{estNumCps, zs::memsrc_e::um, 0}, ncsPT{zsprims[0]->getParticles<true>().get_allocator(), 1},
      ncsEE{zsprims[0]->getParticles<true>().get_allocator(), 1},
      //
      dt{dt}, framedt{dt}, curRatio{0}, estNumCps{estNumCps}, enableGround{withGround}, enableContact{withContact},
      enableMollification{withMollification}, s_groundNormal{gn[0], gn[1], gn[2]},
      augLagCoeff{augLagCoeff}, pnRel{pnRel}, cgRel{cgRel}, PNCap{PNCap}, CGCap{CGCap}, CCDCap{CCDCap}, kappa{kappa0},
      kappa0{kappa0}, kappaMin{0}, kappaMax{kappa0}, fricMu{fricMu}, dHat{dHat_}, epsv{epsv_}, extForce{0, gravity, 0} {
    coOffset = sfOffset = seOffset = svOffset = 0;
    for (auto primPtr : zsprims) {
        if (primPtr->category == ZenoParticles::category_e::curve) {
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<2>{});
        } else if (primPtr->category == ZenoParticles::category_e::surface)
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<3>{});
        else if (primPtr->category == ZenoParticles::category_e::tet)
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<4>{});
    }
    numDofs = coOffset;
    if (coVerts)
        numDofs += coVerts->size();
    numBouDofs = numDofs - coOffset;

    fmt::print("num total obj <verts, bouVerts, surfV, surfE, surfT>: {}, {}, {}, {}, {}\n", coOffset, numBouDofs,
               svOffset, seOffset, sfOffset);

    vtemp = dtiles_t{zsprims[0]->getParticles<true>().get_allocator(),
                     {{"grad", 3},
                      {"P", 9},
                      // dirichlet boundary condition type; 0: NOT, 1: ZERO, 2: NONZERO
                      {"BCorder", 1},
                      {"BCbasis", 9},
                      {"BCtarget", 3},
                      {"BCfixed", 1},
                      {"BCsoft", 1}, // mark if this dof is a soft boundary vert or not
                      {"ws", 1},     // also as constraint jacobian
                      {"cons", 3},
                      {"lambda", 3},

                      {"dir", 3},
                      {"xn", 3},
                      {"vn", 3},
                      {"x0", 3},  // initial positions
                      {"xn0", 3}, // for line search
                      {"xtilde", 3},
                      {"xhat", 3}, // initial positions at the current substep (constraint,
                                   // extforce)
                      {"temp", 3},
                      {"r", 3},
                      {"p", 3},
                      {"q", 3}},
                     numDofs};
    // inertial hessian
    tempI = dtiles_t{vtemp.get_allocator(), {{"Hi", 9}}, coOffset};

    // connect vtemp with "dir", "grad"
    cgtemp = tiles_t{vtemp.get_allocator(),
                     {{"P", 9},

                      {"dir", 3},

                      {"temp", 3},
                      {"r", 3},
                      {"p", 3},
                      {"q", 3}},
                     numDofs};

    auto cudaPol = zs::cuda_exec();
    // average edge length (for CCD filtering)
    initialize(cudaPol); // update vtemp, bvh, boxsize, targetGRes

    // adaptive dhat, targetGRes, kappa
    {
        // dHat (static)
        this->dHat = dHat_ * std::sqrt(boxDiagSize2);
        // adaptive epsv (static)
        if (epsv_ == 0) {
            this->epsv = this->dHat;
        } else {
            this->epsv = epsv_ * this->dHat;
        }
        // kappa (dynamic)
        suggestKappa(cudaPol);
        if (kappa0 != 0) {
            // zeno::log_info("manual kappa: {}\n", this->kappa);
        }
    }

    {
        // check initial self intersections
        // including proximity pairs
        // do once
        markSelfIntersectionPrimitives(cudaPol);
    }

    // output adaptive setups
    // zeno::log_info("auto dHat: {}, epsv (friction): {}\n", this->dHat, this->epsv);
}

void IPCSystem::reinitialize(zs::CudaExecutionPolicy &pol, typename IPCSystem::T framedt) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    dt = framedt;
    this->framedt = framedt;
    curRatio = 0;

    substep = -1;
    projectDBC = false;
    BCsatisfied = false;

    if (enableContact) {
        nPP.setVal(0);
        nPE.setVal(0);
        nPT.setVal(0);
        nEE.setVal(0);

        nPPM.setVal(0);
        nPEM.setVal(0);
        nEEM.setVal(0);

        nFPP.setVal(0);
        nFPE.setVal(0);
        nFPT.setVal(0);
        nFEE.setVal(0);

        ncsPT.setVal(0);
        ncsEE.setVal(0);
    }

    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        auto &verts = primHandle.getVerts();
        // initialize BC info
        // predict pos, initialize augmented lagrangian, constrain weights
        pol(Collapse(verts.size()), [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),
                                     voffset = primHandle.vOffset, dt = dt, asBoundary = primHandle.isBoundary(),
                                     avgNodeMass = avgNodeMass, augLagCoeff = augLagCoeff] __device__(int i) mutable {
            auto x = verts.pack<3>("x", i);
            auto v = verts.pack<3>("v", i);
            int BCorder = 0;
            auto BCtarget = x + v * dt;
            auto BCbasis = mat3::identity();
            int BCfixed = 0;
            if (!asBoundary) {
                BCorder = verts("BCorder", i);
                BCtarget = verts.pack(dim_c<3>, "BCtarget", i);
                BCbasis = verts.pack(dim_c<3, 3>, "BCbasis", i);
                BCfixed = verts("BCfixed", i);
            }
            vtemp("BCorder", voffset + i) = BCorder;
            vtemp.template tuple<3>("BCtarget", voffset + i) = BCtarget;
            vtemp.tuple(dim_c<9>, "BCbasis", voffset + i) = BCbasis;
            vtemp("BCfixed", voffset + i) = BCfixed;
            vtemp("BCsoft", voffset + i) = (int)asBoundary;

            vtemp("ws", voffset + i) = asBoundary || BCorder == 3 ? avgNodeMass * augLagCoeff : zs::sqrt(verts("m", i));
            vtemp.tuple<3>("xtilde", voffset + i) = x + v * dt;
            vtemp.tuple<3>("lambda", voffset + i) = vec3::zeros();
            vtemp.tuple<3>("xn", voffset + i) = x;
            vtemp.tuple<3>("xhat", voffset + i) = x;
            if (BCorder > 0) {
                // recover original BCtarget
                BCtarget = BCbasis * BCtarget;
                vtemp.tuple<3>("vn", voffset + i) = (BCtarget - x) / dt;
            } else {
                vtemp.tuple<3>("vn", voffset + i) = v;
            }
            // vtemp.tuple<3>("xt", voffset + i) = x;
            vtemp.tuple<3>("x0", voffset + i) = verts.pack<3>("x0", i);
        });
    }
    if (coVerts)
        if (auto coSize = coVerts->size(); coSize) {
            pol(Collapse(coSize),
                [vtemp = proxy<space>({}, vtemp), coverts = proxy<space>({}, *coVerts), coOffset = coOffset, dt = dt,
                 augLagCoeff = augLagCoeff, avgNodeMass = avgNodeMass] __device__(int i) mutable {
                    auto x = coverts.pack<3>("x", i);
                    vec3 newX{};
                    if (coverts.hasProperty("BCtarget"))
                        newX = coverts.pack<3>("BCtarget", i);
                    else {
                        auto v = coverts.pack<3>("v", i);
                        newX = x + v * dt;
                    }
                    vtemp("BCorder", coOffset + i) = 3;
                    vtemp.tuple(dim_c<9>, "BCbasis", coOffset + i) = mat3::identity();
                    vtemp.template tuple<3>("BCtarget", coOffset + i) = newX;
                    vtemp("BCfixed", coOffset + i) = (newX - x).l2NormSqr() == 0 ? 1 : 0;

                    vtemp("ws", coOffset + i) = avgNodeMass * augLagCoeff;
                    vtemp.tuple<3>("xtilde", coOffset + i) = newX;
                    vtemp.tuple<3>("lambda", coOffset + i) = vec3::zeros();
                    vtemp.tuple<3>("xn", coOffset + i) = x;
                    vtemp.tuple<3>("vn", coOffset + i) = (newX - x) / dt;
                    // vtemp.tuple<3>("xt", coOffset + i) = x;
                    vtemp.tuple<3>("xhat", coOffset + i) = x;
                    vtemp.tuple<3>("x0", coOffset + i) = coverts.pack<3>("x0", i);
                });
        }

    // spatial accel structs
    frontManageRequired = true;
#define init_front(sInds, front)                                                                                 \
    {                                                                                                            \
        auto numNodes = front.numNodes();                                                                        \
        if (numNodes <= 2) {                                                                                     \
            front.reserve(sInds.size() * numNodes);                                                              \
            front.setCounter(sInds.size() * numNodes);                                                           \
            pol(Collapse{sInds.size()}, [front = proxy<space>(selfStFront), numNodes] ZS_LAMBDA(int i) mutable { \
                for (int j = 0; j != numNodes; ++j)                                                              \
                    front.assign(i *numNodes + j, i, j);                                                         \
            });                                                                                                  \
        } else {                                                                                                 \
            front.reserve(sInds.size());                                                                         \
            front.setCounter(sInds.size());                                                                      \
            pol(Collapse{sInds.size()},                                                                          \
                [front = proxy<space>(front)] ZS_LAMBDA(int i) mutable { front.assign(i, i, 0); });              \
        }                                                                                                        \
    }
    {
        auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, 0);
        stBvh.build(pol, triBvs);
        init_front(svInds, selfStFront);

        auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", seInds, zs::wrapv<2>{}, 0);
        seBvh.build(pol, edgeBvs);
        init_front(seInds, selfSeFront);
    }
    if (coVerts)
        if (coVerts->size()) {
            auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, coOffset);
            bouStBvh.build(pol, triBvs);
            init_front(svInds, boundaryStFront);

            auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, coOffset);
            bouSeBvh.build(pol, edgeBvs);
            init_front(seInds, boundarySeFront);
        }

    updateWholeBoundingBoxSize(pol);
    /// update grad pn residual tolerance
    targetGRes = pnRel * std::sqrt(boxDiagSize2);
    // zeno::log_info("box diag size: {}, targetGRes: {}\n", std::sqrt(boxDiagSize2), targetGRes);

    /// for faster linear solve
    hess1.init(vtemp.get_allocator(), numDofs);
    hess2.init(PP.get_allocator(), estNumCps);
    hess3.init(PP.get_allocator(), estNumCps);
    hess4.init(PP.get_allocator(), estNumCps);
}
void IPCSystem::suggestKappa(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    auto cudaPol = zs::cuda_exec();
    if (kappa0 == 0) {
        /// kappaMin
        initKappa(cudaPol);
        /// adaptive kappa
        { // tet-oriented
            T H_b = computeHb((T)1e-16 * boxDiagSize2, this->dHat * this->dHat);
            kappa = 1e11 * avgNodeMass / (4e-16 * boxDiagSize2 * H_b);
            kappaMax = 100 * kappa;
            if (kappa < kappaMin)
                kappa = kappaMin;
            if (kappa > kappaMax)
                kappa = kappaMax;
        }
        { // surf oriented (use framedt here)
            auto kappaSurf = dt * dt * meanSurfaceArea / 3 * this->dHat * largestMu();
            // zeno::log_info("kappaSurf: {}, auto kappa: {}\n", kappaSurf, kappa);
            if (kappaSurf > kappa && kappaSurf < kappaMax) {
                kappa = kappaSurf;
            }
        }
        // boundaryKappa = kappa;
        // zeno::log_info("average node mass: {}, auto kappa: {} ({} - {})\n", avgNodeMass, this->kappa, this->kappaMin,
        //               this->kappaMax);
    }
}
void IPCSystem::advanceSubstep(zs::CudaExecutionPolicy &pol, typename IPCSystem::T ratio) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    // setup substep dt
    ++substep;
    dt = framedt * ratio;
    curRatio += ratio;

    projectDBC = false;
    BCsatisfied = false;
    pol(Collapse(coOffset), [vtemp = proxy<space>({}, vtemp), coOffset = coOffset, dt = dt] __device__(int vi) mutable {
        int BCorder = vtemp("BCorder", vi);
        auto BCbasis = vtemp.pack<3, 3>("BCbasis", vi);
        auto projVec = [&BCbasis, BCorder](auto &dx) {
            dx = BCbasis.transpose() * dx;
            for (int d = 0; d != BCorder; ++d)
                dx[d] = 0;
            dx = BCbasis * dx;
        };
        auto xn = vtemp.pack(dim_c<3>, "xn", vi);
        vtemp.template tuple<3>("xhat", vi) = xn;
        auto deltaX = vtemp.pack(dim_c<3>, "vn", vi) * dt;
        if (BCorder > 0)
            projVec(deltaX);
        auto newX = xn + deltaX;
        vtemp.template tuple<3>("xtilde", vi) = newX;

        // update "BCfixed", "BCtarget" for dofs under boundary influence
        if (BCorder > 0) {
            vtemp.template tuple<3>("BCtarget", vi) = BCbasis.transpose() * newX;
            vtemp("BCfixed", vi) = deltaX.l2NormSqr() == 0 ? 1 : 0;
        }
    });
    if (coVerts)
        if (auto coSize = coVerts->size(); coSize)
            pol(Collapse(coSize),
                [vtemp = proxy<space>({}, vtemp), coverts = proxy<space>({}, *coVerts), coOffset = coOffset,
                 framedt = framedt, curRatio = curRatio] __device__(int i) mutable {
                    auto xhat = vtemp.pack(dim_c<3>, "xhat", coOffset + i);
                    auto xn = vtemp.pack(dim_c<3>, "xn", coOffset + i);
                    vtemp.template tuple<3>("xhat", coOffset + i) = xn;
                    vec3 newX{};
                    if (coverts.hasProperty("BCtarget"))
                        newX = coverts.pack<3>("BCtarget", i);
                    else {
                        auto v = coverts.pack<3>("v", i);
                        newX = xhat + v * framedt;
                    }
                    // auto xk = xhat + (newX - xhat) * curRatio;
                    auto xk = newX * curRatio + (1 - curRatio) * xhat;
                    vtemp.template tuple<3>("BCtarget", coOffset + i) = xk;
                    vtemp("BCfixed", coOffset + i) = (xk - xn).l2NormSqr() == 0 ? 1 : 0;
                    vtemp.template tuple<3>("xtilde", coOffset + i) = xk;
                });
    for (auto &primHandle : auxPrims) {
        if (primHandle.category == ZenoParticles::category_e::tracker) {
            const auto &eles = primHandle.getEles();
            pol(Collapse(eles.size()), [vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
                                        framedt = framedt, curRatio = curRatio] __device__(int ei) mutable {
                auto inds = eles.pack(dim_c<2>, "inds", ei).template reinterpret_bits<int>();
                // retrieve motion from associated boundary vert
                auto deltaX = vtemp.pack(dim_c<3>, "BCtarget", inds[1]) - vtemp.pack(dim_c<3>, "xhat", inds[1]);
                //
                auto xn = vtemp.pack(dim_c<3>, "xn", inds[0]);
                vtemp.template tuple<3>("BCtarget", inds[0]) = xn + deltaX;
                vtemp.tuple(dim_c<9>, "BCbasis", inds[0]) = mat3::identity();
                vtemp("BCfixed", inds[0]) = deltaX.l2NormSqr() == 0 ? 1 : 0;
                vtemp("BCorder", inds[0]) = 3;
                vtemp("BCsoft", inds[0]) = 0;
                vtemp.template tuple<3>("xtilde", inds[0]) = xn + deltaX;
            });
        }
    }
}
void IPCSystem::updateVelocities(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    pol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp), dt = dt] __device__(int vi) mutable {
        auto newX = vtemp.pack<3>("xn", vi);
        auto dv = (newX - vtemp.pack<3>("xtilde", vi)) / dt;
        auto vn = vtemp.pack<3>("vn", vi);
        if (dv.length() > 4)
            dv = dv.normalized() * 4;
        vn += dv;
        int BCorder = vtemp("BCorder", vi);
        auto BCbasis = vtemp.pack<3, 3>("BCbasis", vi);
        auto projVec = [&BCbasis, BCorder](auto &dx) {
            dx = BCbasis.transpose() * dx;
            for (int d = 0; d != BCorder; ++d)
                dx[d] = 0;
            dx = BCbasis * dx;
        };
        if (BCorder > 0)
            projVec(vn);
        vtemp.tuple<3>("vn", vi) = vn;
    });
}
void IPCSystem::writebackPositionsAndVelocities(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        auto &verts = primHandle.getVerts();
        // update velocity and positions
        pol(zs::range(verts.size()),
            [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts), dt = dt, vOffset = primHandle.vOffset,
             asBoundary = primHandle.isBoundary()] __device__(int vi) mutable {
                verts.tuple<3>("x", vi) = vtemp.pack<3>("xn", vOffset + vi);
                if (!asBoundary)
                    verts.tuple<3>("v", vi) = vtemp.pack<3>("vn", vOffset + vi);
            });
    }
    // not sure if this is necessary for numerical reasons
    if (coVerts && coLowResVerts)
        if (auto coSize = coVerts->size(); coSize)
            pol(Collapse(coSize),
                [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, *const_cast<dtiles_t *>(coVerts)),
                 loVerts = proxy<space>({}, *const_cast<tiles_t *>(coLowResVerts)),
                 coOffset = coOffset] ZS_LAMBDA(int vi) mutable {
                    auto newX = vtemp.pack(dim_c<3>, "xn", coOffset + vi);
                    verts.template tuple<3>("x", vi) = newX;
                    loVerts.template tuple<3>("x", vi) = newX;
                    // no need to update v here. positions are moved accordingly
                    // also, boundary velocies are set elsewhere
                });
}

struct MakeIPCSystem : INode {
    void apply() override {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        auto zstets = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");
        // auto zstets = get_input<ZenoParticles>("ZSParticles");
        std::shared_ptr<ZenoParticles> zsboundary;
        if (has_input<ZenoParticles>("ZSBoundaryPrimitives"))
            zsboundary = get_input<ZenoParticles>("ZSBoundaryPrimitives");

        auto cudaPol = zs::cuda_exec();
        for (auto zstet : zstets) {
            if (!zstet->hasImage(ZenoParticles::s_particleTag)) {
                auto &loVerts = zstet->getParticles();
                auto &verts = zstet->images[ZenoParticles::s_particleTag];
                verts = typename ZenoParticles::dtiles_t{loVerts.get_allocator(), loVerts.getPropertyTags(),
                                                         loVerts.size()};
                cudaPol(range(verts.size()), [loVerts = proxy<space>({}, loVerts),
                                              verts = proxy<space>({}, verts)] __device__(int vi) mutable {
                    // make sure there are no "inds"-like properties in verts!
                    for (int propid = 0; propid != verts._N; ++propid) {
                        auto propOffset = verts._tagOffsets[propid];
                        for (int chn = 0; chn != verts._tagSizes[propid]; ++chn)
                            verts(propOffset + chn, vi) = loVerts(propOffset + chn, vi);
                    }
                });
            }
        }
        if (zsboundary)
            if (!zsboundary->hasImage(ZenoParticles::s_particleTag)) {
                auto &loVerts = zsboundary->getParticles();
                auto &verts = zsboundary->images[ZenoParticles::s_particleTag];
                verts = typename ZenoParticles::dtiles_t{loVerts.get_allocator(), loVerts.getPropertyTags(),
                                                         loVerts.size()};
                cudaPol(range(verts.size()), [loVerts = proxy<space>({}, loVerts),
                                              verts = proxy<space>({}, verts)] __device__(int vi) mutable {
                    // make sure there are no "inds"-like properties in verts!
                    for (int propid = 0; propid != verts._N; ++propid) {
                        auto propOffset = verts._tagOffsets[propid];
                        for (int chn = 0; chn != verts._tagSizes[propid]; ++chn)
                            verts(propOffset + chn, vi) = loVerts(propOffset + chn, vi);
                    }
                });
            }

        const typename IPCSystem::dtiles_t *coVerts =
            zsboundary ? &zsboundary->images[ZenoParticles::s_particleTag] : nullptr;
        const typename IPCSystem::tiles_t *coLowResVerts = zsboundary ? &zsboundary->getParticles() : nullptr;
        const typename IPCSystem::tiles_t *coEdges =
            zsboundary ? &(*zsboundary)[ZenoParticles::s_surfEdgeTag] : nullptr;
        const typename IPCSystem::tiles_t *coEles = zsboundary ? &zsboundary->getQuadraturePoints() : nullptr;

        /// solver parameters
        auto input_est_num_cps = get_input2<int>("est_num_cps");
        auto input_withGround = get_input2<bool>("with_ground");
        auto input_withContact = get_input2<bool>("with_contact");
        auto input_withMollification = get_input2<bool>("with_mollification");
        auto input_contactEE = get_input2<bool>("contact_with_ee");
        auto input_contactSelf = get_input2<bool>("contact_with_self");
        auto input_dHat = get_input2<float>("dHat");
        auto input_epsv = get_input2<float>("epsv");
        auto input_kappa0 = get_input2<float>("kappa0");
        auto input_fricIterCap = get_input2<int>("fric_iter_cap");
        auto input_fric_mu = get_input2<float>("fric_mu");
        auto input_aug_coeff = get_input2<float>("aug_coeff");
        auto input_pn_rel = get_input2<float>("pn_rel");
        auto input_cg_rel = get_input2<float>("cg_rel");
        auto input_pn_cap = get_input2<int>("pn_iter_cap");
        auto input_cg_cap = get_input2<int>("cg_iter_cap");
        auto input_ccd_cap = get_input2<int>("ccd_iter_cap");
        auto input_gravity = get_input2<float>("gravity");
        auto dt = get_input2<float>("dt");
        auto groundNormal = get_input<zeno::NumericObject>("ground_normal")->get<zeno::vec3f>();
        if (auto len2 = lengthSquared(groundNormal); len2 > limits<float>::epsilon() * 10) {
            auto len = std::sqrt(len2);
            groundNormal /= len;
        } else
            groundNormal = zeno::vec3f{0, 1, 0}; // fallback to default up direction when degenerated

        auto A = std::make_shared<IPCSystem>(
            zstets, coVerts, coLowResVerts, coEdges, coEles, dt,
            (std::size_t)(input_est_num_cps ? input_est_num_cps : 1000000), input_withGround, input_withContact,
            input_withMollification, input_aug_coeff, input_pn_rel, input_cg_rel, input_pn_cap, input_cg_cap,
            input_ccd_cap, input_kappa0, input_fric_mu, input_dHat, input_epsv, groundNormal, input_gravity);
        A->enableContactEE = input_contactEE;
        A->enableContactSelf = input_contactSelf;
        A->fricIterCap = input_fricIterCap;

        set_output("ZSIPCSystem", A);
    }
};

ZENDEFNODE(MakeIPCSystem, {{
                               "ZSParticles",
                               "ZSBoundaryPrimitives",
                               {"int", "est_num_cps", "1000000"},
                               {"bool", "with_ground", "0"},
                               {"bool", "with_contact", "1"},
                               {"bool", "with_mollification", "1"},
                               {"bool", "contact_with_ee", "1"},
                               {"bool", "contact_with_self", "1"},
                               {"float", "dt", "0.01"},
                               {"float", "dHat", "0.001"},
                               {"vec3f", "ground_normal", "0,1,0"},
                               {"float", "epsv", "0.0"},
                               {"float", "kappa0", "0"},
                               {"int", "fric_iter_cap", "2"},
                               {"float", "fric_mu", "0"},
                               {"float", "aug_coeff", "1e2"},
                               {"float", "pn_rel", "0.01"},
                               {"float", "cg_rel", "0.001"},
                               {"int", "pn_iter_cap", "1000"},
                               {"int", "cg_iter_cap", "1000"},
                               {"int", "ccd_iter_cap", "20000"},
                               {"float", "gravity", "-9.8"},
                           },
                           {"ZSIPCSystem"},
                           {},
                           {"FEM"}});

} // namespace zeno