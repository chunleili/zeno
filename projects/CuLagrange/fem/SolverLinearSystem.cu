#include "hip/hip_runtime.h"
#include "Solver.cuh"
#include "Utils.hpp"
#include "zensim/geometry/Distance.hpp"
#include "zensim/geometry/Friction.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include "zensim/types/SmallVector.hpp"

namespace zeno {

template <typename VecT, int N = VecT::template range_t<0>::value,
          zs::enable_if_all<N % 3 == 0, N == VecT::template range_t<1>::value> = 0>
__forceinline__ __device__ void rotate_hessian(zs::VecInterface<VecT> &H, const typename IPCSystem::mat3 BCbasis[N / 3],
                                               const int BCorder[N / 3], const int BCfixed[], bool projectDBC) {
    // hessian rotation: trans^T hess * trans
    // left trans^T: multiplied on rows
    // right trans: multiplied on cols
    constexpr int NV = N / 3;
    // rotate and project
    for (int vi = 0; vi != NV; ++vi) {
        int offsetI = vi * 3;
        for (int vj = 0; vj != NV; ++vj) {
            int offsetJ = vj * 3;
            IPCSystem::mat3 tmp{};
            for (int i = 0; i != 3; ++i)
                for (int j = 0; j != 3; ++j)
                    tmp(i, j) = H(offsetI + i, offsetJ + j);
            // rotate
            tmp = BCbasis[vi].transpose() * tmp * BCbasis[vj];
            // project
            if (projectDBC) {
                for (int i = 0; i != 3; ++i) {
                    bool clearRow = i < BCorder[vi];
                    for (int j = 0; j != 3; ++j) {
                        bool clearCol = j < BCorder[vj];
                        if (clearRow || clearCol)
                            tmp(i, j) = (vi == vj && i == j ? 1 : 0);
                    }
                }
            } else {
                for (int i = 0; i != 3; ++i) {
                    bool clearRow = i < BCorder[vi] && BCfixed[vi] == 1;
                    for (int j = 0; j != 3; ++j) {
                        bool clearCol = j < BCorder[vj] && BCfixed[vj] == 1;
                        if (clearRow || clearCol)
                            tmp(i, j) = (vi == vj && i == j ? 1 : 0);
                    }
                }
            }
            for (int i = 0; i != 3; ++i)
                for (int j = 0; j != 3; ++j)
                    H(offsetI + i, offsetJ + j) = tmp(i, j);
        }
    }
    return;
}

/// inertia
void IPCSystem::computeInertialAndGravityPotentialGradient(zs::CudaExecutionPolicy &cudaPol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    // inertial
    cudaPol(zs::range(coOffset), [tempI = proxy<space>({}, tempI), vtemp = proxy<space>({}, vtemp), dt = dt,
                                  projectDBC = projectDBC] ZS_LAMBDA(int i) mutable {
        auto m = zs::sqr(vtemp("ws", i));
        vtemp.tuple<3>("grad", i) =
            vtemp.pack<3>("grad", i) - m * (vtemp.pack<3>("xn", i) - vtemp.pack<3>("xtilde", i));

        auto M = mat3::identity() * m;
        mat3 BCbasis[1] = {vtemp.pack(dim_c<3, 3>, "BCbasis", i)};
        int BCorder[1] = {(int)vtemp("BCorder", i)};
        int BCfixed[1] = {(int)vtemp("BCfixed", i)};
        rotate_hessian(M, BCbasis, BCorder, BCfixed, projectDBC);
        tempI.tuple(dim_c<9>, "Hi", i) = M;
        // prepare preconditioner
        for (int r = 0; r != 3; ++r)
            for (int c = 0; c != 3; ++c)
                vtemp("P", r * 3 + c, i) += M(r, c);
    });
    // extforce (only grad modified)
    for (auto &primHandle : prims) {
        if (primHandle.isBoundary()) // skip soft boundary
            continue;
        cudaPol(zs::range(primHandle.getVerts().size()), [vtemp = proxy<space>({}, vtemp), extForce = extForce, dt = dt,
                                                          vOffset = primHandle.vOffset] ZS_LAMBDA(int vi) mutable {
            auto m = zs::sqr(vtemp("ws", vOffset + vi));
            int BCorder = vtemp("BCorder", vOffset + vi);
            int BCsoft = vtemp("BCsoft", vOffset + vi);
            if (BCsoft == 0 && BCorder != 3)
                vtemp.tuple<3>("grad", vOffset + vi) = vtemp.pack<3>("grad", vOffset + vi) + m * extForce * dt * dt;
        });
    }
    if (vtemp.hasProperty("extf")) {
        cudaPol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp), dt = dt] ZS_LAMBDA(int vi) mutable {
            int BCorder = vtemp("BCorder", vi);
            int BCsoft = vtemp("BCsoft", vi);
            if (BCsoft == 0 && BCorder != 3)
                vtemp.template tuple<3>("grad", vi) =
                    vtemp.pack(dim_c<3>, "grad", vi) + vtemp.pack(dim_c<3>, "extf", vi) * dt * dt;
        });
    }
}
void IPCSystem::computeInertialPotentialGradient(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    // inertial
    cudaPol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp), gTag, dt = dt] ZS_LAMBDA(int i) mutable {
        auto m = zs::sqr(vtemp("ws", i));
        vtemp.tuple<3>(gTag, i) = vtemp.pack<3>(gTag, i) - m * (vtemp.pack<3>("xn", i) - vtemp.pack<3>("xtilde", i));
    });
}

/// elasticity
template <typename Model>
void computeElasticGradientAndHessianImpl(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag,
                                          typename IPCSystem::dtiles_t &vtemp,
                                          typename IPCSystem::PrimitiveHandle &primHandle, const Model &model,
                                          typename IPCSystem::T dt, bool projectDBC, bool includeHessian) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    using mat3 = typename IPCSystem::mat3;
    using vec3 = typename IPCSystem::vec3;
    using T = typename IPCSystem::T;
    if (primHandle.category == ZenoParticles::curve) {
        if (primHandle.isBoundary() && !primHandle.isAuxiliary())
            return;
        /// ref: Fast Simulation of Mass-Spring Systems
        /// credits: Tiantian Liu
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), etemp = proxy<space>({}, primHandle.etemp),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt, projectDBC = projectDBC,
                 vOffset = primHandle.vOffset, includeHessian,
                 n = primHandle.getEles().size()] __device__(int ei) mutable {
                    auto inds = eles.pack(dim_c<2>, "inds", ei).template reinterpret_bits<int>() + vOffset;
                    mat3 BCbasis[2];
                    int BCorder[2];
                    int BCfixed[2];
                    for (int i = 0; i != 2; ++i) {
                        BCbasis[i] = vtemp.pack<3, 3>("BCbasis", inds[i]);
                        BCorder[i] = vtemp("BCorder", inds[i]);
                        BCfixed[i] = vtemp("BCfixed", inds[i]);
                    }

                    if (BCorder[0] == 3 && BCorder[1] == 3) {
                        etemp.tuple<6 * 6>("He", ei) = zs::vec<T, 6, 6>::zeros();
                        return;
                    }

                    auto vole = eles("vol", ei);
                    auto k = eles("k", ei);
                    auto rl = eles("rl", ei);

                    vec3 xs[2] = {vtemp.pack(dim_c<3>, "xn", inds[0]), vtemp.pack(dim_c<3>, "xn", inds[1])};
                    auto xij = xs[1] - xs[0];
                    auto lij = xij.norm();
                    auto dij = xij / lij;
                    auto gij = k * (lij - rl) * dij;

                    // gradient
                    auto vfdt2 = gij * (dt * dt) * vole;
                    for (int d = 0; d != 3; ++d) {
                        atomic_add(exec_cuda, &vtemp(gTag, d, inds[0]), (T)vfdt2(d));
                        atomic_add(exec_cuda, &vtemp(gTag, d, inds[1]), (T)-vfdt2(d));
                    }

                    if (!includeHessian)
                        return;
                    auto H = zs::vec<T, 6, 6>::zeros();
                    auto K = k * (mat3::identity() - rl / lij * (mat3::identity() - dyadic_prod(dij, dij)));
                    // make_pd(K);  // symmetric semi-definite positive, not
                    // necessary

                    for (int i = 0; i != 3; ++i)
                        for (int j = 0; j != 3; ++j) {
                            H(i, j) = K(i, j);
                            H(i, 3 + j) = -K(i, j);
                            H(3 + i, j) = -K(i, j);
                            H(3 + i, 3 + j) = K(i, j);
                        }
                    H *= dt * dt * vole;

                    // rotate and project
                    rotate_hessian(H, BCbasis, BCorder, BCfixed, projectDBC);
                    etemp.tuple<6 * 6>("He", ei) = H;
                    for (int vi = 0; vi != 2; ++vi) {
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j) {
                                atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]), H(vi * 3 + i, vi * 3 + j));
                            }
                    }
                });
    } else if (primHandle.category == ZenoParticles::surface) {
        if (primHandle.isBoundary())
            return;
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), etemp = proxy<space>({}, primHandle.etemp),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt, projectDBC = projectDBC,
                 vOffset = primHandle.vOffset, includeHessian] __device__(int ei) mutable {
                    auto IB = eles.template pack<2, 2>("IB", ei);
                    auto inds = eles.pack(dim_c<3>, "inds", ei).template reinterpret_bits<int>() + vOffset;
                    auto vole = eles("vol", ei);
                    vec3 xs[3] = {vtemp.pack(dim_c<3>, "xn", inds[0]), vtemp.pack(dim_c<3>, "xn", inds[1]),
                                  vtemp.pack(dim_c<3>, "xn", inds[2])};
                    auto x1x0 = xs[1] - xs[0];
                    auto x2x0 = xs[2] - xs[0];

                    mat3 BCbasis[3];
                    int BCorder[3];
                    int BCfixed[3];
                    for (int i = 0; i != 3; ++i) {
                        BCbasis[i] = vtemp.pack<3, 3>("BCbasis", inds[i]);
                        BCorder[i] = vtemp("BCorder", inds[i]);
                        BCfixed[i] = vtemp("BCfixed", inds[i]);
                    }
                    zs::vec<T, 9, 9> H;
                    if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3) {
                        etemp.tuple<9 * 9>("He", ei) = H.zeros();
                        return;
                    }

                    zs::vec<T, 3, 2> Ds{x1x0[0], x2x0[0], x1x0[1], x2x0[1], x1x0[2], x2x0[2]};
                    auto F = Ds * IB;

                    auto dFdX = dFdXMatrix(IB, wrapv<3>{});
                    auto dFdXT = dFdX.transpose();
                    auto f0 = col(F, 0);
                    auto f1 = col(F, 1);
                    auto f0Norm = zs::sqrt(f0.l2NormSqr());
                    auto f1Norm = zs::sqrt(f1.l2NormSqr());
                    auto f0Tf1 = f0.dot(f1);
                    zs::vec<T, 3, 2> Pstretch, Pshear;
                    for (int d = 0; d != 3; ++d) {
                        Pstretch(d, 0) = 2 * (1 - 1 / f0Norm) * F(d, 0);
                        Pstretch(d, 1) = 2 * (1 - 1 / f1Norm) * F(d, 1);
                        Pshear(d, 0) = 2 * f0Tf1 * f1(d);
                        Pshear(d, 1) = 2 * f0Tf1 * f0(d);
                    }
                    auto vecP = flatten(model.mu * Pstretch + (model.mu * 0.3) * Pshear);
                    auto vfdt2 = -vole * (dFdXT * vecP) * (dt * dt);

                    for (int i = 0; i != 3; ++i) {
                        auto vi = inds[i];
                        for (int d = 0; d != 3; ++d)
                            atomic_add(exec_cuda, &vtemp(gTag, d, vi), (T)vfdt2(i * 3 + d));
                    }

                    if (!includeHessian)
                        return;
                    /// ref: A Finite Element Formulation of Baraff-Witkin Cloth
                    // suggested by huang kemeng
                    auto stretchHessian = [&F, &model]() {
                        auto H = zs::vec<T, 6, 6>::zeros();
                        const zs::vec<T, 2> u{1, 0};
                        const zs::vec<T, 2> v{0, 1};
                        const T I5u = (F * u).l2NormSqr();
                        const T I5v = (F * v).l2NormSqr();
                        const T invSqrtI5u = (T)1 / zs::sqrt(I5u);
                        const T invSqrtI5v = (T)1 / zs::sqrt(I5v);

                        H(0, 0) = H(1, 1) = H(2, 2) = zs::max(1 - invSqrtI5u, (T)0);
                        H(3, 3) = H(4, 4) = H(5, 5) = zs::max(1 - invSqrtI5v, (T)0);

                        const auto fu = col(F, 0).normalized();
                        const T uCoeff = (1 - invSqrtI5u >= 0) ? invSqrtI5u : (T)1;
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j)
                                H(i, j) += uCoeff * fu(i) * fu(j);

                        const auto fv = col(F, 1).normalized();
                        const T vCoeff = (1 - invSqrtI5v >= 0) ? invSqrtI5v : (T)1;
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j)
                                H(3 + i, 3 + j) += vCoeff * fv(i) * fv(j);

                        H *= model.mu;
                        return H;
                    };
                    auto shearHessian = [&F, &model]() {
                        using mat6 = zs::vec<T, 6, 6>;
                        auto H = mat6::zeros();
                        const zs::vec<T, 2> u{1, 0};
                        const zs::vec<T, 2> v{0, 1};
                        const T I6 = (F * u).dot(F * v);
                        const T signI6 = I6 >= 0 ? 1 : -1;

                        H(3, 0) = H(4, 1) = H(5, 2) = H(0, 3) = H(1, 4) = H(2, 5) = (T)1;

                        const auto g_ = F * (dyadic_prod(u, v) + dyadic_prod(v, u));
                        zs::vec<T, 6> g{};
                        for (int j = 0, offset = 0; j != 2; ++j) {
                            for (int i = 0; i != 3; ++i)
                                g(offset++) = g_(i, j);
                        }

                        const T I2 = F.l2NormSqr();
                        const T lambda0 = (T)0.5 * (I2 + zs::sqrt(I2 * I2 + (T)12 * I6 * I6));

                        const zs::vec<T, 6> q0 = (I6 * H * g + lambda0 * g).normalized();

                        auto t = mat6::identity();
                        t = 0.5 * (t + signI6 * H);

                        const zs::vec<T, 6> Tq = t * q0;
                        const auto normTq = Tq.l2NormSqr();

                        mat6 dPdF =
                            zs::abs(I6) * (t - (dyadic_prod(Tq, Tq) / normTq)) + lambda0 * (dyadic_prod(q0, q0));
                        dPdF *= (model.mu * 0.3);
                        return dPdF;
                    };
                    auto He = stretchHessian() + shearHessian();
                    H = dFdX.transpose() * He * dFdX;
                    H *= dt * dt * vole;

                    // rotate and project
                    rotate_hessian(H, BCbasis, BCorder, BCfixed, projectDBC);
                    etemp.tuple<9 * 9>("He", ei) = H;
                    for (int vi = 0; vi != 3; ++vi) {
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j) {
                                atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]), H(vi * 3 + i, vi * 3 + j));
                            }
                    }
                });
    } else if (primHandle.category == ZenoParticles::tet)
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), etemp = proxy<space>({}, primHandle.etemp),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt, projectDBC = projectDBC,
                 vOffset = primHandle.vOffset, includeHessian] __device__(int ei) mutable {
                    auto IB = eles.pack(dim_c<3, 3>, "IB", ei);
                    auto inds = eles.pack(dim_c<4>, "inds", ei).template reinterpret_bits<int>() + vOffset;
                    auto vole = eles("vol", ei);
                    vec3 xs[4] = {vtemp.pack<3>("xn", inds[0]), vtemp.pack<3>("xn", inds[1]),
                                  vtemp.pack<3>("xn", inds[2]), vtemp.pack<3>("xn", inds[3])};

                    mat3 BCbasis[4];
                    int BCorder[4];
                    int BCfixed[4];
                    for (int i = 0; i != 4; ++i) {
                        BCbasis[i] = vtemp.pack<3, 3>("BCbasis", inds[i]);
                        BCorder[i] = vtemp("BCorder", inds[i]);
                        BCfixed[i] = vtemp("BCfixed", inds[i]);
                    }
                    zs::vec<T, 12, 12> H;
                    if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 && BCorder[3] == 3) {
                        etemp.tuple<12 * 12>("He", ei) = H.zeros();
                        return;
                    }
                    mat3 F{};
                    {
                        auto x1x0 = xs[1] - xs[0];
                        auto x2x0 = xs[2] - xs[0];
                        auto x3x0 = xs[3] - xs[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1], x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        F = Ds * IB;
                    }
                    auto P = model.first_piola(F);
                    auto vecP = flatten(P);
                    auto dFdX = dFdXMatrix(IB);
                    auto dFdXT = dFdX.transpose();
                    auto vfdt2 = -vole * (dFdXT * vecP) * dt * dt;

                    for (int i = 0; i != 4; ++i) {
                        auto vi = inds[i];
                        for (int d = 0; d != 3; ++d)
                            atomic_add(exec_cuda, &vtemp(gTag, d, vi), (T)vfdt2(i * 3 + d));
                    }

                    if (!includeHessian)
                        return;
                    auto Hq = model.first_piola_derivative(F, true_c);
                    H = dFdXT * Hq * dFdX * vole * dt * dt;

                    // rotate and project
                    rotate_hessian(H, BCbasis, BCorder, BCfixed, projectDBC);
                    etemp.tuple<12 * 12>("He", ei) = H;
                    for (int vi = 0; vi != 4; ++vi) {
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j) {
                                atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]), H(vi * 3 + i, vi * 3 + j));
                            }
                    }
                });
}

void IPCSystem::computeElasticGradientAndHessian(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag,
                                                 bool includeHessian) {
    using namespace zs;
    for (auto &primHandle : prims) {
        match([&](auto &elasticModel) {
            computeElasticGradientAndHessianImpl(cudaPol, gTag, vtemp, primHandle, elasticModel, dt, projectDBC,
                                                 includeHessian);
        })(primHandle.getModels().getElasticModel());
    }
    for (auto &primHandle : auxPrims) {
        using ModelT = RM_CVREF_T(primHandle.getModels().getElasticModel());
        const ModelT &model = primHandle.modelsPtr ? primHandle.getModels().getElasticModel() : ModelT{};
        match([&](auto &elasticModel) {
            computeElasticGradientAndHessianImpl(cudaPol, gTag, vtemp, primHandle, elasticModel, dt, projectDBC,
                                                 includeHessian);
        })(model);
    }
}

void IPCSystem::computeBoundaryBarrierGradientAndHessian(zs::CudaExecutionPolicy &pol, bool includeHessian) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    for (auto &primHandle : prims) {
        if (primHandle.isBoundary()) // skip soft boundary
            continue;
        const auto &svs = primHandle.getSurfVerts();
        pol(range(svs.size()),
            [vtemp = proxy<space>({}, vtemp), svtemp = proxy<space>({}, primHandle.svtemp), svs = proxy<space>({}, svs),
             gn = s_groundNormal, dHat2 = dHat * dHat, kappa = kappa, projectDBC = projectDBC, includeHessian,
             svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                const auto vi = reinterpret_bits<int>(svs("inds", svi)) + svOffset;
                auto x = vtemp.pack<3>("xn", vi);
                auto dist = gn.dot(x);
                auto dist2 = dist * dist;
                auto t = dist2 - dHat2;
                auto g_b = t * zs::log(dist2 / dHat2) * -2 - (t * t) / dist2;
                auto H_b = (zs::log(dist2 / dHat2) * -2.0 - t * 4.0 / dist2) + 1.0 / (dist2 * dist2) * (t * t);
                if (dist2 < dHat2) {
                    auto grad = -gn * (kappa * g_b * 2 * dist);
                    for (int d = 0; d != 3; ++d)
                        atomic_add(exec_cuda, &vtemp("grad", d, vi), grad(d));
                }

                if (!includeHessian)
                    return;
                auto param = 4 * H_b * dist2 + 2 * g_b;
                auto hess = mat3::zeros();
                if (dist2 < dHat2 && param > 0) {
                    auto nn = dyadic_prod(gn, gn);
                    hess = (kappa * param) * nn;
                }

                // make_pd(hess);
                mat3 BCbasis[1] = {vtemp.pack<3, 3>("BCbasis", vi)};
                int BCorder[1] = {(int)vtemp("BCorder", vi)};
                int BCfixed[1] = {(int)vtemp("BCfixed", vi)};
                rotate_hessian(hess, BCbasis, BCorder, BCfixed, projectDBC);
                svtemp.tuple<9>("H", svi) = hess;
                for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j) {
                        atomic_add(exec_cuda, &vtemp("P", i * 3 + j, vi), hess(i, j));
                    }
            });

        if (s_enableFriction)
            if (fricMu != 0) {
                pol(range(svs.size()), [vtemp = proxy<space>({}, vtemp), svtemp = proxy<space>({}, primHandle.svtemp),
                                        svs = proxy<space>({}, svs), epsvh = epsv * dt, gn = s_groundNormal,
                                        fricMu = fricMu, projectDBC = projectDBC, includeHessian,
                                        svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                    const auto vi = reinterpret_bits<int>(svs("inds", svi)) + svOffset;
                    auto dx = vtemp.pack<3>("xn", vi) - vtemp.pack<3>("xhat", vi);
                    auto fn = svtemp("fn", svi);
                    if (fn == 0) {
                        return;
                    }
                    auto coeff = fn * fricMu;
                    auto relDX = dx - gn.dot(dx) * gn;
                    auto relDXNorm2 = relDX.l2NormSqr();
                    auto relDXNorm = zs::sqrt(relDXNorm2);

                    vec3 grad{};
                    if (relDXNorm2 > epsvh * epsvh)
                        grad = -relDX * (coeff / relDXNorm);
                    else
                        grad = -relDX * (coeff / epsvh);
                    for (int d = 0; d != 3; ++d)
                        atomic_add(exec_cuda, &vtemp("grad", d, vi), grad(d));

                    if (!includeHessian)
                        return;

                    auto hess = mat3::zeros();
                    if (relDXNorm2 > epsvh * epsvh) {
                        zs::vec<T, 2, 2> mat{relDX[0] * relDX[0] * -coeff / relDXNorm2 / relDXNorm + coeff / relDXNorm,
                                             relDX[0] * relDX[2] * -coeff / relDXNorm2 / relDXNorm,
                                             relDX[0] * relDX[2] * -coeff / relDXNorm2 / relDXNorm,
                                             relDX[2] * relDX[2] * -coeff / relDXNorm2 / relDXNorm + coeff / relDXNorm};
                        make_pd(mat);
                        hess(0, 0) = mat(0, 0);
                        hess(0, 2) = mat(0, 1);
                        hess(2, 0) = mat(1, 0);
                        hess(2, 2) = mat(1, 1);
                    } else {
                        hess(0, 0) = coeff / epsvh;
                        hess(2, 2) = coeff / epsvh;
                    }

                    mat3 BCbasis[1] = {vtemp.pack<3, 3>("BCbasis", vi)};
                    int BCorder[1] = {(int)vtemp("BCorder", vi)};
                    int BCfixed[1] = {(int)vtemp("BCfixed", vi)};
                    rotate_hessian(hess, BCbasis, BCorder, BCfixed, projectDBC);
                    svtemp.tuple(dim_c<9>, "H", svi) = svtemp.pack(dim_c<3, 3>, "H", svi) + hess;
                    for (int i = 0; i != 3; ++i)
                        for (int j = 0; j != 3; ++j) {
                            atomic_add(exec_cuda, &vtemp("P", i * 3 + j, vi), hess(i, j));
                        }
                });
            }
    }
    return;
}

void IPCSystem::convertHessian(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr execspace_e space = execspace_e::cuda;
    constexpr auto execTag = wrapv<space>{};

    hess1.reset(true, numDofs); // additive style
    hess2.reset(false, 0);      // overwrite style
    hess3.reset(false, 0);
    hess4.reset(false, 0);
    // inertial
    pol(zs::range(coOffset), [tempI = proxy<space>({}, tempI), hess1 = proxy<space>(hess1)] __device__(int i) mutable {
        auto Hi = tempI.pack(dim_c<3, 3>, "Hi", i);
        hess1.hess[i] = Hi;
        hess1.inds[i][0] = i;
    });

    // elasticity
    for (auto &primHandle : prims) {
        auto &eles = primHandle.getEles();
        // elasticity
        if (primHandle.category == ZenoParticles::curve) {
            if (primHandle.isBoundary() && !primHandle.isAuxiliary())
                continue;
            auto offset = hess2.increaseCount(eles.size());
            pol(zs::range(eles.size()),
                [etemp = proxy<space>({}, primHandle.etemp), eles = proxy<space>({}, eles), hess2 = proxy<space>(hess2),
                 vOffset = primHandle.vOffset, offset] ZS_LAMBDA(int ei) mutable {
                    auto He = etemp.pack(dim_c<6, 6>, "He", ei);
                    auto inds = eles.pack(dim_c<2>, "inds", ei).template reinterpret_bits<int>() + vOffset;
                    hess2.hess[offset + ei] = He;
                    hess2.inds[offset + ei] = inds;
                });
        } else if (primHandle.category == ZenoParticles::surface) {
            if (primHandle.isBoundary())
                continue;
            auto offset = hess3.increaseCount(eles.size());
            pol(zs::range(eles.size()),
                [etemp = proxy<space>({}, primHandle.etemp), eles = proxy<space>({}, eles), hess3 = proxy<space>(hess3),
                 vOffset = primHandle.vOffset, offset] ZS_LAMBDA(int ei) mutable {
                    auto He = etemp.pack(dim_c<9, 9>, "He", ei);
                    auto inds = eles.pack(dim_c<3>, "inds", ei).template reinterpret_bits<int>() + vOffset;
                    hess3.hess[offset + ei] = He;
                    hess3.inds[offset + ei] = inds;
                });
        } else if (primHandle.category == ZenoParticles::tet) {
            auto offset = hess4.increaseCount(eles.size());
            pol(zs::range(eles.size()),
                [etemp = proxy<space>({}, primHandle.etemp), eles = proxy<space>({}, eles), hess4 = proxy<space>(hess4),
                 vOffset = primHandle.vOffset, offset] ZS_LAMBDA(int ei) mutable {
                    auto He = etemp.pack(dim_c<12, 12>, "He", ei);
                    auto inds = eles.pack(dim_c<4>, "inds", ei).template reinterpret_bits<int>() + vOffset;
                    hess4.hess[offset + ei] = He;
                    hess4.inds[offset + ei] = inds;
                });
        }
        for (auto &primHandle : auxPrims) {
            auto &eles = primHandle.getEles();
            // soft bindings
            if (primHandle.category == ZenoParticles::curve) {
                auto offset = hess2.increaseCount(eles.size());
                pol(zs::range(eles.size()),
                    [etemp = proxy<space>({}, primHandle.etemp), eles = proxy<space>({}, eles),
                     hess2 = proxy<space>(hess2), vOffset = primHandle.vOffset, offset] ZS_LAMBDA(int ei) mutable {
                        auto He = etemp.pack(dim_c<6, 6>, "He", ei);
                        auto inds = eles.pack(dim_c<2>, "inds", ei).template reinterpret_bits<int>() + vOffset;
                        hess2.hess[offset + ei] = He;
                        hess2.inds[offset + ei] = inds;
                    });
            }
        }

        // contacts
        if (enableContact) {
            auto numPP = nPP.getVal();
            auto offset = hess2.increaseCount(numPP);
            pol(zs::range(numPP), [tempPP = proxy<space>({}, tempPP), PP = proxy<space>(PP),
                                   hess2 = proxy<space>(hess2), offset] ZS_LAMBDA(int ppi) mutable {
                auto H = tempPP.pack(dim_c<6, 6>, "H", ppi);
                auto inds = PP[ppi];
                hess2.hess[offset + ppi] = H;
                hess2.inds[offset + ppi] = inds;
            });

            auto numPE = nPE.getVal();
            offset = hess3.increaseCount(numPE);
            pol(zs::range(numPE), [tempPE = proxy<space>({}, tempPE), PE = proxy<space>(PE),
                                   hess3 = proxy<space>(hess3), offset] ZS_LAMBDA(int pei) mutable {
                auto H = tempPE.pack(dim_c<9, 9>, "H", pei);
                auto inds = PE[pei];
                hess3.hess[offset + pei] = H;
                hess3.inds[offset + pei] = inds;
            });

            auto numPT = nPT.getVal();
            offset = hess4.increaseCount(numPT);
            pol(zs::range(numPT), [tempPT = proxy<space>({}, tempPT), PT = proxy<space>(PT),
                                   hess4 = proxy<space>(hess4), offset] ZS_LAMBDA(int pti) mutable {
                auto H = tempPT.pack(dim_c<12, 12>, "H", pti);
                auto inds = PT[pti];
                hess4.hess[offset + pti] = H;
                hess4.inds[offset + pti] = inds;
            });

            auto numEE = nEE.getVal();
            offset = hess4.increaseCount(numEE);
            pol(zs::range(numEE), [tempEE = proxy<space>({}, tempEE), EE = proxy<space>(EE),
                                   hess4 = proxy<space>(hess4), offset] ZS_LAMBDA(int eei) mutable {
                auto H = tempEE.pack(dim_c<12, 12>, "H", eei);
                auto inds = EE[eei];
                hess4.hess[offset + eei] = H;
                hess4.inds[offset + eei] = inds;
            });

            if (enableMollification) {
                auto numEEM = nEEM.getVal();
                offset = hess4.increaseCount(numEEM);
                pol(zs::range(numEEM), [tempEEM = proxy<space>({}, tempEEM), EEM = proxy<space>(EEM),
                                        hess4 = proxy<space>(hess4), offset] ZS_LAMBDA(int eemi) mutable {
                    auto H = tempEEM.pack(dim_c<12, 12>, "H", eemi);
                    auto inds = EEM[eemi];
                    hess4.hess[offset + eemi] = H;
                    hess4.inds[offset + eemi] = inds;
                });

                auto numPPM = nPPM.getVal();
                offset = hess4.increaseCount(numPPM);
                pol(zs::range(numPPM), [tempPPM = proxy<space>({}, tempPPM), PPM = proxy<space>(PPM),
                                        hess4 = proxy<space>(hess4), offset] ZS_LAMBDA(int ppmi) mutable {
                    auto H = tempPPM.pack(dim_c<12, 12>, "H", ppmi);
                    auto inds = PPM[ppmi];
                    hess4.hess[offset + ppmi] = H;
                    hess4.inds[offset + ppmi] = inds;
                });

                auto numPEM = nPEM.getVal();
                offset = hess4.increaseCount(numPEM);
                pol(zs::range(numPEM), [tempPEM = proxy<space>({}, tempPEM), PEM = proxy<space>(PEM),
                                        hess4 = proxy<space>(hess4), offset] ZS_LAMBDA(int pemi) mutable {
                    auto H = tempPEM.pack(dim_c<12, 12>, "H", pemi);
                    auto inds = PEM[pemi];
                    hess4.hess[offset + pemi] = H;
                    hess4.inds[offset + pemi] = inds;
                });
            } // end mollification

            if (s_enableFriction) {
                if (fricMu != 0) {
                    if (s_enableSelfFriction) {
                        auto numFPP = nFPP.getVal();
                        offset = hess2.increaseCount(numFPP);
                        pol(zs::range(numFPP), [fricPP = proxy<space>({}, fricPP), FPP = proxy<space>(FPP),
                                                hess2 = proxy<space>(hess2), offset] ZS_LAMBDA(int fppi) mutable {
                            auto H = fricPP.pack(dim_c<6, 6>, "H", fppi);
                            auto inds = FPP[fppi];
                            hess2.hess[offset + fppi] = H;
                            hess2.inds[offset + fppi] = inds;
                        });

                        auto numFPE = nFPE.getVal();
                        offset = hess3.increaseCount(numFPE);
                        pol(zs::range(numFPE), [fricPE = proxy<space>({}, fricPE), FPE = proxy<space>(FPE),
                                                hess3 = proxy<space>(hess3), offset] ZS_LAMBDA(int fpei) mutable {
                            auto H = fricPE.pack(dim_c<9, 9>, "H", fpei);
                            auto inds = FPE[fpei];
                            hess3.hess[offset + fpei] = H;
                            hess3.inds[offset + fpei] = inds;
                        });

                        auto numFPT = nFPT.getVal();
                        offset = hess4.increaseCount(numFPT);
                        pol(zs::range(numFPT), [fricPT = proxy<space>({}, fricPT), FPT = proxy<space>(FPT),
                                                hess4 = proxy<space>(hess4), offset] ZS_LAMBDA(int fpti) mutable {
                            auto H = fricPT.pack(dim_c<12, 12>, "H", fpti);
                            auto inds = FPT[fpti];
                            hess4.hess[offset + fpti] = H;
                            hess4.inds[offset + fpti] = inds;
                        });

                        auto numFEE = nFEE.getVal();
                        offset = hess4.increaseCount(numFEE);
                        pol(zs::range(numFEE), [fricEE = proxy<space>({}, fricEE), FEE = proxy<space>(FEE),
                                                hess4 = proxy<space>(hess4), offset] ZS_LAMBDA(int feei) mutable {
                            auto H = fricEE.pack(dim_c<12, 12>, "H", feei);
                            auto inds = FEE[feei];
                            hess4.hess[offset + feei] = H;
                            hess4.inds[offset + feei] = inds;
                        });
                    } // self friction
                }     //fricmu
            }         //enable friction
        }             //enable contact

        // ground contact
        if (enableGround) {
            for (auto &primHandle : prims) {
                if (primHandle.isBoundary()) // skip soft boundary
                    continue;
                const auto &svs = primHandle.getSurfVerts();

                pol(zs::range(svs.size()),
                    [svtemp = proxy<space>({}, primHandle.svtemp), svs = proxy<space>({}, svs),
                     svOffset = primHandle.svOffset, hess1 = proxy<space>(hess1), execTag] __device__(int svi) mutable {
                        const auto vi = reinterpret_bits<int>(svs("inds", svi)) + svOffset;
                        auto pbHess = svtemp.pack(dim_c<3, 3>, "H", svi);
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j)
                                atomic_add(execTag, &hess1.hess[vi](i, j), (float)pbHess(i, j));
                        // hess1.hess[i] = Hi;
                    });
            }
        }

        // constraint hessian
        if (!BCsatisfied) {
            pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), hess1 = proxy<space>(hess1),
                                     boundaryKappa = boundaryKappa, execTag] __device__(int vi) mutable {
                auto w = vtemp("ws", vi);
                int BCfixed = vtemp("BCfixed", vi);
                if (!BCfixed) {
                    int BCorder = vtemp("BCorder", vi);
                    for (int d = 0; d != BCorder; ++d)
                        atomic_add(execTag, &hess1.hess[vi](d, d), (float)(boundaryKappa * w));
                }
            });
        }
    }
}

void IPCSystem::compactHessian(zs::CudaExecutionPolicy &pol) {
    using CsrT = RM_CVREF_T(linMat);
    using T = CsrT::value_type;
    using Tn = CsrT::size_type;
    using table_type = CsrT::table_type;
    auto &ap = linMat.ap;
    auto &aj = linMat.aj;
    auto &ax = linMat.ax;
    auto &nnz = linMat.nnz;
    auto &tab = linMat.tab;
    const auto numExpectedEntries = numDofs * 8;
    if (linMat.nrows != numDofs) { // init csr mat
        linMat.nrows = linMat.ncols = numDofs;
        ap = zs::Vector<Tn>{vtemp.get_allocator(), numDofs + 1};
        aj = zs::Vector<int>{vtemp.get_allocator(), numExpectedEntries};
        ax = zs::Vector<T>{vtemp.get_allocator(), numExpectedEntries};
        nnz = zs::Vector<Tn>{vtemp.get_allocator(), numDofs + 1};
        tab = table_type{vtemp.get_allocator(), numExpectedEntries};
    }
    nnz.reset(0);
    tab.reset(true);
}

} // namespace zeno