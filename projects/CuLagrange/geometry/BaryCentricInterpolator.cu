#include "hip/hip_runtime.h"
#include "kernel/bary_centric_weights.hpp"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

#include <iostream>

namespace zeno{

using T = float;
using vec3 = zs::vec<T,3>;
using vec4 = zs::vec<T,4>;
using mat3 = zs::vec<T,3,3>;
using mat4 = zs::vec<T,4,4>;

struct ZSComputeBaryCentricWeights : INode {
    void apply() override {
        using namespace zs;

        // fmt::print("ENTERING NODES\n");
        // std::cout << "ENTERING NODES" << std::endl;

        auto zsvolume = get_input<ZenoParticles>("zsvolume");
        auto zssurf = get_input<ZenoParticles>("zssurf");
        // the bvh of zstets
        // auto lbvh = get_input<zeno::LBvh>("lbvh");
        auto thickness = get_param<float>("bvh_thickness");
        auto fitting_in = get_param<int>("fitting_in");

        auto bvh_channel = get_param<std::string>("bvh_channel");
        auto tag = get_param<std::string>("tag");

        const auto& verts = zsvolume->getParticles();
        const auto& eles = zsvolume->getQuadraturePoints();

        const auto& everts = zssurf->getParticles();
        const auto& e_eles = zssurf->getQuadraturePoints();

        auto &bcw = (*zsvolume)[tag];
        bcw = typename ZenoParticles::particles_t({{"inds",1},{"w",4},{"cnorm",1}},everts.size(),zs::memsrc_e::device,0);

        auto cudaExec = zs::cuda_exec();
        const auto numFEMVerts = verts.size();
        const auto numFEMEles = eles.size();
        const auto numEmbedVerts = bcw.size();
        const auto numEmbedEles = e_eles.size();
        constexpr auto space = zs::execspace_e::cuda;

        // fmt::print("TRY COMPUTE BARYCENTRIC WEIGHTS\n");

        // std::cout << "TRY COMPUTE BARYCENTRIC WEIGHTS" << std::endl;


        // cudaExec(zs::range(eles.size()),
        //     [eles = proxy<space>({},eles)] __device__(int ei) mutable {
        //         auto quad = eles.template pack<4>("inds", ei).template reinterpret_bits<int>();
        //         if(quad[0] < 0 || quad[1] < 0 || quad[2] < 0 || quad[3] < 0)
        //             printf("invalid quad : %d %d %d %d\n",quad[0],quad[1],quad[2],quad[3]);
        //         if(quad[0] > 13572 || quad[1] > 13572 || quad[2] > 13572 || quad[3] > 13572)
        //             printf("invalid quad : %d %d %d %d\n",quad[0],quad[1],quad[2],quad[3]);
        // });

        compute_barycentric_weights(cudaExec,verts,eles,everts,"x",bcw,"inds","w",thickness,fitting_in);
        // set_output("zsvolume", zsvolume);return;

        // fmt::print("FINISH COMPUTING BARYCENTRIC WEIGHTS\n");

        cudaExec(zs::range(numEmbedVerts),
            [bcw = proxy<space>({},bcw),fitting_in] ZS_LAMBDA(int vi) mutable {
                auto idx = reinterpret_bits<int>(bcw("inds",vi));
                if(fitting_in && idx < 0)
                    printf("Unbind vert %d under fitting-in mode\n",vi);
            }
        );

        auto e_dim = e_eles.getPropertySize("inds");

        cudaExec(zs::range(numEmbedVerts),
            [bcw = proxy<space>({},bcw)] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(bcw)::value_type;
                bcw("cnorm",vi) = (T)0.;
        });

        zs::Vector<T> nmEmbedVerts(eles.get_allocator(),eles.size());
        cudaExec(zs::range(eles.size()),[nmEmbedVerts = proxy<space>(nmEmbedVerts)]
            ZS_LAMBDA(int ei) mutable{
                using T = typename RM_CVREF_T(bcw)::value_type;
                nmEmbedVerts[ei] = (T)0.;
        });

        if(e_dim !=3 && e_dim !=4) {
            throw std::runtime_error("INVALID EMBEDDED PRIM TOPO");
        }

        cudaExec(zs::range(bcw.size()),
            [everts = proxy<space>({},everts),bcw = proxy<space>({},bcw),execTag = wrapv<space>{},nmEmbedVerts = proxy<space>(nmEmbedVerts)]
                ZS_LAMBDA (int vi) mutable {
                    using T = typename RM_CVREF_T(bcw)::value_type;
                    auto ei = reinterpret_bits<int>(bcw("inds",vi));
                    if(ei < 0)
                        return;
                    atomic_add(execTag,&nmEmbedVerts[ei],(T)1.0);                  
        });

        cudaExec(zs::range(bcw.size()),
            [bcw = proxy<space>({},bcw),nmEmbedVerts = proxy<space>(nmEmbedVerts)] 
                ZS_LAMBDA(int vi) mutable{
                    auto ei = reinterpret_bits<int>(bcw("inds",vi));
                    if(ei < 0)
                        bcw("cnorm",vi) = (T)0.0;
                    else
                        bcw("cnorm",vi) = (T)1.0/(T)nmEmbedVerts[ei];
        });


        set_output("zsvolume", zsvolume);
    }
};

ZENDEFNODE(ZSComputeBaryCentricWeights, {{{"interpolator","zsvolume"}, {"embed surf", "zssurf"}},
                            {{"interpolator on gpu", "zsvolume"}},
                            {{"float","bvh_thickness","0"},{"int","fitting_in","1"},{"string","tag","skin_bw"},{"string","bvh_channel","x"}},
                            {"ZSGeometry"}});



struct ZSSampleEmbedVectorField : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zstets = get_input<ZenoParticles>("volume");
        auto sampler = get_input<ZenoParticles>("vec_field");

        auto tag = get_param<std::string>("bcw_channel");
        auto sample_attr = get_param<std::string>("sampleAttr");
        auto out_attr = get_param<std::string>("outAttr");
        auto tag_type = get_param<std::string>("type");

        auto cudaExec = zs::cuda_exec();

        auto& verts = zstets->getParticles();
        if(!verts.hasProperty(out_attr))
            verts.append_channels(cudaExec,{{out_attr,3}});

        const auto& sample_verts = sampler->getParticles();
        const auto& sample_eles = sampler->getQuadraturePoints();
  
        if(!sampler->hasAuxData(tag)){
            fmt::print("no specified bcw channel detected, create a new one...\n");
            auto& sample_bcw = (*sampler)[tag];
            sample_bcw = typename ZenoParticles::particles_t({{"inds",1},{"w",4}},verts.size(),zs::memsrc_e::device,0);
        }
        const auto& sample_bcw = (*sampler)[tag];

        if(sample_bcw.size() != verts.size())
            throw std::runtime_error("SMAPLE_BCW SIZE UNEQUAL VOLUME SIZE");
        
        constexpr auto space = zs::execspace_e::cuda;

        auto default_val = vec3::from_array(get_input<zeno::NumericObject>("default")->get<zeno::vec3f>());
        bool on_elm = tag_type == "element";

        cudaExec(zs::range(sample_bcw.size()),
            [sample_bcw = proxy<space>({},sample_bcw),verts = proxy<space>({},verts),sample_eles = proxy<space>({},sample_eles),sample_verts = proxy<space>({},sample_verts),
                sample_attr = zs::SmallString(sample_attr),out_attr = zs::SmallString(out_attr),default_val,on_elm] ZS_LAMBDA(int vi) mutable {
                    auto ei = reinterpret_bits<int>(sample_bcw("inds",vi));
                    if(ei < 0){
                        verts.template tuple<3>(out_attr,vi) = default_val;
                        return;
                    }
                    if(on_elm){
                        verts.template tuple<3>(out_attr,vi) = sample_eles.template pack<3>(sample_attr,ei);
                        return;
                    }

                    const auto& w = sample_bcw.pack<4>("w",vi);
                    verts.template tuple<3>(out_attr,vi) = vec3::zeros();
                    for(int i = 0;i < 4;++i){
                        auto idx = sample_eles.template pack<4>("inds",ei).template reinterpret_bits<int>()[i];
                        verts.template tuple<3>(out_attr,vi) = verts.template pack<3>(out_attr,vi) + w[i] * sample_verts.template pack<3>(sample_attr,idx);
                    }
        });


        set_output("volume",zstets);
    }

};

ZENDEFNODE(ZSSampleEmbedVectorField, {{{"volume"}, {"embed vec field", "vec_field"},{"default value","default"}},
                            {{"out volume", "volume"}},
                            {{"string","bcw_channel","bcw"},{"string","sampleAttr","vec_field"},{"string","outAttr"," vec_field"},{"enum element vert","type","element"}},
                            {"ZSGeometry"}});

struct ZSSampleEmbedTagField : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zstets = get_input<ZenoParticles>("volume");
        auto sampler = get_input<ZenoParticles>("tag_field");

        auto tag = get_param<std::string>("bcw_channel");
        auto sample_attr = get_param<std::string>("tagAttr");
        auto out_attr = get_param<std::string>("outAttr");
        auto tag_type = get_param<std::string>("type");

        auto default_tag_value = get_param<int>("default");

        auto cudaExec = zs::cuda_exec();

        auto& verts = zstets->getParticles();
        if(!verts.hasProperty(out_attr))
            verts.append_channels(cudaExec,{{out_attr,1}});

        const auto& sample_verts = sampler->getParticles();
        const auto& sample_eles = sampler->getQuadraturePoints();

        if(!sampler->hasAuxData(tag)){
            fmt::print("no specified bcw channel detected, create a new one...\n");
            auto& sample_bcw = (*sampler)[tag];
            sample_bcw = typename ZenoParticles::particles_t({{"inds",1},{"w",4}},verts.size(),zs::memsrc_e::device,0);
        }
        const auto& sample_bcw = (*sampler)[tag];

        if(sample_bcw.size() != verts.size())
            throw std::runtime_error("SMAPLE_BCW SIZE UNEQUAL VOLUME SIZE");
        
        constexpr auto space = zs::execspace_e::cuda;

        bool on_elm = tag_type == "element";

        cudaExec(zs::range(sample_bcw.size()),
            [sample_bcw = proxy<space>({},sample_bcw),verts = proxy<space>({},verts),sample_eles = proxy<space>({},sample_eles),sample_verts = proxy<space>({},sample_verts),
                sample_attr = zs::SmallString(sample_attr),out_attr = zs::SmallString(out_attr),default_tag_value,on_elm] ZS_LAMBDA(int vi) mutable {
                     auto ei = reinterpret_bits<int>(sample_bcw("inds",vi));
                     if(ei < 0){
                         verts(out_attr,vi) = reinterpret_bits<float>(default_tag_value);
                         return;
                     }

                    if(on_elm)
                        verts(out_attr,vi) = sample_eles(sample_attr,ei);
                    else{
                        auto idx = sample_eles.pack<4>("inds",ei).reinterpret_bits<int>()[0];
                        verts(out_attr,vi) = sample_verts(sample_attr,idx);
                    }
        });


        set_output("volume",zstets);
    }

};

ZENDEFNODE(ZSSampleEmbedTagField, {{{"volume"}, {"embed tag field", "tag_field"},{"default value","default"}},
                            {{"out volume", "volume"}},
                            {{"string","interpolate_tag","bws"},{"string","sampleAttr","vec_field"},{"string","outAttr"," vec_field"},{"enum element vert","type","element"}},
                            {"ZSGeometry"}});


struct ZSInterpolateEmbedAttr : zeno::INode {
    template<int DIM,typename SRC_TILEVEC,typename DST_TILEVEC,typename TOPO_TIELVEC,typename BCW_TILEVEC>
    void interpolate_p2p_imp(const std::string& srcAttr,const std::string& dstAttr,
            const SRC_TILEVEC& src_tilevec,DST_TILEVEC& dst_tilevec,const TOPO_TIELVEC& src_topo,const BCW_TILEVEC& bcw) {
        auto cudaExec = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        if(!dst_tilevec.hasProperty(dstAttr))
            dst_tilevec.append_channels(cudaExec, {{dstAttr, DIM}});

        cudaExec(zs::range(dst_tilevec.size()),
            [srcAttr = zs::SmallString{srcAttr},dstAttr = zs::SmallString{dstAttr},
                    src_tilevec = zs::proxy<space>({},src_tilevec), bcw = zs::proxy<space>({},bcw),
                    dst_tilevec = zs::proxy<space>({},dst_tilevec),
                    src_topo = zs::proxy<space>({},src_topo)] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(dst_tilevec)::value_type;
                const auto& ei = bcw.template pack<1>("inds",vi).template reinterpret_bits<int>()[0];
                if(ei < 0)
                    return;
                const auto& inds = src_topo.template pack<4>("inds",ei).template reinterpret_bits<int>();

                const auto& w = bcw.template pack<4>("w",vi);
                dst_tilevec.template tuple<DIM>(dstAttr,vi) = zs::vec<T,DIM>::zeros();
                for(size_t i = 0;i < 4;++i){
                    auto idx = inds[i];
                    dst_tilevec.template tuple<DIM>(dstAttr,vi) = dst_tilevec.template pack<DIM>(dstAttr,vi) + w[i] * src_tilevec.template pack<DIM>(srcAttr, idx);
                }

        });
    }

    template<int DIM,typename QUAD_TILEVEC,typename POINT_TILEVEC,typename BCW_TILEVEC>
    void interpolate_q2p_imp(const std::string& quadAttr,const std::string& pointAttr,
            const QUAD_TILEVEC& quad_tilevec,POINT_TILEVEC& point_tilevec,const BCW_TILEVEC& bcw) {
        auto cudaExec = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        if(!point_tilevec.hasProperty(pointAttr))
            point_tilevec.append_channels(cudaExec, {{pointAttr, DIM}});   
             
        cudaExec(zs::range(point_tilevec.size()),
            [pointAttr = zs::SmallString{pointAttr},quadAttr = zs::SmallString{quadAttr},
                point_tilevec = zs::proxy<space>({},point_tilevec),bcw = zs::proxy<space>({},bcw),
                quad_tilevec = zs::proxy<space>({},quad_tilevec)] ZS_LAMBDA (int vi) mutable {
            using T = typename RM_CVREF_T(point_tilevec)::value_type;
            const auto& ei = bcw.template pack<1>("inds",vi).template reinterpret_bits<int>()[0];
            if(ei < 0)
                return;
            point_tilevec.template tuple<DIM>(pointAttr,vi) = quad_tilevec.template pack<DIM>(quadAttr,ei);
        });
    }


    void apply() override {
        using namespace zs;
        auto source = get_input<ZenoParticles>("source");
        auto dest = get_input<ZenoParticles>("dest");

        auto srcAttr = get_param<std::string>("srcAttr");
        auto dstAttr = get_param<std::string>("dstAttr");
        auto bcw_tag = get_param<std::string>("bcw_tag");
        auto strategy = get_param<std::string>("strategy");
        const auto& bcw = (*source)[bcw_tag];
        auto& dest_pars = dest->getParticles();

        if(bcw.size() != dest_pars.size()) {
            fmt::print("the dest and bcw's size not match\n");
            throw std::runtime_error("the dest and bcw's size not match");
        }

        
        if(strategy == "p2p") {
            const auto& source_pars = source->getParticles();
            const auto& topo = source->getQuadraturePoints();
            if(!source_pars.hasProperty(srcAttr)) {
                fmt::print("the source have no {} channel\n",srcAttr);
                throw std::runtime_error("the source have no specified channel");
            }           
            if(topo.getChannelSize("inds") != 4) {
                fmt::print("only support tetrahedra mesh as source\n");
                throw std::runtime_error("only support tetrahedra mesh as source");
            }
            if(dest_pars.hasProperty(dstAttr) && dest_pars.getChannelSize(dstAttr) != source_pars.getChannelSize(srcAttr)){
                fmt::print("the dest attr_{} and source attr_{} not match in size\n",dstAttr,srcAttr);
                throw std::runtime_error("the dest attr and source attr not match in size");
            }

            if(source_pars.getChannelSize(srcAttr) == 1)
                interpolate_p2p_imp<1>(srcAttr,dstAttr,source_pars,dest_pars,topo,bcw);
            if(source_pars.getChannelSize(srcAttr) == 2)
                interpolate_p2p_imp<2>(srcAttr,dstAttr,source_pars,dest_pars,topo,bcw);
            if(source_pars.getChannelSize(srcAttr) == 3)
                interpolate_p2p_imp<3>(srcAttr,dstAttr,source_pars,dest_pars,topo,bcw);
        }else if(strategy == "q2p") {
            const auto& source_quads = source->getQuadraturePoints();
            if(!source_quads.hasProperty(srcAttr)) {
                fmt::print("the source have no {} channel\n",srcAttr);
                throw std::runtime_error("the source have no specified channel");
            }    
            if(dest_pars.hasProperty(dstAttr) && dest_pars.getChannelSize(dstAttr) != source_quads.getChannelSize(srcAttr)){
                fmt::print("the dest attr_{} and source attr_{} not match in size\n",dstAttr,srcAttr);
                throw std::runtime_error("the dest attr and source attr not match in size");
            }

            if(source_quads.getChannelSize(srcAttr) == 1)
                interpolate_q2p_imp<1>(srcAttr,dstAttr,source_quads,dest_pars,bcw);
            if(source_quads.getChannelSize(srcAttr) == 2)
                interpolate_q2p_imp<2>(srcAttr,dstAttr,source_quads,dest_pars,bcw);
            if(source_quads.getChannelSize(srcAttr) == 3)
                interpolate_q2p_imp<3>(srcAttr,dstAttr,source_quads,dest_pars,bcw);
        }
        set_output("dest",dest);
    }
};


ZENDEFNODE(ZSInterpolateEmbedAttr, {{{"source"}, {"dest"}},
                            {{"dest"}},
                            {
                                {"string","srcAttr","x"},
                                {"string","dstAttr","x"},
                                {"string","bcw_tag","skin_bw"},
                                {"enum p2p q2p","strategy","p2p"}

                            },
                            {"ZSGeometry"}});

// deprecated
struct ZSInterpolateEmbedPrim : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zstets = get_input<ZenoParticles>("zsvolume");
        auto zssurf = get_input<ZenoParticles>("zssurf");

        auto tag = get_param<std::string>("tag");
        auto inAttr = get_param<std::string>("inAttr");
        auto outAttr = get_param<std::string>("outAttr");
        // auto refAttr = get_param<std::string>("refAttr");

        // auto useDispMap = get_param<int>("useDispMap");
        // auto refDispMapTag = get_param<std::string>("refDispMapTag");
        // auto outDispMapTag = get_param<std::string>("outDispMapTag");

        // auto use_xform = get_param<int>("use_xform");

        auto &everts = zssurf->getParticles();
    
        const auto& verts = zstets->getParticles();
        const auto& eles = zstets->getQuadraturePoints();
        const auto& bcw = (*zstets)[tag];

        // if(useDispMap && (!everts.hasProperty(refDispMapTag) || !everts.hasProperty(outDispMapTag))) {
        //     fmt::print("the input everts have no {} or {} dispMap when useDispMap is on\n",refDispMapTag,outDispMapTag);
        //     throw std::runtime_error("the input everts have no specified dispMap when useDispMap is on");
        // }


        // if(use_xform && !everts.hasProperty(refAttr)) {
        //     fmt::print("the input everts have no {} channel when use_xform is on\n",refAttr);
        //     throw std::runtime_error("the input everts have no refAttr channel when use_xform is on");
        // }
        // if(use_xform && !verts.hasProperty(refAttr)) {
        //     fmt::print("the input verts have no {} channel when use_xform is on\n",refAttr);
        //     throw std::runtime_error("the input verts have no refAttr channel when use_xform is on");
        // }

        const auto nmEmbedVerts = bcw.size();
        if(everts.size() != nmEmbedVerts)
            throw std::runtime_error("INPUT SURF SIZE AND BCWS SIZE DOES NOT MATCH");


        auto cudaExec = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        cudaExec(zs::range(nmEmbedVerts),
            [inAttr = zs::SmallString{inAttr},outAttr = zs::SmallString{outAttr},
                    verts = proxy<space>({},verts),eles = proxy<space>({},eles),
                    bcw = proxy<space>({},bcw),everts = proxy<space>({},everts)
                    // use_xform,refAttr = zs::SmallString{refAttr},
                    // useDispMap,
                    // refDispMapTag = zs::SmallString{refDispMapTag},
                    // outDispMapTag = zs::SmallString{outDispMapTag}
                    ] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(verts)::value_type;
                const auto& ei = bcw.pack<1>("inds",vi).reinterpret_bits<int>()[0];
                if(ei < 0)
                    return;
                const auto& inds = eles.template pack<4>("inds",ei).template reinterpret_bits<int>();
                // if(use_xform || useDispMap) {
                //     zs::vec<T,3,3> F{};
                //     zs::vec<T,3> b{};

                //     LSL_GEO::deformation_xform(
                //         verts.template pack<3>(inAttr,inds[0]),
                //         verts.template pack<3>(inAttr,inds[1]),
                //         verts.template pack<3>(inAttr,inds[2]),
                //         verts.template pack<3>(inAttr,inds[3]),
                //         verts.template pack<3>(refAttr,inds[0]),
                //         eles.template pack<3,3>("IB",ei),F,b);
                    
                //     everts.template tuple<3>(outAttr,vi) = F * everts.template pack<3>(refAttr,vi) + b;

                //     // if(vi == 0){
                //     //     printf("F : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
                //     //         (float)F(0,0),(float)F(0,1),(float)F(0,2),
                //     //         (float)F(1,0),(float)F(1,1),(float)F(1,2),
                //     //         (float)F(2,0),(float)F(2,1),(float)F(2,2));
                //     //     printf("b : %f %f %f\n",(float)b[0],(float)b[1],(float)b[2]);
                //     // }

                //     if(useDispMap) {
                //         everts.template tuple<3>(outDispMapTag,vi) = F * everts.template pack<3>(refDispMapTag,vi);
                //     }
                // }else{
                    const auto& w = bcw.pack<4>("w",vi);
                    everts.tuple<3>(outAttr,vi) = vec3::zeros();
                    for(size_t i = 0;i < 4;++i){
                        // const auto& idx = eles.pack<4>("inds",ei).reinterpret_bits<int>()[i];
                        // const auto idx = reinterpret_bits<int>(eles("inds", i, ei));
                        auto idx = inds[i];
                        everts.tuple<3>(outAttr,vi) = everts.pack<3>(outAttr,vi) + w[i] * verts.pack<3>(inAttr, idx);
                    }
#if 0
                    if(vi == 100){
                        auto vert = everts.pack<3>(outAttr,vi);
                        printf("V<%d>->E<%d>(%f,%f,%f,%f) :\t%f\t%f\t%f\n",vi,ei,w[0],w[1],w[2],w[3],vert[0],vert[1],vert[2]);
                    }
#endif

                // }
        });
        set_output("zssurf",zssurf);
    }
};

ZENDEFNODE(ZSInterpolateEmbedPrim, {{{"zsvolume"}, {"embed primitive", "zssurf"}},
                            {{"embed primitive", "zssurf"}},
                            {
                                {"string","inAttr","x"},
                                {"string","outAttr","x"},
                                // {"string","refAttr","X"},
                                {"string","tag","skin_bw"}
                                // {"int","use_xform","0"},
                                // {"int","useDispMap","0"},
                                // {"string","refDispMapTag","dX"},
                                // {"string","outDispMapTag","dx"}
                                },
                            {"ZSGeometry"}});


struct ZSDeformEmbedPrim : zeno::INode {
    void apply() override {
        using namespace zs;
        auto zsvolume = get_input<ZenoParticles>("zsvolume");
        auto zssurf = get_input<ZenoParticles>("zssurf");

        auto tag = get_param<std::string>("tag");
        auto inAttr = get_param<std::string>("inAttr");
        auto outAttr = get_param<std::string>("outAttr");

        auto deformField = get_param<std::string>("deformField");

        auto &everts = zssurf->getParticles();

        auto cudaExec = zs::cuda_exec();

        if(!everts.hasProperty(inAttr)) {
            fmt::print("the embed prim has no {} attribute as input\n",inAttr);
            throw std::runtime_error("the embed prim has no attribute as input");
        }
        if(!everts.hasProperty(outAttr))
            everts.append_channels(cudaExec, {{outAttr, 3}});

        
        const auto& verts = zsvolume->getParticles();
        const auto& eles = zsvolume->getQuadraturePoints();
        const auto& bcw = (*zsvolume)[tag];

        if(!eles.hasProperty(deformField)) {
            fmt::print("the embed prim has no {} deformField\n",deformField);
            throw std::runtime_error("the embed prim has no deformField");
        }

        const auto nmEmbedVerts = bcw.size();

        if(everts.size() != nmEmbedVerts)
            throw std::runtime_error("INPUT SURF SIZE AND BCWS SIZE DOES NOT MATCH");


        constexpr auto space = zs::execspace_e::cuda;

        cudaExec(zs::range(nmEmbedVerts),
            [inAttr = zs::SmallString{inAttr},outAttr = zs::SmallString{outAttr},
                    everts = proxy<space>({},everts),eles = proxy<space>({},eles),
                    bcw = proxy<space>({},bcw),
                    deformField = zs::SmallString{deformField}] ZS_LAMBDA (int vi) mutable {
                using T = typename RM_CVREF_T(verts)::value_type;
                const auto& ei = bcw.pack<1>("inds",vi).reinterpret_bits<int>()[0];
                if(ei < 0)
                    return;
                everts.template tuple<3>(outAttr,vi) = eles.template pack<3,3>(deformField,ei) * everts.template pack<3>(inAttr,vi);
                // if(vi == 114754){
                //     auto dx = everts.template pack<3>(outAttr,vi);
                //     auto dX = everts.template pack<3>(inAttr,vi);
                //     auto F = eles.template pack<3,3>(deformField,ei);
                //     printf("F : %f %f %f\n%f %f %f\n%f %f %f\n",
                //         (float)F(0,0),(float)F(0,1),(float)F(0,2),
                //         (float)F(1,0),(float)F(1,1),(float)F(1,2),
                //         (float)F(2,0),(float)F(2,1),(float)F(2,2)
                //     );
                //     printf("Fdet : %f\n",(float)zs::determinant(F));
                //     printf("dX : %f %f %f with length %f\n",(float)dX[0],(float)dX[1],(float)dX[2],(float)dX.norm());
                //     printf("dx : %f %f %f with length %f\n",(float)dx[0],(float)dx[1],(float)dx[2],(float)dx.norm());
                // }

        });
        set_output("zssurf",zssurf);
        set_output("zsvolume",zsvolume);
    }
};

ZENDEFNODE(ZSDeformEmbedPrim, {{{"zsvolume"}, {"embed primitive", "zssurf"}},
                            {{"embed primitive", "zssurf"},{"zsvolume"}},
                            {
                                {"string","inAttr","V"},
                                {"string","outAttr","v"},
                                {"string","tag","skin_bw"},
                                {"string","deformField","F"}
                                },
                            {"ZSGeometry"}});

} // namespace zeno